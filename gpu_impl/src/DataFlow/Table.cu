#include "hip/hip_runtime.h"
#include <string>
#include <sstream>
#include <Util/Util.hpp>
#include "DataFlow/Table.hpp"
#include "../../inc/DataFlow/Table.hpp"

std::string Table::toString(uint32_t print_row_num, bool raw) {
  print_row_num = print_row_num < row_num ? print_row_num : row_num;
  std::stringstream ss;
  for (uint32_t i = 0; i < print_row_num; i++) {
    ss << "row[" << std::to_string(i) << "]: [";
    for (Column *column: columns) {
      switch (column->type) {
        case LONG: {
          ss << ((long *) column->data_ptr)[i];
          break;
        }
        case DOUBLE: {
          ss << ((double *) column->data_ptr)[i];
          break;
        }
        case INT: {
          ss << ((int32_t *) column->data_ptr)[i];
          break;
        }
        case STRING: {
          char *chars = (char *) column->data_ptr;
          int32_t *indices = (int32_t *) column->data_ptr_aux;
          if (raw) {
            for (int j = indices[2 * i]; j < indices[2 * i + 1]; j ++) {
              ss << (int)chars[j] << ";";
            }
          } else {
            ss << std::string(chars + indices[2 * i], indices[2 * i + 1] - indices[2 * i]);
          }
          break;
        }
        case DEPEND:break;
      }
      if (column != columns.back()) {
        ss << ", ";
      }
    }
    ss << "]\n";
  }
  return ss.str();
}

std::string Table::toString() {
  std::stringstream ss;
  ss << "row number: " << row_num << "\n";
  ss << "column types: ";
  for (uint32_t i = 0; i < columns.size(); i ++) {
    ss << columns[i]->type;
    if (i != columns.size() - 1) {
      ss << "; ";
    }
  }
  ss << std::endl;
  ss << toString(min(20, row_num));
  return ss.str();
}

Table::Table(std::vector<ColumnType> _types, uint32_t _row_num) : types(_types), row_num(_row_num) {

}

Table::Table() {}

std::string Table::rowToString(uint32_t row_idx) {
  std::stringstream ss;
  ss << "row[" << std::to_string(row_idx) << "]: [";
  for (Column *column: columns) {
    switch (column->type) {
      case LONG: {
        ss << std::to_string(((long *) column->data_ptr)[row_idx]);
        break;
      }
      case DOUBLE: {
        ss << std::to_string(((double *) column->data_ptr)[row_idx]);
        break;
      }
      case INT: {
        ss << std::to_string(((int32_t *) column->data_ptr)[row_idx]);
        break;
      }
      case STRING: {
        char *chars = (char *) column->data_ptr;
        int32_t *indices = (int32_t *) column->data_ptr_aux;
        ss << std::string(chars + indices[2 * row_idx],
                          indices[2 * row_idx + 1] - indices[2 * row_idx]);
        break;
      }
      case DEPEND:break;
    }
    if (column != columns.back()) {
      ss << ", ";
    }
  }
  ss << "]\n";
  return ss.str();
}
