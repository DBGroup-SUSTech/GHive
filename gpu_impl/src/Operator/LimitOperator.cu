#include "hip/hip_runtime.h"
#include <Operator/LimitOperator.hpp>

void LimitOperator::execute() {
  for (auto child: this->children) {
    child->execute();
  }
  std::cout << "GHive-CPP [LimitOperator-process]: " << this->operator_name << " starts to execute, with input" <<
            std::endl << children[0]->op_result->toString() << std::endl;

  this->op_result = children[0]->op_result;
  this->op_result->row_num = min(limit_num, children[0]->op_result->row_num);

  std::cout << "GHive-CPP [LimitOperator-process]: " << this->operator_name << " ends to execute, with input" <<
            std::endl << children[0]->op_result->toString() << std::endl;
}

std::string LimitOperator::toString() {
  std::string ret = "limit_num: " + std::to_string(limit_num);
  return Operator::toString() + " " + ret;
}

std::string LimitOperator::toString(int level) {
  std::string ret;
  for (int i = 0; i < level; i++) {
    ret += " ";
  }
  ret += this->toString() + "\n";
  return ret;
}
