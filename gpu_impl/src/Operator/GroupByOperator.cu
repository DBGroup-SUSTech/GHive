#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include "Operator/GroupByOperator.hpp"
#include "Util/Util.hpp"

GroupByOperator::GroupByOperator(std::string name,
                                 std::vector<std::string> cols,
                                 std::vector<std::string> aggs,
                                 std::vector<std::string> keys) {
  this->operator_name = name;
  this->output_cols = cols;
  this->aggs = aggs;
  this->keys = keys;
}

void GroupByOperator::parseExtended() {
  std::vector<uint32_t> offset_keys;
  std::smatch match_result;
  for (std::string str_key: keys) {
    if (std::regex_match(children[0]->operator_name, std::regex("RS_[0-9]+")) ||
        std::regex_match(children[0]->operator_name, std::regex("Union [0-9]+"))
        ) {
      if (std::regex_match(str_key, match_result,
                           std::regex("KEY.*([0-9]+)"))) {
        int key_offset = std::stoi(match_result[1]);
        offset_keys.push_back(key_offset);
      } else if (std::regex_match(str_key, match_result,
                                  std::regex("VALUE.*([0-9]+)"))) {
        int value_offset = std::stoi(match_result[1]);
        offset_keys.push_back(value_offset +
            keys.size());
      }
    } else {
      for (uint32_t i = 0; i < children[0]->output_cols.size(); i++) {
        if (str_key == children[0]->output_cols[i]) {
          offset_keys.push_back(i);
          break;
        }
      }
    }
  }

  GroupByPredicate group_by_desc;
  group_by_desc.keys = offset_keys;
  this->predicate = group_by_desc;

  for (std::string agg_str: aggs) {
    std::cout << "GHive-CPP [GroupByOperator-parseExtended]: aggstr: aggs: = " << agg_str << std::endl;
    if (std::regex_match(agg_str, match_result, std::regex("(.*)\\((.*)\\)"))) {
      std::string agg_type_str = match_result[1];
      std::string agg_col = match_result[2];
      std::cout << "GHive-CPP [GroupByOperator-parseExtended]: agg_str=" << agg_str
                << " agg_type_str=" << agg_type_str
                << " agg_col=" << agg_col << std::endl;
      AggregationType agg_type = UNKNOWN;
      uint32_t aggregation_col = 0xffffffff;

      // TODO: More aggregation types to be added.
      // TODO: literal details should be justified.
      if (agg_type_str == "sum") {
        agg_type = SUM;
      } else if (agg_type_str == "max") {
        agg_type = MAX;
      } else if (agg_type_str == "min") {
        agg_type = MIN;
      } else if (agg_type_str == "avg") {
        agg_type = AVG;
      } else if (agg_type_str == "count") {
        agg_type = CNT;
      }

      if (std::regex_match(agg_col, match_result,
                           std::regex("KEY.*([0-9]+)"))) {
        int key_offset = std::stoi(match_result[1]);

        aggregation_col = key_offset;
      } else if (std::regex_match(agg_col, match_result,
                                  std::regex("VALUE.*([0-9]+)"))) {
        int value_offset = std::stoi(match_result[1]);
        std::cout << "GHive-CPP [GroupByOperator-parseExtended]: value_offset: " << value_offset << endl;
        aggregation_col = value_offset + children[0]->op_result->key_num;
      } else {
        for (uint32_t i = 0; i < children[0]->output_cols.size(); i++) {
          std::cout << "GHive-CPP [GroupByOperator-parseExtended]: children[0]'s output_col[i]="
                    << children[0]->output_cols[i] << std::endl;
          if (agg_col == children[0]->output_cols[i]) {
            aggregation_col = i;
            break;
          }
        }
      }
      if (aggregation_col == 0xffffffff) { // empty key
        aggregation_col = 0; // default: column 0
      }
      AggregationDesc aggregation_desc(agg_type, aggregation_col);
      this->predicate.aggregation_descs.push_back(aggregation_desc);
    }
  }
}

void GroupByOperator::execute() {
  std::cout << "GHive-CPP [GroupByOperator-execute]: inside operator_name: "
            << operator_name << std::endl;
  if (children.size() != 0) {
    for (auto child: this->children) {
      child->execute();
    }
  }
  long start = profiler_millis_timestamp();
  profiler.start_op();
  std::cout << "Operator [" + operator_name + "] starts at: " << profiler_millis_timestamp() << std::endl;
  assert(this->children.size() == 1);
  Table *input_tbl = children[0]->op_result;
  std::cout << "GHive-CPP [GroupByOperator-execute]: " << operator_name << " starts to execute with input "
            << input_tbl->toString() << std::endl;

  op_result = predicate.gpu_execute(input_tbl, profiler);

  std::cout << "GHive-CPP [GroupByOperator-execute] " << operator_name << " ends to execute, with result: "
            << op_result->toString() << std::endl;
  std::cout << "Operator [" + operator_name + "] ends at: " << profiler_millis_timestamp() << std::endl;
  long end = profiler_millis_timestamp();
  std::cout << "Operator [" + operator_name + "] takes time: " << (end - start) << std::endl;
  std::cout << "Operator [" + operator_name + "-pcie_h2d" + "] takes time: " << profiler.total_pcie_h2d << std::endl;
  std::cout << "Operator [" + operator_name + "-exe" + "] takes time: " << profiler.gpu_exec_total << std::endl;
  std::cout << "Operator [" + operator_name + "-pcie_d2h" + "] takes time: " << profiler.total_pcie_d2h << std::endl;
  std::cout << "Operator [" + operator_name + "-recover" + "] takes time: " << profiler.data_recover_total << std::endl;
  profiler.end_op();
  std::cout << profiler.toString() << std::endl;
}

std::string GroupByOperator::toString() {
  return "[" + operator_name + "]" + "; " +
      "aggs: " + vector_to_string(aggs, "") +
      "keys: " + vector_to_string(keys, "") +
      "output_cols: " + vector_to_string(output_cols, "");
}

std::string GroupByOperator::toString(int level) {
  std::string ret = "";
  for (int i = 0; i < level; i++) {
    ret += "  ";
  }
  ret += this->toString() + "\n";
  for (auto child: children) {
    ret += child->toString(level + 1);
  }
  return ret;
}

