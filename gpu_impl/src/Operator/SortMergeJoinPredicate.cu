#include "hip/hip_runtime.h"
#include <list>
#include <thrust/device_vector.h>

#include "Operator/SortMergeJoinPredicate.hpp"
#include "Operator/GroupByPredicate.hpp"
#include "Util/Util.hpp"

Table *SortMergeJoinPredicate::process(const std::vector<Table *> &tables) {
  profiler.start_op();


  std::cout << "GHive-CPP [SortMergeJoinPredicate-process]: process starts with " << tables.size() << " tables." << std::endl;
  // 1. Sort each data flow, get sorted index.
  // 1.1 extract the key columns and copy them to GPU.
  uint32_t **sorted_index = new uint32_t *[join_cols.size()];

  for (uint32_t i = 0; i < join_cols.size(); i++) {

    profiler.start_pci_host2device();
    std::vector<uint32_t> each_tbl_cols = join_cols[i]; // keys cols for table i
    std::cout << "table [" << i << "]: " << tables[i]->toString() << std::endl;
    std::cout << "GHive-CPP [SortMergeJoinPredicate-process]: join_cols[i] "
              << vector_to_string<uint32_t>(each_tbl_cols) << std::endl;
    Table *tbl = tables[i];
    uint32_t key_space_size = each_tbl_cols.size();
    for (uint32_t j = 0; j < each_tbl_cols.size(); j++) {
      if (tbl->columns[each_tbl_cols[j]]->type == STRING) {
        key_space_size ++;
      }
    }
//    profiler.start_gpu_alloc();
    thrust::device_vector<const void *> d_join_keys(key_space_size);
    thrust::device_vector<int> d_keys_type(each_tbl_cols.size());
//    profiler.end_gpu_alloc();
    for (uint32_t j = 0, l = 0; j < each_tbl_cols.size(); j++, l++) {
      // uint32_t ith_col = each_tbl_cols[j];
      const auto &col = tbl->columns[each_tbl_cols[j]];
      switch (col->type) {
        case LONG: {
          d_keys_type[j] = 0;
          long *d_long_ptr;
//          profiler.start_gpu_alloc();
          hipMalloc((void **) &d_long_ptr, col->row_num * sizeof(long));
//          profiler.end_gpu_alloc();
//          profiler.start_pci_host2device();
          hipMemcpy(d_long_ptr, col->data_ptr, col->row_num * sizeof(long), hipMemcpyHostToDevice);
//          profiler.end_pci_host2device();
          d_join_keys[l] = d_long_ptr;
          break;
        }
        case DOUBLE: {
          d_keys_type[j] = 1;
          double *d_double_ptr;
//          profiler.start_gpu_alloc();
          hipMalloc((void **) &d_double_ptr, col->row_num * sizeof(double));
//          profiler.end_gpu_alloc();
//          profiler.start_pci_host2device();
          hipMemcpy(d_double_ptr, col->data_ptr, col->row_num * sizeof(double), hipMemcpyHostToDevice);
//          profiler.end_pci_host2device();
          d_join_keys[l] = d_double_ptr;
          break;
        }
        case INT: {
          d_keys_type[j] = 2;
          int32_t *d_int_ptr;
//          profiler.start_gpu_alloc();
          hipMalloc((void **) &d_int_ptr, col->row_num * sizeof(int));
//          profiler.end_gpu_alloc();
//          profiler.start_pci_host2device();
          hipMemcpy(d_int_ptr, col->data_ptr, tbl->row_num * sizeof(int), hipMemcpyHostToDevice);
//          profiler.end_pci_host2device();
          d_join_keys[l] = d_int_ptr;
          break;
        }
        case STRING: {
          d_keys_type[j] = 3;
          char *d_str_ptr;
          int32_t *d_str_idx_ptr;
          uint32_t size_char = 0;
          for (uint32_t x = 0; x < col->row_num; x++) {
            size_char = max(static_cast<int32_t *>(col->data_ptr_aux)[2 * x + 1], size_char);
          }
//          profiler.start_gpu_alloc();
          hipMalloc((void **) &d_str_ptr, (size_char + 1) * sizeof(char));
          hipMalloc((void **) &d_str_idx_ptr, 2 * col->row_num * sizeof(int32_t));
//          profiler.end_gpu_alloc();
//          profiler.start_pci_host2device();
          hipMemcpy(d_str_ptr, col->data_ptr, (size_char + 1) * sizeof(char), hipMemcpyHostToDevice);
          hipMemcpy(d_str_idx_ptr, col->data_ptr_aux, 2 * col->row_num * sizeof(int32_t), hipMemcpyHostToDevice);
//          profiler.end_pci_host2device();
          d_join_keys[l++] = d_str_ptr;
          d_join_keys[l++] = d_str_idx_ptr;
          break;
        }
        case DEPEND: break;
        default:break;
      }
    }
//    profiler.start_gpu_alloc();
    const void **d_keys_data_ptr = thrust::raw_pointer_cast(d_join_keys.data());
    const int *d_keys_type_ptr = thrust::raw_pointer_cast(d_keys_type.data());

    thrust::device_vector<int> d_result_idx(tbl->row_num);
    thrust::sequence(d_result_idx.begin(), d_result_idx.end());
//    profiler.start_gpu_alloc();
    profiler.end_pci_host2device();
    // 1.2 invoking thrust::sort_by_key to acquire the sorted index.
    profiler.start_sort();
    profiler.start_gpu_exec();
    thrust::sort(d_result_idx.begin(),
                 d_result_idx.end(),
                 sort_comparator(d_keys_data_ptr, d_keys_type_ptr, each_tbl_cols.size()));
    profiler.end_gpu_exec();
    profiler.end_sort();
//    for (uint32_t x = 0; x < d_result_idx.size(); x++) {
//      std::cout << d_result_idx[x] << " ";
//    }
//    std::cout << std::endl;
    std::cout << "GHive-CPP [SortMergeJoinPredicate-process]: finish sort." << std::endl;
    // 1.3 copy back the result to CPU.
    profiler.start_cpu_alloc();
    uint32_t *result_index = new uint32_t[tbl->row_num];
    profiler.end_cpu_alloc();
    profiler.start_pci_device2host();
    hipMemcpy(result_index,
               thrust::raw_pointer_cast(d_result_idx.data()),
               tbl->row_num * sizeof(int32_t),
               hipMemcpyDeviceToHost);
    profiler.end_pci_device2host();
    sorted_index[i] = result_index;
  }

  std::cout << "GHive-CPP: [BaseJoinPredicate-sm_join]: Sorting ends." << std::endl;

  // 2. Merge the data according to the sorted indexes.

  // 2.1 Calculate the size of each offset pair. Semi Join only maintain 1 column.
  profiler.start_merge();
  uint32_t pair_counter = 0;
  std::vector<int32_t> corresponding_index(join_types.size() + 1);
  for (uint32_t i = 0; i < join_types.size(); i++) {
    uint32_t idx_second = join_types[i].second.second;
    if (i == 0) {
      result_to_tbl.push_back(i);
      corresponding_index[i] = pair_counter++;
    }
    if (join_types[i].first == LEFT_SEMI_JOIN) {
      corresponding_index[idx_second] = -1;
    } else {
      result_to_tbl.push_back(idx_second);
      corresponding_index[idx_second] = pair_counter;
      pair_counter++;
    }
  }
  std::cout << "GHive-CPP: [BaseJoinPredicate-sm_join]: corresponding index: " <<
            vector_to_string<int32_t>(corresponding_index) << std::endl;
  // 2.2 according to the join types, recording the result to tmp vector
  //     push back all the offsets to the result vector (result_offsets) finally.
  //     For Outer Join, -1 represents null.
  // TODO: join_col_size = 0;
  uint32_t join_col_size = join_cols[0].size(); // key的个数

  std::list<std::vector<int32_t>> result_pairs;
  for (uint32_t i = 0; i < join_types.size(); i++) {
    auto join_type = join_types[i];
    uint32_t tbl_idx_first = join_type.second.first;
    uint32_t tbl_idx_second = join_type.second.second;
    uint32_t result_idx_first = corresponding_index[tbl_idx_first];
    uint32_t result_idx_second = corresponding_index[tbl_idx_second];
    Table *tbl_first = tables[tbl_idx_first];
    Table *tbl_second = tables[tbl_idx_second];
    std::cout << "GHive-CPP: [BaseJoinPredicate-sm_join]: tbl_idx_first: "
              << tbl_idx_first << std::endl;
    std::cout << "GHive-CPP: [BaseJoinPredicate-sm_join]: tbl_idx_second: "
              << tbl_idx_second << std::endl;
    uint32_t ptr_alloc_size = join_col_size;
    uint32_t *sorted_idx_1 = sorted_index[tbl_idx_first];
    uint32_t *sorted_idx_2 = sorted_index[tbl_idx_second];
    for (auto idx: join_cols[tbl_idx_first]) {
      const auto &col = tbl_first->columns[idx];
      if (col->type == ColumnType::STRING) {
        ptr_alloc_size++;
      }
    }
    const void **data_ptr_1 = new const void *[ptr_alloc_size];
    const void **data_ptr_2 = new const void *[ptr_alloc_size];
    int *types = new int[join_col_size];
    for (uint32_t j = 0, l = 0; j < join_cols[tbl_idx_first].size(); j++, l++) {
      uint32_t idx = join_cols[tbl_idx_first][j];
      const auto &col = tbl_first->columns[idx];
      // std::cout << "df_first_data: ";
      // for (uint32_t s = 0; s < 4; s++) {
      //   std::cout << tbl_first->longCols[idx][s] << " ";
      // }
      switch (col->type) {
        case LONG: {
          types[j] = 0;
          data_ptr_1[l] = col->data_ptr;
          break;
        }
        case DOUBLE: {
          types[j] = 1;
          data_ptr_1[l] = col->data_ptr;
          break;
        }
        case INT: {
          types[j] = 2;
          data_ptr_1[l] = col->data_ptr;
          break;
        }
        case STRING: {
          types[j] = 3;
          data_ptr_1[l++] = col->data_ptr;
          data_ptr_1[l] = col->data_ptr_aux;
          break;
        }
        case DEPEND:break;
        default:break;
      }
    }

    for (uint32_t j = 0, l = 0; j < join_cols[tbl_idx_second].size(); j++, l++) {
      uint32_t idx = join_cols[tbl_idx_second][j];
      const auto &col = tbl_second->columns[idx];
      // std::cout << "df_second_data: ";
      // for (uint32_t s = 0; s < 4; s++) {
      //   std::cout << tbl_second->longCols[idx][s] << " ";
      // }
      switch (col->type) {
        case LONG: {
          data_ptr_2[l] = col->data_ptr;
          break;
        }
        case DOUBLE: {
          data_ptr_2[l] = col->data_ptr;
          break;
        }
        case INT: {
          data_ptr_2[l] = col->data_ptr;
          break;
        }
        case STRING: {
          data_ptr_2[l++] = col->data_ptr;
          data_ptr_2[l] = col->data_ptr_aux;
          break;
        }
        case DEPEND:break;
        default:break;
      }
    }
    struct two_key_comparator comparator(data_ptr_1, data_ptr_2, types, join_col_size);
    if (i == 0) {
      std::vector<int32_t> result_pair(pair_counter, 0x8fffffff);
      uint32_t iter_first = 0;
      uint32_t iter_second = 0;
      switch (join_type.first) {
        case INNER_JOIN: {
          while (iter_first < tbl_first->row_num) {
            while (comparator.compare(sorted_idx_1[iter_first], sorted_idx_2[iter_second]) < 0) {
              iter_second++;
              if (iter_second >= tbl_second->row_num) {
                goto finish_inner_join;
              }
            }
            uint32_t tmp_iter_second = iter_second;
            while (comparator.compare(sorted_idx_1[iter_first], sorted_idx_2[tmp_iter_second]) == 0) {
              result_pair[result_idx_first] = sorted_idx_1[iter_first];
              result_pair[result_idx_second] = sorted_idx_2[tmp_iter_second];
              result_pairs.push_back(result_pair);
              tmp_iter_second++;
              if (tmp_iter_second >= tbl_second->row_num) {
                break;
              }
            }
            iter_first++;
          }
          finish_inner_join:
          break;
        }
        case LEFT_OUTER_JOIN: {
          while (iter_first < tbl_first->row_num) {
            while (comparator.compare(sorted_idx_1[iter_first], sorted_idx_2[iter_second]) < 0) {
              iter_second++;
              if (iter_second >= tbl_second->row_num) {
                while (iter_first < tbl_first->row_num) {
                  result_pair[result_idx_first] = sorted_idx_1[iter_first];
                  result_pair[result_idx_second] = -1;
                  result_pairs.push_back(result_pair);
                  iter_first++;
                }
                goto finish_left_outer_join;
              }
            }
            if (comparator.compare(sorted_idx_1[iter_first], sorted_idx_2[iter_second]) > 0) {
              result_pair[result_idx_first] = sorted_idx_1[iter_first];
              result_pair[result_idx_second] = -1;
              result_pairs.push_back(result_pair);
            }

            uint32_t tmp_iter_second = iter_second;
            while (comparator.compare(sorted_idx_1[iter_first], sorted_idx_2[tmp_iter_second]) == 0) {
              result_pair[result_idx_first] = sorted_idx_1[iter_first];
              result_pair[result_idx_second] = sorted_idx_2[tmp_iter_second];
              result_pairs.push_back(result_pair);
              tmp_iter_second++;
              if (tmp_iter_second >= tbl_second->row_num) {
                break;
              }
            }
            iter_first++;
          }
          finish_left_outer_join:
          break;
        }
        case RIGHT_OUTER_JOIN: {
          break;
        }
        case FULL_OUTER_JOIN: {
          break;
        }
        case LEFT_SEMI_JOIN: {
          while (iter_first < tbl_first->row_num) {
            while (comparator.compare(sorted_idx_1[iter_first], sorted_idx_2[iter_second]) < 0) {
              iter_second++;
              if (iter_second >= tbl_second->row_num) {
                goto finish_left_semi_join;
              }
            }
            if (comparator.compare(sorted_idx_1[iter_first], sorted_idx_2[iter_second]) == 0) {
              result_pair[result_idx_first] = sorted_idx_1[iter_first];
              result_pairs.push_back(result_pair);
            }
            iter_first++;
          }
          finish_left_semi_join:
          break;
        }
        case CROSS_JOIN: {
          break;
        }
      }
    } else {

      switch (join_type.first) {
        case INNER_JOIN: {
          auto first_vec_iter = result_pairs.begin();
          uint32_t iter_second = 0;
          while (first_vec_iter != result_pairs.end()) {
            uint32_t first_idx = (*first_vec_iter)[result_idx_first];
            while (comparator.compare(first_idx, sorted_idx_2[iter_second]) < 0) {
              iter_second++;
              if (iter_second >= tbl_second->row_num) {
                goto finish_inner_join_2;
              }
            }
            if (comparator.compare(first_idx, sorted_idx_2[iter_second]) > 0) {
              result_pairs.erase(first_vec_iter++);
              continue;
            }
            uint32_t tmp_iter_second = iter_second;
            if (comparator.compare(first_idx, sorted_idx_2[tmp_iter_second]) == 0) {
              (*first_vec_iter)[result_idx_second] = sorted_idx_2[tmp_iter_second];
              tmp_iter_second++;
            }
            while (tmp_iter_second < tbl_second->row_num
                && comparator.compare(first_idx, sorted_idx_2[tmp_iter_second]) == 0) {
              result_pairs.insert(first_vec_iter, *first_vec_iter);
              (*first_vec_iter)[result_idx_second] = sorted_idx_2[tmp_iter_second];
              tmp_iter_second++;
            }
          }
          finish_inner_join_2:
          break;
        }
        case LEFT_OUTER_JOIN: {
          auto first_vec_iter = result_pairs.begin();
          uint32_t iter_second = 0;
          while (first_vec_iter != result_pairs.end()) {
            uint32_t first_idx = (*first_vec_iter)[result_idx_first];
            while (comparator.compare(first_idx, sorted_idx_2[iter_second]) < 0) {
              iter_second++;
              if (iter_second >= tbl_second->row_num) {
                while (first_vec_iter != result_pairs.end()) {
                  (*first_vec_iter)[result_idx_second] = -1;
                  first_vec_iter++;
                }
                goto finish_left_outer_join_2;
              }
            }
            if (comparator.compare(first_idx, sorted_idx_2[iter_second]) > 0) {
              (*first_vec_iter)[result_idx_second] = -1;
              first_vec_iter++;
              continue;
            }
            uint32_t tmp_iter_second = iter_second;
            if (comparator.compare(first_idx, sorted_idx_2[tmp_iter_second]) == 0) {
              (*first_vec_iter)[result_idx_second] = sorted_idx_2[tmp_iter_second];
              tmp_iter_second++;
            }
            while (tmp_iter_second < tbl_second->row_num
                && comparator.compare(first_idx, sorted_idx_2[tmp_iter_second]) == 0) {
              result_pairs.insert(first_vec_iter, *first_vec_iter);
//              first_vec_iter++;
              (*first_vec_iter)[result_idx_second] = sorted_idx_2[tmp_iter_second];
              tmp_iter_second++;
            }
            first_vec_iter++;
          }
          finish_left_outer_join_2:
          break;
        }
        case RIGHT_OUTER_JOIN: {

          break;
        }
        case FULL_OUTER_JOIN: {

          break;
        }
        case LEFT_SEMI_JOIN: {
          auto first_vec_iter = result_pairs.begin();
          uint32_t iter_second = 0;
          while (first_vec_iter != result_pairs.end()) {
            uint32_t first_idx = (*first_vec_iter)[result_idx_first];
            while (comparator.compare(first_idx, sorted_idx_2[iter_second]) < 0) {
              iter_second++;
              if (iter_second >= tbl_second->row_num) {
                goto finish_left_semi_join_2;
              }
            }
            if (comparator.compare(first_idx, sorted_idx_2[iter_second]) > 0) {
              result_pairs.erase(first_vec_iter++);
              continue;
            }
            if (comparator.compare(first_idx, sorted_idx_2[iter_second]) == 0) {
              first_vec_iter ++;
            }
          }
          finish_left_semi_join_2:
          break;
        }
        case CROSS_JOIN: {

          break;
        }
      }
    }
  }
  cardinality = result_pairs.size();
  std::cout << "GHive-CPP: [BaseJoinPredicate-sm_join]: cardinality: " << cardinality << std::endl;
  for (std::vector<int32_t> vec: result_pairs) {
    result_offsets.insert(result_offsets.end(), vec.begin(), vec.end());
  }
  profiler.end_merge();
  // 3. Generate the result dataflow according to the maintain_cols.
  profiler.start_data_recover();
  Table *result_tbl = generate_result(tables);
  profiler.end_data_recover();
  profiler.end_op();
  return result_tbl;
}

SortMergeJoinPredicate::SortMergeJoinPredicate() { }
