#include "hip/hip_runtime.h"
#include <iostream>
#include "Operator/JoinPredicate.hpp"
#include "Operator/Operator.hpp"
#include "DataFlow/DataFlow.hpp"
//#include "Operator/hashjoin.cuh"
#include "unordered_map"
//#include "Util/crystal.cuh"
//#include <hipcub/hipcub.hpp>
//#include <cub/util_allocator.cuh>
//#include <cub/block/block_shuffle.cuh>

/*
 * This method will call GPU has join.
 * */
//int *hashJoin(int *tableR, int num_tuples_R, int *tableS, int num_tuples_S,
//              int *result_count, Profiler &profiler) {
//
//  int *tagR = (int *) malloc(num_tuples_R * sizeof(int));
//  int *tagS = (int *) malloc(num_tuples_S * sizeof(int));
//
//  for (int i = 0; i < num_tuples_R; i++) {
//    tagR[i] = i;
//  }
//
//  for (int i = 0; i < num_tuples_S; i++) {
//    tagS[i] = i;
//  }
//
//  int *join_res;
//  if (num_tuples_R < 128000001 & num_tuples_S < 128000001) {
//    join_res =
//        inGPU_Hashjoin(tableR, tagR, num_tuples_R, tableS, tagS, num_tuples_S,
//                       log_parts1, log_parts2, 5 + p_d, result_count, profiler);
//    return join_res;
//  } else {
//    std::cout << "Out-of-GPU Hash join is not supported currently.";
//    return nullptr;
//  }
//}

JoinPredicate::JoinPredicate() {
  this->join_condition = UNKNOWN_CONDITION;
  this->join_type = UNKNOWN_TYPE;
  this->left_join_col = 0xffffffff;
  this->right_join_col = 0xffffffff;
}

JoinPredicate::JoinPredicate(JoinCondition join_condition, JoinType join_type,
                             uint32_t left_join_col, uint32_t right_join_col) :
    join_condition(join_condition), join_type(join_type),
    left_join_col(left_join_col), right_join_col(right_join_col) {}


/*
 * Receives the pointer of the data from two tables.
 * The rows with join keys the same will be merged together.
 * The right table will be after the left table.
 * */
//
// // GPU hash join impl with crystal
// // ---------------------------------------------------------------------------------------------------
//
// /**
//  * @brief build_hash_table hash_table of R
//  * @tparam BT BLOCK_THREADS
//  * @tparam IPT ITEMS_PER_THREAD
//  * @param keyR [input]
//  * @param idxR [input]
//  * @param rowNumR [input]
//  * @param hashTableR [output]
//  * @param slotNum [input] todo: optimized
//  *
//  * @note TILE <---> BLOCK
//  * num_slots为项数
//  * hash_table大小设置为2倍
//  */
// /// todo input selection flags
// template<uint32_t BT, uint32_t IPT>
// __global__ void
// buildHashTable(int64_t *keyR, int64_t *idxR, int rowNumR, int64_t *hashTableR, int slotNum) {
//
//     using ull_t = unsigned long long int; // uint64_t cast to unsigned long long int
//
//     ull_t keyItems[IPT];
//     ull_t idxItems[IPT];
//     int selectionFlags[IPT];
//
//     static const uint32_t TILE_SIZE = BT * IPT;
//     uint32_t tileOffset = blockIdx.x * TILE_SIZE;
//     uint32_t tileNum = (rowNumR + TILE_SIZE - 1) / TILE_SIZE;
//     uint32_t itemNumPerTile = TILE_SIZE;
//     if (blockIdx.x == tileNum - 1) {
//         itemNumPerTile = rowNumR - tileOffset; // the last block
//     }
//
//     InitFlags<BT, IPT>(selectionFlags);
//     BlockLoad<ull_t, BT, IPT>(reinterpret_cast<ull_t *>(keyR + tileOffset), keyItems, itemNumPerTile);
//     BlockLoad<ull_t, BT, IPT>(reinterpret_cast<ull_t *>(idxR + tileOffset), idxItems, itemNumPerTile);
//
//     // init hash table
//     int64_t *blockItr = hashTableR + tileOffset * 3;
//     int64_t *threadItr = blockItr + threadIdx.x * 3 * IPT;
// #pragma unroll
//     for (int i = 0; i < IPT; ++i) {
//         if (threadIdx.x * IPT + i < itemNumPerTile) {
//             threadItr[3 * i] = 0;
//             threadItr[3 * i + 1] = 0;
//             threadItr[3 * i + 2] = slotNum;
//         }
//     }
//
//     __syncthreads();
//
//     BlockBuildSelectivePHT_3<ull_t, ull_t, BT, IPT>(keyItems, idxItems, selectionFlags,
//                                                     reinterpret_cast<ull_t *>(hashTableR), slotNum,
//                                                     (int) itemNumPerTile);
// }
//
// /**
//  * @brief For each row in S, use R's hashTable to probe and count matched entries.
//  * @tparam BT BLOCK_THREADS
//  * @tparam IPT ITEMS_PER_THREAD
//
//  * @param keyS [input]
//  * @param idxS [input]
//  * @param rowNumS [input]
//  *
//  * @param hashTableR [input]
//  * @param slotNum [input]
//  *
//  * @param prefixSumS [output] blockWide
//  // * @param flagsS [output] todo: can be optimized.
//  * @param counter [input & output] length of final result
//  * @param writeOffset [output] every block write from this. length = number of block
//  */
// template<uint32_t BT, uint32_t IPT>
// __global__ void
// probeAndCount(int64_t *keyS, int64_t *idxS, int rowNumS,
//               int64_t *hashTableR, int slotNum,
//               int *prefixSumS, int *counter, int *writeOffset) {
//
//     using ull_t = unsigned long long int; // cast uint64_t to unsigned long long int
//     using BlockScanT = hipcub::BlockScan<int, BT, hipcub::BLOCK_SCAN_RAKING>;
//
//     __shared__ struct TempStorage {
//         ull_t out[BT * IPT]; // not enough
//         // int outOffset;
//         typename BlockScanT::TempStorage scan;
//     } tempStorage;
//
//     ull_t keyItemS[IPT];
//     ull_t idxItemS[IPT];
//     int selectionFlagsS[IPT];
//     int matchCountS[IPT];
//     int matchCountPrefixSumS[IPT];
//     int selectionNum;
//
//     static const uint32_t TILE_SIZE = BT * IPT;
//     uint32_t tileOffset = blockIdx.x * TILE_SIZE;
//     uint32_t tileNum = (rowNumS + TILE_SIZE - 1) / TILE_SIZE;
//     uint32_t itemNumPerTile = TILE_SIZE;
//     if (blockIdx.x == tileNum - 1) {
//         itemNumPerTile = rowNumS - tileOffset; // the last block
//     }
//
//     // printf("tileOffset = %d blockIdx = %d, itemNumPerTile = %d TILE_SIZE = %d tileNum = %d rowNumS = %d\n", tileOffset,
//     //        blockIdx.x,
//     //        itemNumPerTile, TILE_SIZE, tileNum, rowNumS);
//
//     BlockLoad<ull_t, BT, IPT>(reinterpret_cast<ull_t *>(keyS + tileOffset), keyItemS, itemNumPerTile);
//     BlockLoad<ull_t, BT, IPT>(reinterpret_cast<ull_t *>(idxS + tileOffset), idxItemS, itemNumPerTile);
//
//     InitCounts<BT, IPT>(matchCountS);
//     BlockProbeAndDo_3<ull_t, ull_t, BT, IPT>(
//             keyItemS, reinterpret_cast<ull_t *>(hashTableR), slotNum, itemNumPerTile,
//             [&]__device__(int i, ull_t k, ull_t v) { matchCountS[i]++; }
//     );
//
//     InitFlags<BT, IPT>(selectionFlagsS);
// #pragma unroll
//     for (int i = 0; i < IPT; ++i) {
//         selectionFlagsS[i] = !!matchCountS[i];
//     }
//     // calculate prefix sum per block =====================================================================
//
//     // int selectionFlagPrefixSum[IPT]; // block-wide prefix sum
//
//     BlockScanT(tempStorage.scan).ExclusiveSum(matchCountS, matchCountPrefixSumS, selectionNum);
//
//     // printf("**debug block scan: sum=%d \n", selectionNum);
//     // // for (int i = threadIdx.x * IPT; i < (threadIdx.x + 1) * IPT; ++i) {
//     // for (int i = 0; i < IPT; ++i) {
//     //     printf("blockID = %d selectionNum = %d matchCount = %d prefixSumS = %d key = %d\n",
//     //            blockIdx.x, selectionNum, matchCountS[i], matchCountPrefixSumS[i], keyItemS[i]);
//     // }
//     // printf("\n");
//
//     if (threadIdx.x == 0) {
//         writeOffset[blockIdx.x] = atomicAdd(counter, selectionNum);
//     }
//
//     BlockStore<int, BT, IPT>(prefixSumS + tileOffset, matchCountPrefixSumS, itemNumPerTile);
//     // BlockStore<int, BT, IPT>(flagsS + tileOffset, selectionFlagsS, itemNumPerTile);
//     //
//     // for (auto i = threadIdx.x; i < BT * IPT; i += blockDim.x) tempStorage.out[i] = 0;
//     // __syncthreads();
//     // BlockShuffle<ull_t, BT, IPT>(keyItemS, selectionFlagsS, selectionFlagPrefixSum, tempStorage.out);
//     // if (threadIdx.x == 0) {
//     //     memcpy(keyS + tempStorage.outOffset, tempStorage.out, sizeof(int64_t) * selectionNum);
//     // }
//     //
//     // for (auto i = threadIdx.x; i < BT * IPT; i += blockDim.x) tempStorage.out[i] = 0;
//     // __syncthreads();
//     // BlockShuffle<ull_t, BT, IPT>(idxItemS, selectionFlagsS, selectionFlagPrefixSum, tempStorage.out);
//     // if (threadIdx.x == 0) {
//     //     memcpy(idxS + tempStorage.outOffset, tempStorage.out, sizeof(int64_t) * selectionNum);
//     // }
//     //
//     // for (auto i = threadIdx.x; i < BT * IPT; i += blockDim.x) tempStorage.out[i] = 0;
//     // __syncthreads();
//     // BlockShuffle<ull_t, BT, IPT>(joinValueR, selectionFlagsS, selectionFlagPrefixSum, tempStorage.out);
//     // if (threadIdx.x == 0) {
//     //     memcpy(idxRJoinByS + tempStorage.outOffset, tempStorage.out, sizeof(int64_t) * selectionNum);
//     // }
//
//     // for (auto i = threadIdx.x; i < BT * IPT; i += blockDim.x) tempStorage.out[i] = 0;
//     // __syncthreads();
//     // BlockShuffle<int, BT, IPT>(selectionFlagsS, selectionFlagsS, selectionFlagPrefixSum, tempStorage.out);
//     // if (threadIdx.x == 0) {
//     //     memcpy(flagsS + tempStorage.outOffset, tempStorage.out, sizeof(int) * selectionNum);
//     // }
//     // --------------------------------------------------------------------------------------------------------
//
//     // for (int item = 0; item < IPT; ++item) {
//     //     if (threadIdx.x + BT * item < itemNumPerTile) {
//     //         // printf("item = %d, flag = %d: ", item, selectionFlagsS[item]);
//     //         if (selectionFlagsS[item]) {
//     //             printf("idxR: %4ld, key: %4ld, idxS: %4ld\n", joinValueR[item], keyItemS[item], idxItemS[item]);
//     //         }
//     //     }
//     // }
// }
//
// /**
//  * @brief
//  * @tparam BT BLOCK_THREADS
//  * @tparam IPT ITEMS_PER_THREAD
//  *
//  * @param keyS [input]
//  * @param idxS [input]
//  * @param rowNumS [input]
//  * @param prefixSumS [input]
//  * @param writeOffsetS [input] block wide
//  *
//  * @param hashTableR [input]
//  * @param slotNum [input]
//  *
//  * @param keyJoin [output]
//  * @param idxRJoin [output]
//  * @param idxSJoin [output]
//  */
// template<uint32_t BT, uint32_t IPT>
// __global__ void
// probeAndGenerate(int64_t *keyS, int64_t *idxS, int rowNumS, int *prefixSumS, int *writeOffsetS,
//                  int64_t *hashTableR, int slotNum,
//                  int64_t *keyJoin, int64_t *idxRJoin, int64_t *idxSJoin) {
//     using ull_t = unsigned long long int; // cast uint64_t to unsigned long long int
//
//     ull_t keyItemS[IPT];
//     ull_t idxItemS[IPT];
//     int prefixSumItemS[IPT]; // block wide
//     // int flagsItemS[IPT];
//
//     static const uint32_t TILE_SIZE = BT * IPT;
//     uint32_t tileOffset = blockIdx.x * TILE_SIZE;
//     uint32_t tileNum = (rowNumS + TILE_SIZE - 1) / TILE_SIZE;
//     uint32_t itemNumPerTile = TILE_SIZE;
//     if (blockIdx.x == tileNum - 1) {
//         itemNumPerTile = rowNumS - tileOffset; // the last block
//     }
//
//     BlockLoad<ull_t, BT, IPT>(reinterpret_cast<ull_t *>(keyS + tileOffset), keyItemS, itemNumPerTile);
//     BlockLoad<ull_t, BT, IPT>(reinterpret_cast<ull_t *>(idxS + tileOffset), idxItemS, itemNumPerTile);
//     BlockLoad<int, BT, IPT>(prefixSumS + tileOffset, prefixSumItemS, itemNumPerTile);
//     // BlockLoad<ull_t, BT, IPT>(reinterpret_cast<ull_t *>(flagsS + tileOffset), flagsItemS, itemNumPerTile);
//
//     int blockWriteOffset = writeOffsetS[blockIdx.x];
//
//     auto callback = [&]__device__(int i, ull_t k, ull_t v) {
//         int offset = blockWriteOffset + prefixSumItemS[i];
//
//         idxRJoin[offset] = (int64_t) v;
//         keyJoin[offset] = (int64_t) k; // keyItemS[i]
//         idxSJoin[offset] = (int64_t) idxItemS[i];
//         prefixSumItemS[i]++;
//
//         printf("Callback: hashtable[%d] = [%lld->%lld],\t\tgen row[%d=%d+%d]=[%lld %lld %lld]\n",
//                i, k, v,
//                offset, blockWriteOffset, prefixSumItemS[i] - 1,
//                idxRJoin[offset], keyJoin[offset], idxSJoin[offset]);
//     };
//
//     BlockProbeAndDo_3<ull_t, ull_t, BT, IPT>(
//             keyItemS, reinterpret_cast<ull_t *>(hashTableR), slotNum, itemNumPerTile,
//             callback
//     );
// }
//
//
// // --------------------------------------------------------------------------------------
// struct TableWithIndex {
//     int64_t *d_key;
//     std::vector<int64_t *> h_values;
//     uint32_t row_num;
//     hipcub::CachingDeviceAllocator &g_allocator;
//
//     TableWithIndex(int64_t *d_key_col, std::vector<int64_t *> &&h_val_cols, uint32_t row_num,
//                    hipcub::CachingDeviceAllocator &g_allocator)
//             : d_key(d_key_col),
//               h_values(h_val_cols),
//               row_num(row_num),
//               g_allocator(g_allocator) {}
//
//     TableWithIndex(DataFlow *df, uint32_t key_col, hipcub::CachingDeviceAllocator &g_allocator)
//             : g_allocator(g_allocator) {
//         // assert(join_type == IJ);
//         // assert(join_condition == EQ);
//
//         assert(key_col < df->longColNum + df->doubleColNum);
//
//         uint32_t key_index = df->sequence[key_col];
//         assert(key_index < df->longColNum);
//         //  Assert that 2 cols used for join are of the same data type -> long type
//
//         this->row_num = df->get_row_num();
//
//         int64_t *h_key = df->get_long_cols()[key_index];
//
//         this->d_key = nullptr;
//         CubDebugExit(this->g_allocator.DeviceAllocate((void **) &this->d_key, sizeof(int64_t) * row_num));
//         CubDebugExit(hipMemcpy(this->d_key, h_key, sizeof(int64_t) * row_num, hipMemcpyHostToDevice));
//
//         int64_t *h_idx;
//         h_idx = new int64_t[row_num];
//         init_index_sequence(h_idx, h_idx + row_num);
//
//         this->h_values.push_back(h_idx);
//         // printf("table constructed\n");
//         // print();
//     }
//
//     ~TableWithIndex() {
//         clear_memory();
//     }
//
//     void print() {
//         auto *h_key = new int64_t[row_num];
//         hipMemcpy(h_key, d_key, sizeof(int64_t) * row_num, hipMemcpyDeviceToHost);
//         auto col_num_join = h_values.size();
//         printf("With %lu cols, first col is key\n", col_num_join);
//
//         printf("+~~~~~~~~~+");
//         for (int c = 0; c < col_num_join; ++c) {
//             printf("---------+");
//         }
//         printf("\n");
//
//         for (int r = 0; r < row_num; ++r) {
//
//             printf("|");
//             printf(" %7ld |", h_key[r]);
//             for (int c = 0; c < col_num_join; ++c) {
//                 printf(" %7ld |", h_values[c][r]);
//             }
//             printf("\n");
//
//             printf("+~~~~~~~~~+");
//             for (int c = 0; c < col_num_join; ++c) {
//                 printf("---------+");
//             }
//             printf("\n");
//         }
//
//         delete[] h_key;
//     }
//
//     void clear_memory() {
//         // printf("Memory cleared:\n");
//         // print();
//
//         CubDebugExit(this->g_allocator.DeviceFree(d_key));
//         for (auto i: h_values) delete[] i;
//     }
//
// };
//
// TableWithIndex
// gpuHashJoinTwoTable(const TableWithIndex &R, const TableWithIndex &S, hipcub::CachingDeviceAllocator &gAllocator) {
//     int64_t *hIdxR, *hIdxS;
//     int64_t *dIdxR, *dIdxS;
//     hIdxR = new int64_t[R.row_num];
//     hIdxS = new int64_t[S.row_num];
//     init_index_sequence(hIdxR, hIdxR + R.row_num);
//     init_index_sequence(hIdxS, hIdxS + S.row_num);
//
//     CubDebugExit(gAllocator.DeviceAllocate((void **) &dIdxR, sizeof(int64_t) * R.row_num));
//     CubDebugExit(gAllocator.DeviceAllocate((void **) &dIdxS, sizeof(int64_t) * S.row_num));
//     CubDebugExit(hipMemcpy(dIdxR, hIdxR, sizeof(int64_t) * R.row_num, hipMemcpyHostToDevice));
//     CubDebugExit(hipMemcpy(dIdxS, hIdxS, sizeof(int64_t) * S.row_num, hipMemcpyHostToDevice));
//
//     int64_t *dHashTableR;       // 使用R建立的hash table
//     // int64_t *dIdxRJoinByS;      // 与Sjoin后的IdxR
//     // int *dSelectionFlagsR;   // R是build的时候输入用的。
//     int *dSelectionFlagsS;      // S是用S去probe之后输出用的。
//     int *dPrefixSum;            // 中间结果,每个threadBlock对应的prefixSum
//     int *dCounter;              // 中间用于同步的counter
//     int *dWriteOffset;          // 每个block写入最终结果的offset
//
//     CubDebugExit(gAllocator.DeviceAllocate((void **) &dHashTableR, sizeof(int64_t) * R.row_num * 3));
//     // CubDebugExit(g_allocator.DeviceAllocate((void **) &dIdxRJoinByS, sizeof(int64_t) * rowNumS));
//     // CubDebugExit(g_allocator.DeviceAllocate((void **) &dSelectionFlagsR, sizeof(int) * rowNumR));
//     CubDebugExit(gAllocator.DeviceAllocate((void **) &dSelectionFlagsS, sizeof(int) * S.row_num));
//     CubDebugExit(gAllocator.DeviceAllocate((void **) &dPrefixSum, sizeof(int) * S.row_num));
//     CubDebugExit(gAllocator.DeviceAllocate((void **) &dCounter, sizeof(int)));
//     CubDebugExit(gAllocator.DeviceAllocate((void **) &dWriteOffset, sizeof(int)));
//
//     hipMemset(dHashTableR, 0, sizeof(int64_t) * R.row_num * 3);
//     // hipMemset(dSelectionFlagsR, 1, sizeof(int) * rowNumR);
//     hipMemset(dSelectionFlagsS, 1, sizeof(int) * S.row_num);
//     // hipMemset(dIdxRJoinByS, 1, sizeof(int64_t) * rowNumR);
//     hipMemset(dCounter, 0, sizeof(int));
//
//     static const uint32_t BLOCK_THREADS = 6;
//     static const uint32_t ITEMS_PER_THREAD = 1;
//     static const uint32_t BLOCKS_PER_GRID =
//             (R.row_num + BLOCK_THREADS * ITEMS_PER_THREAD - 1) / (BLOCK_THREADS * ITEMS_PER_THREAD);
//
//     int slotNum = (int) R.row_num;
//
//     // printf("buildHashTable<%u, %u><<<%u, %u>>>()\n", BLOCK_THREADS, ITEMS_PER_THREAD, BLOCKS_PER_GRID, BLOCK_THREADS);
//
//     buildHashTable<BLOCK_THREADS, ITEMS_PER_THREAD><<<BLOCKS_PER_GRID, BLOCK_THREADS>>>(
//             R.d_key, dIdxR, R.row_num, dHashTableR,
//             slotNum
//     );
//
//     hipDeviceSynchronize();
//
//     // print hash table------------------------------------------------------------------------
//     auto *hHashTableR = new int64_t[R.row_num * 3];
//     hipMemcpy(hHashTableR, dHashTableR, sizeof(int64_t) * R.row_num * 3, hipMemcpyDeviceToHost);
//     printf("hash table R: ");
//     for (int i = 0; i < R.row_num; ++i) {
//         printf("[%ld -> %ld -> %ld] ", hHashTableR[i * 3], hHashTableR[i * 3 + 1], hHashTableR[i * 3 + 2]);
//     }
//     printf("\n");
//     delete[] hHashTableR;
//     //-----------------------------------------------------------------------------------------
//
//     probeAndCount<BLOCK_THREADS, ITEMS_PER_THREAD><<<BLOCKS_PER_GRID, BLOCK_THREADS>>>(
//             S.d_key, dIdxS, S.row_num, dHashTableR,
//             slotNum,
//             dPrefixSum, dCounter, dWriteOffset
//     );
//
//     hipDeviceSynchronize();
//
//     int hCounter;
//     hipMemcpy(&hCounter, dCounter, sizeof(int), hipMemcpyDeviceToHost);
//
//     int64_t *dKeyJoin;
//     int64_t *dIdxRJoin;
//     int64_t *dIdxSJoin;
//     CubDebugExit(gAllocator.DeviceAllocate((void **) &dKeyJoin, sizeof(int64_t) * hCounter));
//     CubDebugExit(gAllocator.DeviceAllocate((void **) &dIdxRJoin, sizeof(int64_t) * hCounter));
//     CubDebugExit(gAllocator.DeviceAllocate((void **) &dIdxSJoin, sizeof(int64_t) * hCounter));
//
//     probeAndGenerate<BLOCK_THREADS, ITEMS_PER_THREAD><<<BLOCKS_PER_GRID, BLOCK_THREADS>>>(
//             S.d_key, dIdxS, S.row_num, dPrefixSum, dWriteOffset,
//             dHashTableR, slotNum,
//             dKeyJoin, dIdxRJoin, dIdxSJoin
//     );
//
//     hipDeviceSynchronize();
//
//     auto *hIdxRJoin = new int64_t[hCounter];
//     auto *hIdxSJoin = new int64_t[hCounter];
//
//     hipMemcpy(hIdxRJoin, dIdxRJoin, sizeof(int64_t) * hCounter, hipMemcpyDeviceToHost);
//     hipMemcpy(hIdxSJoin, dIdxSJoin, sizeof(int64_t) * hCounter, hipMemcpyDeviceToHost);
//
//     // | R.val | S.val |
//     int rowNumJoin = hCounter;
//     std::vector<int64_t *> hValuesJoin;
//
//     for (auto col: R.h_values) {
//         auto *colJoin = new int64_t[rowNumJoin];
//         for (int i = 0; i < rowNumJoin; ++i) {
//             colJoin[i] = col[hIdxRJoin[i]];
//         }
//         hValuesJoin.push_back(colJoin);
//     }
//
//     for (auto col: S.h_values) {
//         auto *colJoin = new int64_t[rowNumJoin];
//         for (int i = 0; i < rowNumJoin; ++i) {
//             colJoin[i] = col[hIdxSJoin[i]];
//         }
//         hValuesJoin.push_back(colJoin);
//     }
//
//     delete[]hIdxRJoin;
//     delete[]hIdxSJoin;
//
//     // CubDebugExit(g_allocator.DeviceFree(dKeyJoin));
//     CubDebugExit(gAllocator.DeviceFree(dIdxRJoin));
//     CubDebugExit(gAllocator.DeviceFree(dIdxSJoin));
//
//     CubDebugExit(gAllocator.DeviceFree(dHashTableR));
//     // CubDebugExit(g_allocator.DeviceFree(dIdxRJoinByS));
//     // CubDebugExit(g_allocator.DeviceFree(dSelectionFlagsR));
//     CubDebugExit(gAllocator.DeviceFree(dSelectionFlagsS));
//     CubDebugExit(gAllocator.DeviceFree(dPrefixSum));
//     CubDebugExit(gAllocator.DeviceFree(dCounter));
//     CubDebugExit(gAllocator.DeviceFree(dWriteOffset));
//
//     CubDebugExit(gAllocator.DeviceFree(dIdxR));
//     CubDebugExit(gAllocator.DeviceFree(dIdxS));
//     delete[] hIdxR;
//     delete[] hIdxS;
//
//     return TableWithIndex{dKeyJoin, std::move(hValuesJoin), (uint32_t) rowNumJoin, gAllocator};
//
//
//     // delete[] hIdxRJoin;
//     // delete[] hIdxSJoin;
//
//     // // print final table: --------------------------------------------------------------
//     // auto *hKeyJoin = new int64_t[hCounter];
//     // hipMemcpy(hKeyJoin, dKeyJoin, sizeof(int64_t) * hCounter, hipMemcpyDeviceToHost);
//     // // printf("Table R: \n");
//     // // printf("+---------+---------+\n");
//     // // printf("| idxR    | keyR    |\n");
//     // // printf("+---------+---------+\n");
//     // // for (int i = 0; i < R.row_num; ++i) {
//     // //     printf("| %7ld | %7ld |\n", hIdxR[i], hKeyR[i]);
//     // //     printf("+---------+---------+\n");
//     // // }
//     // //
//     // // printf("Table S: \n");
//     // // printf("+---------+---------+\n");
//     // // printf("| idxS    | keyS    |\n");
//     // // printf("+---------+---------+\n");
//     // // for (int i = 0; i < rowNumS; ++i) {
//     // //     printf("| %7ld | %7ld |\n", hIdxS[i], hKeyS[i]);
//     // //     printf("+---------+---------+\n");
//     // // }
//     // printf("Final table: \n");
//     // printf("+---------+---------+---------+\n");
//     // printf("| idxR    | keyR/S  | idxS    |\n");
//     // printf("+---------+---------+---------+\n");
//     // for (int i = 0; i < hCounter; ++i) {
//     //     printf("| %7ld | %7ld | %7ld |\n", hIdxRJoin[i], hKeyJoin[i], hIdxSJoin[i]);
//     //     printf("+---------+---------+---------+\n");
//     // }
//     // delete[] hKeyJoin;
//     // // ---------------------------------------------------------------------------------------
//
//
//
//     // print result ---------------------------------------------------------------------------
//     // auto *hPrefixSum = new int[rowNumS];
//     // auto *hSelectionFlagsS = new int[rowNumS];
//     // auto *hIdxRJoinByS = new int64_t[rowNumS];
//     // hipMemcpy(hPrefixSum, dPrefixSum, sizeof(int) * rowNumS, hipMemcpyDeviceToHost);
//     // hipMemcpy(hSelectionFlagsS, dSelectionFlagsS, sizeof(int) * rowNumS, hipMemcpyDeviceToHost);
//     // // hipMemcpy(hIdxRJoinByS, dIdxRJoinByS, sizeof(int64_t) * rowNumS, hipMemcpyDeviceToHost);
//     //
//     // hipMemcpy(hKeyS, dKeyS, sizeof(int64_t) * rowNumS, hipMemcpyDeviceToHost);
//     // hipMemcpy(hIdxS, dIdxS, sizeof(int64_t) * rowNumS, hipMemcpyDeviceToHost);
//     // hipMemcpy(&rowNumS, dCounter, sizeof(int), hipMemcpyDeviceToHost);
//     //
//     // printf("Final Table:\n%10s %10s %10s %10s %10s\n", "flag", "w_idx", "R.idx", "(R/S).key", "S.idx");
//     // for (int i = 0; i < rowNumS; ++i) {
//     //     printf("%10d %10d %10ld %10ld %10ld\n", hSelectionFlagsS[i], hPrefixSum[i], hIdxRJoinByS[i], hKeyS[i],
//     //            hIdxS[i]);
//     // }
//     //
//     // delete[] hPrefixSum;
//     // delete[] hIdxRJoinByS;
//     // ----------------------------------------------------------------------------------------
//
// }
//
// DataFlow *JoinPredicate::gpu_hash_join(DataFlow *left_ptr, DataFlow *right_ptr, Profiler &profiler) {
//
//     /**
//      * @TODO template support
//      * @TODO multi key support
//      * @DONE multi table support
//      * @note
//      *  1. build index table for every dataflow
//      *  2. gpuHashJoinTwoTable() join all dataflow
//      *  3. link with other cols
//      * @DONE multi-val hash table support
//      * @note
//      *  Linear probeAndCount.
//      *  Use CAS for each slot.
//      *  Partitioned and shared hash table.
//      *  unsolved problems:
//      *  1. how to probeAndCount with partition:
//      *     1. Probe once and create thread-wide matchCountS[IPT];
//      *     2. Count block-wide prefixSum[BT * IPT] and selectionNum.
//      *     3. Use selectionNum to allocate global memory to store result
//      *     3. Probe again and write to global result;
//      *
//      *  shuffle or not ?
//      *  allocate how many space ?
//      *  DONE [list]
//      *      1. 简化输入输出接口 [done]
//      *      2. 实现近乎完整的两表join [done]
//      *      3. 思考多表join的思路 [done]
//      *      4. 讨论空间分配问题 [done]
//      * TODO distinct 0 and NULL
//      */
//     hipcub::CachingDeviceAllocator gAllocator(true);
//
//     TableWithIndex R(left_ptr, left_join_col, gAllocator), S(right_ptr, right_join_col, gAllocator);
//     TableWithIndex(gpuHashJoinTwoTable(R, S, gAllocator)).print();
//
//     // joinRS.print();
//     /*
//     // print RS: ------------------------------------------------------------------------------
//     // auto *hKeyJoin = new int64_t[joinRS.row_num];
//     // hipMemcpy(hKeyJoin, joinRS.d_key, sizeof(int64_t) * joinRS.row_num, hipMemcpyDeviceToHost);
//     // auto colNumJoin = joinRS.h_values.size();
//     // printf("Final table: with %lu cols\n", colNumJoin);
//     // printf("+---------+---------+---------+\n");
//     // printf("| idxR    | keyR/S  | idxS    |\n");
//     // printf("+---------+---------+---------+\n");
//     // for (int i = 0; i < joinRS.row_num; ++i) {
//     //     printf("| %7ld | %7ld | %7ld |\n",
//     //            joinRS.h_values[colNumJoin - 2][i], hKeyJoin[i], joinRS.h_values[colNumJoin - 1][i]);
//     //     printf("+---------+---------+---------+\n");
//     // }
//     // delete[] hKeyJoin;
//     // ----------------------------------------------------------------------------------------------
//
//     // free result memory
//     // joinRS.clearMemory(g_allocator);
//     // R.clearMemory(g_allocator);
//     // S.clear_memory(g_allocator);
//     // free device memory
//     */
//     // todo build dataflow
//     return nullptr;
// }
//
// /**
//  * @brief gpu_hash_join for N tables and different join types
//  * @param tables  length=Ns
//  * @param key_cols length=N
//  * @param types   length=N - 1;
//  * @return
//  */
// DataFlow *
// JoinPredicate::gpu_hash_join(const std::vector<DataFlow *> &tables, const std::vector<std::vector<uint32_t>> &key_cols,
//                              const std::vector<JoinType> &types) {
//     /**
//      * todo: add join type support
//      * todo: add multi key support
//      */
//     hipcub::CachingDeviceAllocator gAllocator(true);
//
//     auto tableNum = tables.size();
//     std::vector<TableWithIndex *> indexTables;
//
//     for (int i = 0; i < tableNum; ++i) {
//         // printf("table %d. key = %d\n", i, key_cols[i][0]);
//         indexTables.push_back(new TableWithIndex(tables[i], key_cols[i][0], gAllocator));
//     }
//
//     printf("First table: ");
//     indexTables[0]->print();
//     printf("Join table: ");
//     indexTables[1]->print();
//
//     auto *lastJoinResult = new TableWithIndex(gpuHashJoinTwoTable(*indexTables[0], *indexTables[1], gAllocator));
//     lastJoinResult->print();
//
//     for (int i = 2; i < tableNum; ++i) {
//         printf("Join table: ");
//         indexTables[i]->print();
//
//         auto *joinResult = new TableWithIndex(gpuHashJoinTwoTable(*lastJoinResult, *indexTables[i], gAllocator));
//         delete lastJoinResult;
//         lastJoinResult = joinResult;
//
//         lastJoinResult->print();
//     }
//
//
//     DataFlow *dfJoin = nullptr;
//     // todo: build dataflow
//
//     delete lastJoinResult;
//     for (auto ptr: indexTables) {
//         delete ptr;
//     }
//
//     return dfJoin;
// }
