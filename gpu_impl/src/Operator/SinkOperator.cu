#include "Operator/SinkOperator.hpp"

#include "Operator/GroupByPredicate.hpp"

SinkOperator::SinkOperator(std::string name, std::string vertex_name, vector<string> partitions) {
  operator_name = name;
  this->vertex_name = vertex_name;
  partition_cols = partitions;
  if (table_map.find(vertex_name) != table_map.end()) {
    this->op_result = table_map[vertex_name];
    is_input = true;
  } else {
    is_input = false;
  }
}

void SinkOperator::parseExtended() {
  std::vector<std::string> lines;
  split(extended_info, lines, "\n");
  for (std::string line: lines) { // todo: parse
    std::smatch match_result;
    if (std::regex_search(line, match_result,
                          std::regex("key expressions: (.*)"))) {
      std::string all_expressions = match_result[1];
      std::cout << "GHive-CPP [SinkOperator-parseExtended]: all_expression: " << match_result[1] << std::endl;
      std::vector<std::string> raw_expressions;
      split(all_expressions, raw_expressions, ", ");
      for (std::string each_expression: raw_expressions) {
        std::cout << "GHive-CPP [SinkOperator-parseExtended]: each_raw_expression: " << each_expression << std::endl;
        std::string each_expression_trim = trim(each_expression);
        std::cout << "GHive-CPP [SinkOperator-parseExtended]: each_trim_expression: " << each_expression_trim
                  << std::endl;
        if (std::regex_search(each_expression_trim, match_result, std::regex("(.*) \\(type: (.*)\\)"))) {
          std::string expression = match_result[1];
          this->key_expressions.push_back(expression);
          std::cout << "GHive-CPP [SinkOperator-parseExtended]: expression: " << expression << std::endl;
          //std::string expression_type = match_result[2];
          for (uint32_t i = 0; i < children[0]->output_cols.size(); i++) {
            std::cout << "GHive-CPP [SinkOperator-parseExtended]: children[0]->output_cols[" << i << "]: "
                      << children[0]->output_cols[i]
                      << std::endl;
            std::cout << vector_to_string(children[0]->output_cols) << std::endl;
            if (expression == children[0]->output_cols[i]) {
              key_offsets.push_back(i);
              break;
            }
          }
        }
      }

    } else if (std::regex_search(line, match_result,
                                 std::regex("null sort order: (.*)"))) { // todo: null sort order
      std::cout << "GHive-CPP [SinkOperator-parseExtended]: null sort order: " << match_result[1] << std::endl;

    } else if (std::regex_search(line, match_result,
                                 std::regex("sort order: (.*)"))) {
      this->sort_orders = match_result[1];
      std::cout << "GHive-CPP [SinkOperator-parseExtended]: sort order: " << match_result[1] << std::endl;
      // std::cout << "GHive-CPP [SinkOperator-parseExtended]: debug: ";
      // for (char c: this->sort_orders) {
      //   switch (SortOrder(c)) {
      //     case SortOrder::ASC: std::cout << "ASC" << std::endl;
      //       break;
      //     case SortOrder::DESC: std::cout << "DESC" << std::endl;
      //       break;
      //     default:break;
      //   }
      // }
      // std::cout << std::endl;
    }

  }
  std::vector<std::string> new_output_cols;

  // push key expressions first
  for (std::string key: key_expressions) {
    new_output_cols.push_back(key);
  }
  for (std::string col: output_cols) {
    bool find = false;
    for (std::string s: key_expressions) {
      if (s == col) {
        find = true;
      }
    }
    if (!find) {
      new_output_cols.push_back(col);
    }
  }
  for (std::string col: output_cols) {
    std::cout << "original pushback: " << col << std::endl;
    original_output_cols.push_back(col);
  }
  this->output_cols = new_output_cols;
  std::cout << "new output cols: " << vector_to_string(this->output_cols) << std::endl;
}

void SinkOperator::execute() {
  std::cout << "GHive-CPP [SinkOperator-execute]: inside operator_name: " << operator_name << std::endl;
  if (!is_input) {
    for (auto op: children) {
      op->execute();
    }
    Table *input_tbl = children[0]->op_result;
    std::cout << "GHive-CPP [SinkOperator-execute]: " << operator_name
              << " starts to execute with input: " << input_tbl->toString(100) << std::endl;

    Profiler profiler1;
    profiler1.start_op();
    profiler1.start_pci_host2device();
    std::vector<Column *> new_columns;
    for (uint32_t key_offset: key_offsets) {
      new_columns.push_back(input_tbl->columns[key_offset]);
    }
    for (uint32_t i = 0; i < input_tbl->columns.size(); i++) {
      bool is_key = false;
      for (uint32_t offset: key_offsets) {
        if (offset == i) {
          is_key = true;
          break;
        }
      }
      if (!is_key) {
        new_columns.push_back(input_tbl->columns[i]);
      }
    }


    auto millisecondsUTC =
        std::chrono::duration_cast<std::chrono::milliseconds>(
            std::chrono::system_clock::now().time_since_epoch()
        ).count();
    std::cout << "START_PCIE1: " << millisecondsUTC << std::endl;
    // sort new_columns -----------------------------------------------------------------------
    if (this->require_sort) {
      std::cout << "GHive-CPP [SinkOperator-execute]: Collect data and prepare sorting, order="
                << this->sort_orders
                << std::endl;

      auto key_num = this->sort_orders.size();
      auto row_num = input_tbl->rowNum();

      auto key_vec_size = key_num;
      for (uint32_t k_idx = 0; k_idx < key_num; ++k_idx) {
        if (new_columns[k_idx]->type == ColumnType::STRING) {
          key_vec_size++;
        }
      }
      thrust::device_vector<const void *> d_keys_vec;
      thrust::device_vector<int> d_keys_type;
      thrust::device_vector<SortOrder> d_sort_orders(sort_orders.size());

      for (uint32_t k_idx = 0; k_idx < key_num; ++k_idx) {
        d_sort_orders[k_idx] = SortOrder(sort_orders[k_idx]);
      }
      for (uint32_t k_idx = 0; k_idx < key_num; k_idx++) {
        auto *column = new_columns[k_idx];
        auto col_row_num = column->row_num;
        switch (column->type) {
          case LONG: {
            void *d_data_ptr = nullptr;
            hipMalloc((void **) &d_data_ptr, sizeof(long) * col_row_num);
            hipMemcpy(d_data_ptr, column->data_ptr, sizeof(long) * col_row_num, hipMemcpyHostToDevice);
            d_keys_vec.push_back(d_data_ptr);
            d_keys_type.push_back(LONG);
            break;
          }
          case DOUBLE: {
            void *d_data_ptr = nullptr;
            hipMalloc((void **) &d_data_ptr, sizeof(double) * col_row_num);
            hipMemcpy(d_data_ptr, column->data_ptr, sizeof(double) * col_row_num, hipMemcpyHostToDevice);
            d_keys_vec.push_back(d_data_ptr);
            d_keys_type.push_back(DOUBLE);
            break;
          }
          case INT: {
            void *d_data_ptr = nullptr;
            hipMalloc((void **) &d_data_ptr, sizeof(int32_t) * col_row_num);
            hipMemcpy(d_data_ptr, column->data_ptr, sizeof(int32_t) * col_row_num, hipMemcpyHostToDevice);
            d_keys_vec.push_back(d_data_ptr);
            d_keys_type.push_back(INT);
            break;
          }
          case STRING: {
            void *d_data_ptr = nullptr;
            void *d_data_ptr_aux = nullptr;
            int32_t size_char = 0;
            for (int x = 0; x < col_row_num; x++) {
              size_char = ((int32_t *) column->data_ptr_aux)[2 * x] > size_char ?
                          ((int32_t *) column->data_ptr_aux)[2 * x] : size_char;
              size_char = ((int32_t *) column->data_ptr_aux)[2 * x + 1] > size_char ?
                          ((int32_t *) column->data_ptr_aux)[2 * x + 1] : size_char;
            }
            hipMalloc((void **) &d_data_ptr, sizeof(char) * size_char);
            hipMalloc((void **) &d_data_ptr_aux, sizeof(int32_t) * col_row_num * 2);
            hipMemcpy(d_data_ptr, column->data_ptr, sizeof(char) * size_char, hipMemcpyHostToDevice);
            hipMemcpy(d_data_ptr_aux, column->data_ptr_aux, sizeof(int32_t) * col_row_num * 2, hipMemcpyHostToDevice);
            d_keys_vec.push_back(d_data_ptr);
            d_keys_vec.push_back(d_data_ptr_aux);
            d_keys_type.push_back(STRING);
            break;
          }
          default: break;
        }
      }
      profiler1.end_pci_host2device();

      auto millisecondsUTC2 =
          std::chrono::duration_cast<std::chrono::milliseconds>(
              std::chrono::system_clock::now().time_since_epoch()
          ).count();
      std::cout << "START_EXE: " << millisecondsUTC2 << std::endl;
      profiler1.start_gpu_exec();
      const void **d_keys_vec_ptr = thrust::raw_pointer_cast(d_keys_vec.data());
      const int *d_keys_type_ptr = thrust::raw_pointer_cast(d_keys_type.data());
      const SortOrder *d_sort_orders_ptr = thrust::raw_pointer_cast(d_sort_orders.data());

      thrust::device_vector<uint32_t> d_result_idx(row_num);
      thrust::sequence(d_result_idx.begin(), d_result_idx.end());
      std::cout << "GHive-CPP [SinkOperator-execute]: Keys number =" << key_num << std::endl;
      thrust::sort(d_result_idx.begin(), d_result_idx.end(),
                   sort_comparator(d_keys_vec_ptr, d_keys_type_ptr, static_cast<int>(key_num), d_sort_orders_ptr));
      std::cout << "GHive-CPP [SinkOperator-execute]: Finish gpu sorting" << std::endl;

      auto millisecondsUTC3 =
          std::chrono::duration_cast<std::chrono::milliseconds>(
              std::chrono::system_clock::now().time_since_epoch()
          ).count();
      std::cout << "START_PCIE2: " << millisecondsUTC3 << std::endl;

      profiler1.end_gpu_exec();
      thrust::host_vector<uint32_t> h_result_idx(d_result_idx.begin(), d_result_idx.end());


      auto millisecondsUTC4 = std::chrono::duration_cast<std::chrono::milliseconds>(
          std::chrono::system_clock::now().time_since_epoch()
      ).count();
      std::cout << "END_PCIE2: " << millisecondsUTC4 << std::endl;


      for (auto &col: new_columns) {
        switch (col->type) {
          case LONG: {
            auto *old_data_ptr = static_cast<long *>(col->data_ptr);
            auto *new_data_ptr = new long[row_num];
            for (uint32_t r = 0; r < row_num; ++r) {
              new_data_ptr[r] = old_data_ptr[h_result_idx[r]];
            }
            col->data_ptr = new_data_ptr;
            // delete[]old_data_ptr;
            break;
          }
          case DOUBLE: {
            auto *old_data_ptr = static_cast<double *>(col->data_ptr);
            auto *new_data_ptr = new double[row_num];
            for (uint32_t r = 0; r < row_num; ++r) {
              new_data_ptr[r] = old_data_ptr[h_result_idx[r]];
            }
            col->data_ptr = new_data_ptr;
            // delete[]old_data_ptr;
            break;
          }
          case INT: {
            auto *old_data_ptr = static_cast<int32_t *>(col->data_ptr);
            auto *new_data_ptr = new int32_t[row_num];
            for (uint32_t r = 0; r < row_num; ++r) {
              new_data_ptr[r] = old_data_ptr[h_result_idx[r]];
            }
            col->data_ptr = new_data_ptr;
            // delete[]old_data_ptr;
            break;
          }
          case STRING: {
            auto *old_data_ptr_aux = static_cast<int32_t *>(col->data_ptr_aux);
            auto *new_data_ptr_aux = new int32_t[row_num * 2];
            for (uint32_t r = 0; r < row_num; ++r) {
              new_data_ptr_aux[r * 2] = old_data_ptr_aux[h_result_idx[r] * 2];
              new_data_ptr_aux[r * 2 + 1] = old_data_ptr_aux[h_result_idx[r] * 2 + 1];
            }
            col->data_ptr_aux = new_data_ptr_aux;
            // delete[]old_data_ptr;
            break;
          }
          default:break;
        }
      }
      std::cout << "GHive-CPP [SinkOperator-execute]: Finish cpu attaching" << std::endl;
    }

     auto millisecondsUTC5 = std::chrono::duration_cast<std::chrono::milliseconds>(
         std::chrono::system_clock::now().time_since_epoch()
     ).count();
     std::cout << "END_DATA_RECOVER: " << millisecondsUTC5 << std::endl;

    // ---------------------------------------------------------------------------------------------
    profiler1.end_op();
    std::cout << profiler1.toString() << std::endl;

    input_tbl->columns = new_columns;
    op_result = input_tbl;
  }
  std::cout << "GHive-CPP [SinkOperator-execute]: " << operator_name
            << " ends to execute, with result: " << op_result->toString(100) << std::endl;
  std::cout << "GHive-CPP [SinkOperator-execute]: " << operator_name
            << " ends to execute, with result.... " << std::endl;
}

std::string SinkOperator::toString() {
  std::string partition_string = "partition cols: ";
  if (!partition_cols.empty()) {
    partition_string += partition_cols[0];
    for (uint32_t i = 1; i < partition_cols.size(); i++) {
      partition_string += ",";
      partition_string += partition_cols[i];
    }
  }
  return "[" + operator_name + "]; " + "vertex name: " + vertex_name + "; " +
      partition_string + ";";
}

std::string SinkOperator::toString(int level) {
  std::string ret = "";
  for (int i = 0; i < level; i++) {
    ret += "  ";
  }
  ret += this->toString() + "\n";
  if (!is_input) {
    for (auto op: children) {
      ret += op->toString(level + 1);
    }
  }
  return ret;
}
