#include "hip/hip_runtime.h"
#include <Operator/PTFOperator.hpp>
#include <Operator/GroupByPredicate.hpp>

void PTFOperator::parseExtended() {
  std::cout << "parse: " << operator_name << std::endl;
  std::vector<std::string> extended_plan_lines;
  split(this->extended_info, extended_plan_lines, "\n");

  std::regex col_order_by("order by: (*)");
  std::regex col_partition_by("partition by: (*)");
  std::regex col_expression0("(.*) ASC NULL (.*)");
  std::regex col_expression1("(.*) DESC NULL (.*)");
  std::regex ASC("ASC");
  std::regex functions_begin("window functions:");
  std::regex window_function_definition("window function definition");
  std::regex function_alias("alias: (.*)");
  std::regex function_argument("arguments: (.*)");
  std::regex function_type("name: (.*)");
  std::regex output_col_name("(_col[0-9]+)");
  std::regex output_shape("output shape: (.*)");

  std::smatch result;
  auto *predicates = new PTFPredicate();
  std::vector<uint32_t> order_by_col;
  std::vector<SortOrder> asc;
  std::vector<bool> null_first;
  std::vector<uint32_t> partition_by_col;
  std::vector<WindowFunction> window_functions;

  int line_index = 0;
  for (; line_index < extended_plan_lines.size(); line_index++) {
    std::string line = extended_plan_lines[line_index];
    std::cout << line << std::endl;
    if (std::regex_search(line,result,output_shape)){
      std::vector<std::string> token;
      split_unquote(result[1], token , ",");
      for (auto & i : token) {
        regex_search(i,result,output_shape);
        this->output_cols.push_back(result[1]);
      }
    }

    if (regex_search(line,result,col_order_by)) {
      std::vector<std::string> token;
      split_unquote(result[1], token, ",");
      for (auto &i: token) {
        if (std::regex_search(i, result, col_expression0)) {
          std::string col_name = result[1];
          for (uint32_t j = 0; j < children[0]->output_cols.size(); j++) {
            if (col_name == children[0]->output_cols[j]) {
              order_by_col.push_back(j);
              break;
            }
          }
          if (result[2] == "FIRST")
            null_first.push_back(true);
          else null_first.push_back(false);
          asc.push_back(SortOrder::ASC);
        } else if (std::regex_search(i, result, col_expression1)) {
          std::string col_name = result[1];
          for (uint32_t j = 0; j < children[0]->output_cols.size(); j++) {
            if (col_name == children[0]->output_cols[j]) {
              order_by_col.push_back(j);
              break;
            }
          }
          if (result[2] == "FIRST")
            null_first.push_back(true);
          else null_first.push_back(false);
          asc.push_back(SortOrder::DESC);
        }
      }
      predicates->order_by_col=order_by_col;
      predicates->null_first=null_first;
      predicates->asc=asc;
    }

    if (regex_search(line,result,col_partition_by)){
      std::vector<std::string> token;
      split_unquote(result[1], token, ",");
      for (const auto &x: token) {
        for (uint32_t j = 0; j < children[0]->output_cols.size(); j++) {
          if (x == children[0]->output_cols[j]) {
            order_by_col.push_back(j);
            break;
          }
        }
      }
      predicates->partition_by_col=partition_by_col;
    }

    else if (!regex_search(line, result, functions_begin)) continue;
    else {
      line = extended_plan_lines[++line_index];
      std::cout << line << std::endl;
      if (regex_search(line, result, window_function_definition)) {
        WindowFunction windowFunction{};
        line = extended_plan_lines[++line_index];
        regex_search(line, result, function_alias);
        windowFunction.alias = result[1];
        line = extended_plan_lines[line_index++];
        regex_search(line, result, function_argument);
        std::vector<std::string> token;
        split_unquote(result[1], token, ",");
        //only need one argument.
        for (uint32_t j = 0; j < children[0]->output_cols.size(); j++) {
          if (token[0] == children[0]->output_cols[j]) {
            windowFunction.arguments=j;
            break;
          }
        }
        line = extended_plan_lines[++line_index];
        regex_search(line, result, function_type);

        if (result[1]=="sum")
          windowFunction.type =SUM;
        else if (result[1]=="max")
          windowFunction.type =MAX;
        else if (result[1]=="min")
          windowFunction.type=MIN;
        else if (result[1]=="count")
          windowFunction.type=CNT;
        else if (result[1]=="rank")
          windowFunction.type=RANK;

        predicates->window_functions.push_back(windowFunction);
      }
    }
  }

  this->predicate = predicates;

  for (const auto& x: this->predicate->window_functions) {
    this->output_cols.push_back(x.alias);
  }
}





