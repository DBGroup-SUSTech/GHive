#include "hip/hip_runtime.h"
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/reverse_iterator.h>
#include <thrust/partition.h>
#include <thrust/device_vector.h>
#include <algorithm>
#include <Profile/FilterProfiler.hpp>
#include "Operator/FilterOperator.hpp"
#include "Operator/FilterPredicate.hpp"

FilterPredicate::FilterPredicate() {}

void FilterPredicate::gpu_string_filter(char *filter_col, int *index_col, std::string *val, Profiler &profiler) {
  std::cout << "GHive-CPP [FilterPredicate-process]: processing string starts." << std::endl;
  int size_char = 0;
  for (int x = 0; x < row_num; x++) {
    size_char = ((int32_t *) index_col)[2 * x] > size_char ?
                ((int32_t *) index_col)[2 * x] : size_char;
    size_char = ((int32_t *) index_col)[2 * x + 1] > size_char ?
                ((int32_t *) index_col)[2 * x + 1] : size_char;
  }
  thrust::device_vector<char> d_filter_col(filter_col, filter_col + size_char);
  thrust::device_vector<int> d_index_col(index_col, index_col + 2 * row_num);
  thrust::device_vector<char> d_val_0_vec(val[0].begin(), val[0].end());

  thrust::counting_iterator<int> iter(0);

  int size_0 = val[0].size();
  char *d_data = thrust::raw_pointer_cast(d_filter_col.data());
  int *d_index = thrust::raw_pointer_cast(d_index_col.data());
  char *d_val_0 = thrust::raw_pointer_cast(d_val_0_vec.data());

  switch (mode) {
    case FILTER_EQ: {
      thrust::transform(iter, iter + row_num, result_bitmap.begin(),
                        filter_str_unary_eq(d_data, d_index, d_val_0, size_0));
      break;
    }
    case FILTER_LE: {
      thrust::transform(iter, iter + row_num, result_bitmap.begin(),
                        filter_str_unary_le(d_data, d_index, d_val_0, size_0));
      break;
    }
    case FILTER_GE: {
      thrust::transform(iter, iter + row_num, result_bitmap.begin(),
                        filter_str_unary_ge(d_data, d_index, d_val_0, size_0));
      break;
    }
    case FILTER_LT: {
      thrust::transform(iter, iter + row_num, result_bitmap.begin(),
                        filter_str_unary_lt(d_data, d_index, d_val_0, size_0));

      break;
    }
    case FILTER_GT: {
      thrust::transform(iter, iter + row_num, result_bitmap.begin(),
                        filter_str_unary_gt(d_data, d_index, d_val_0, size_0));
      break;
    }
    case FILTER_EQ_RANGE: {
      thrust::device_vector<char> d_val_1_vec(val[1].begin(), val[1].end());
      char *d_val_1 = thrust::raw_pointer_cast(d_val_1_vec.data());
      int size_1 = val[1].size();
      thrust::transform(iter, iter + row_num, result_bitmap.begin(),
                        filter_str_unary_eq_range(d_data, d_index, d_val_0, size_0, d_val_1, size_1));
      break;
    }
    case FILTER_NOT_EQ: {
      thrust::transform(iter, iter + row_num, result_bitmap.begin(),
                        filter_str_unary_ne(d_data, d_index, d_val_0, size_0));
      break;
    }
    case FILTER_NOT_RANGE: {
      thrust::device_vector<char> d_val_1_vec(val[1].begin(), val[1].end());
      char *d_val_1 = thrust::raw_pointer_cast(d_val_1_vec.data());
      int size_1 = val[1].size();
      thrust::transform(iter, iter + row_num, result_bitmap.begin(),
                        filter_str_unary_not_range(d_data, d_index, d_val_0, size_0, d_val_1, size_1));
      break;
    }
    case FILTER_RANGE:
    case FILTER_NOT_EQ_RANGE:
    case FILTER_NOT_NULL:
    case FILTER_AND:
    case FILTER_OR:break;
  }
}

void FilterPredicate::gpu_string_filter_not_null(int *index_col, Profiler &profiler) {
  std::cout << "GHive-CPP [FilterPredicate-process]: processing string starts." << std::endl;
  profiler.start_pci_host2device();
  thrust::device_vector<int> d_index_col(index_col, index_col + 2 * row_num);
  profiler.end_pci_host2device();
  int *d_index = thrust::raw_pointer_cast(d_index_col.data());
  thrust::counting_iterator<int> iter(0);
  thrust::transform(iter, iter + row_num, result_bitmap.begin(),
                    filter_str_not_null(d_index));

}

void FilterPredicate::cpu_string_filter_not_null(int *index_col) {
  thrust::counting_iterator<int> iter(0);
  thrust::transform(iter, iter + row_num, host_result_bitmap.begin(),
                    filter_str_not_null(index_col));

}

void FilterPredicate::cpu_string_filter(char *filter_col, int *index_col, std::string *val) {
  std::cout << "GHive-CPP [FilterPredicate-process]: processing string starts." << std::endl;

  thrust::counting_iterator<int> iter(0);
  char *v_0 = const_cast<char *>(val[0].data());
  uint32_t v_size = val[0].size();

  switch (mode) {
    case FILTER_EQ: {
      thrust::transform(iter, iter + row_num, host_result_bitmap.begin(),
                        filter_str_unary_eq(filter_col, index_col, v_0, v_size));
      break;
    }
    case FILTER_LE: {
      thrust::transform(iter, iter + row_num, host_result_bitmap.begin(),
                        filter_str_unary_le(filter_col, index_col, v_0, v_size));
      break;
    }
    case FILTER_GE: {
      thrust::transform(iter, iter + row_num, host_result_bitmap.begin(),
                        filter_str_unary_ge(filter_col, index_col, v_0, v_size));
      break;
    }
    case FILTER_LT: {
      thrust::transform(iter, iter + row_num, host_result_bitmap.begin(),
                        filter_str_unary_lt(filter_col, index_col, v_0, v_size));

      break;
    }
    case FILTER_GT: {
      thrust::transform(iter, iter + row_num, host_result_bitmap.begin(),
                        filter_str_unary_gt(filter_col, index_col, v_0, v_size));
      break;
    }
    case FILTER_EQ_RANGE: {
      char *v_1 = const_cast<char *>(val[1].data());
      uint32_t v_1_size = val[1].size();

      thrust::transform(iter, iter + row_num, host_result_bitmap.begin(),
                        filter_str_unary_eq_range(filter_col, index_col, v_0, v_size, v_1, v_1_size));
      break;
    }
    case FILTER_NOT_EQ: {
      thrust::transform(iter, iter + row_num, host_result_bitmap.begin(),
                        filter_str_unary_ne(filter_col, index_col, v_0, v_size));
      break;
    }
    case FILTER_NOT_RANGE: {
      char *v_1 = const_cast<char *>(val[1].data());
      uint32_t v_1_size = val[1].size();
      thrust::transform(iter, iter + row_num, host_result_bitmap.begin(),
                        filter_str_unary_not_range(filter_col, index_col, v_0, v_size, v_1, v_1_size));
      break;
    }
    case FILTER_RANGE:
    case FILTER_NOT_EQ_RANGE:
    case FILTER_NOT_NULL:
    case FILTER_AND:
    case FILTER_OR:break;
  }
}

template<typename T>
void FilterPredicate::gpu_filter(T *filter_col, T *val, FilterProfiler &profiler) {
  profiler.start_pci_host2device();
  std::cout << "row number: " << row_num << std::endl;
  thrust::device_vector<T> d_filter_col(filter_col, filter_col + row_num);
  profiler.end_pci_host2device();
  profiler.start_gpu_exec();
  profiler.start_transform();
  switch (mode) {
    case FILTER_EQ: // val[0] is never null
      thrust::transform(d_filter_col.begin(), d_filter_col.end(), result_bitmap.begin(), filter_unary_eq<T>(val[0]));
      break;
    case FILTER_LE:
      thrust::transform(d_filter_col.begin(),
                        d_filter_col.end(),
                        result_bitmap.begin(),
                        filter_unary_le<T>(val[0]));
      break;
    case FILTER_GE: // val[0] is never null
      thrust::transform(d_filter_col.begin(), d_filter_col.end(), result_bitmap.begin(), filter_unary_ge<T>(val[0]));
      break;
    case FILTER_LT:
      thrust::transform(d_filter_col.begin(),
                        d_filter_col.end(),
                        result_bitmap.begin(),
                        filter_unary_lt<T>(val[0]));
      break;
    case FILTER_GT:
      thrust::transform(d_filter_col.begin(),
                        d_filter_col.end(),
                        result_bitmap.begin(),
                        filter_unary_gt<T>(val[0]));
      break;
    case FILTER_NOT_EQ:
      thrust::transform(d_filter_col.begin(),
                        d_filter_col.end(),
                        result_bitmap.begin(),
                        filter_unary_ne<T>(val[0]));
      break;
    case FILTER_NOT_NULL:
      thrust::transform(d_filter_col.begin(),
                        d_filter_col.end(),
                        result_bitmap.begin(),
                        filter_not_null<T>());
      break;
    case FILTER_AND:
    case FILTER_OR:break;
    case FILTER_EQ_RANGE:
      thrust::transform(d_filter_col.begin(),
                        d_filter_col.end(),
                        result_bitmap.begin(),
                        filter_eq_range<T>(val[0], val[1]));
      break;
    case FILTER_NOT_RANGE:
      thrust::transform(d_filter_col.begin(),
                        d_filter_col.end(),
                        result_bitmap.begin(),
                        filter_not_range<T>(val[0], val[1]));
      break;
    case FILTER_NOT_EQ_RANGE:
      thrust::transform(d_filter_col.begin(),
                        d_filter_col.end(),
                        result_bitmap.begin(),
                        filter_not_eq_range<T>(val[0], val[1]));
      break;
    case FILTER_RANGE:
      thrust::transform(d_filter_col.begin(),
                        d_filter_col.end(),
                        result_bitmap.begin(),
                        filter_range<T>(val[0], val[1]));
      break;
  }
  profiler.end_gpu_exec();
  profiler.end_transform();
}

template<typename T>
void FilterPredicate::cpu_filter(T *filter_col, T *val) {
  host_result_bitmap = thrust::host_vector<T>(row_num);
  switch (mode) {
    case FILTER_EQ:
      thrust::transform(filter_col,
                        filter_col + row_num,
                        host_result_bitmap.begin(),
                        filter_unary_eq<T>(val[0]));
      break;
    case FILTER_LE:
      thrust::transform(filter_col,
                        filter_col + row_num,
                        host_result_bitmap.begin(),
                        filter_unary_le<T>(val[0]));
      break;
    case FILTER_GE:
      thrust::transform(filter_col,
                        filter_col + row_num,
                        host_result_bitmap.begin(),
                        filter_unary_ge<T>(val[0]));
      break;
    case FILTER_LT:
      thrust::transform(filter_col,
                        filter_col + row_num,
                        host_result_bitmap.begin(),
                        filter_unary_lt<T>(val[0]));
      break;
    case FILTER_GT:
      thrust::transform(filter_col,
                        filter_col + row_num,
                        host_result_bitmap.begin(),
                        filter_unary_gt<T>(val[0]));
      break;
    case FILTER_NOT_EQ:
      thrust::transform(filter_col,
                        filter_col + row_num,
                        host_result_bitmap.begin(),
                        filter_unary_ne<T>(val[0]));
      break;
    case FILTER_NOT_NULL:
      thrust::transform(filter_col,
                        filter_col + row_num,
                        host_result_bitmap.begin(),
                        filter_not_null<T>());
      break;
    case FILTER_AND:
    case FILTER_OR:break;
    case FILTER_EQ_RANGE:
      thrust::transform(filter_col,
                        filter_col + row_num,
                        host_result_bitmap.begin(),
                        filter_eq_range<T>(val[0], val[1]));
      break;
    case FILTER_NOT_RANGE:
      thrust::transform(filter_col,
                        filter_col + row_num,
                        host_result_bitmap.begin(),
                        filter_not_range<T>(val[0], val[1]));
      break;
    case FILTER_NOT_EQ_RANGE:
      thrust::transform(filter_col,
                        filter_col + row_num,
                        host_result_bitmap.begin(),
                        filter_not_eq_range<T>(val[0], val[1]));
      break;
    case FILTER_RANGE:
      thrust::transform(filter_col,
                        filter_col + row_num,
                        host_result_bitmap.begin(),
                        filter_range<T>(val[0], val[1]));
      break;
  }
}

template<typename T, typename F>
void FilterPredicate::cpu_filter_two(T *filter_col_1, F *filter_col_2) {
  host_result_bitmap = thrust::host_vector<T>(row_num);
  switch (mode) {
    case FILTER_EQ:
      thrust::transform(filter_col_1,
                        filter_col_1,
                        filter_col_2,
                        host_result_bitmap.begin(),
                        filter_binary_eq<T, F>());
      break;
    case FILTER_LE:
      thrust::transform(filter_col_1,
                        filter_col_1 + row_num,
                        filter_col_2,
                        host_result_bitmap.begin(),
                        filter_binary_le<T, F>());
      break;
    case FILTER_GE:
      thrust::transform(filter_col_1,
                        filter_col_1 + row_num,
                        filter_col_2,
                        host_result_bitmap.begin(),
                        filter_binary_ge<T, F>());
      break;
    case FILTER_LT:
      thrust::transform(filter_col_1,
                        filter_col_1 + row_num,
                        filter_col_2,
                        host_result_bitmap.begin(),
                        filter_binary_lt<T, F>());
      break;
    case FILTER_GT:
      thrust::transform(filter_col_1,
                        filter_col_1 + row_num,
                        filter_col_2,
                        host_result_bitmap.begin(),
                        filter_binary_gt<T, F>());
      break;
    case FILTER_NOT_EQ:
      thrust::transform(filter_col_1,
                        filter_col_1 + row_num,
                        filter_col_2,
                        host_result_bitmap.begin(),
                        filter_binary_ne<T, F>());
      break;
    case FILTER_NOT_NULL:
    case FILTER_AND:
    case FILTER_OR:
    case FILTER_EQ_RANGE:
    case FILTER_NOT_RANGE:
    case FILTER_NOT_EQ_RANGE:
    case FILTER_RANGE:break;
  }
//  result_bitmap = host_result_bitmap;
}

template<typename T, typename F>
void FilterPredicate::gpu_filter_two(T *filter_col_1, F *filter_col_2, Profiler &profiler) {
  thrust::device_vector<T> d_filter_col_1(filter_col_1, filter_col_1 + row_num);
  thrust::device_vector<T> d_filter_col_2(filter_col_2, filter_col_2 + row_num);
  switch (mode) {
    case FILTER_EQ:
      thrust::transform(d_filter_col_1.begin(),
                        d_filter_col_1.end(),
                        d_filter_col_2.begin(),
                        result_bitmap.begin(),
                        filter_binary_eq<T, F>());
      break;
    case FILTER_LE:
      thrust::transform(d_filter_col_1.begin(),
                        d_filter_col_1.end(),
                        d_filter_col_2.begin(),
                        result_bitmap.begin(),
                        filter_binary_le<T, F>());
      break;
    case FILTER_GE:
      thrust::transform(d_filter_col_1.begin(),
                        d_filter_col_1.end(),
                        d_filter_col_2.begin(),
                        result_bitmap.begin(),
                        filter_binary_ge<T, F>());
      break;
    case FILTER_LT:
      thrust::transform(d_filter_col_1.begin(),
                        d_filter_col_1.end(),
                        d_filter_col_2.begin(),
                        result_bitmap.begin(),
                        filter_binary_lt<T, F>());
      break;
    case FILTER_GT:
      thrust::transform(d_filter_col_1.begin(),
                        d_filter_col_1.end(),
                        d_filter_col_2.begin(),
                        result_bitmap.begin(),
                        filter_binary_gt<T, F>());
      break;
    case FILTER_NOT_EQ:
      thrust::transform(d_filter_col_1.begin(),
                        d_filter_col_1.end(),
                        d_filter_col_2.begin(),
                        result_bitmap.begin(),
                        filter_binary_ne<T, F>());
      break;
    case FILTER_NOT_NULL:
    case FILTER_AND:
    case FILTER_OR:
    case FILTER_EQ_RANGE:
    case FILTER_NOT_RANGE:
    case FILTER_NOT_EQ_RANGE:
    case FILTER_RANGE:break;
  }
}

void FilterPredicate::process(Table *table, FilterProfiler &profiler) {
  std::cout << "GHive-CPP [FilterPredicate-process]: processing starts." << std::endl;
  std::cout << this->toString(0) << std::endl;
  row_num = table->row_num;
  profiler.start_gpu_alloc();
  result_bitmap = thrust::device_vector<int>(row_num);
  profiler.end_gpu_alloc();
//  int *bitmap;
//  profiler.start_pci_host2device();
//  hipMalloc((void **)&bitmap, row_num * sizeof(int));
//  profiler.end_pci_host2device();

  if (predColNum == 0) { // and / or
    if (mode == FILTER_AND) {
      childrenPredicate[0]->process(table, profiler);
      childrenPredicate[1]->process(table, profiler);
      std::cout << "childrenPredicate[0] with result_bitmap.size() = "
                << childrenPredicate[0]->result_bitmap.size() << ":" << std::endl;
      auto len0 = std::min<::size_t>(100, childrenPredicate[0]->result_bitmap.size());
      // auto len0 = childrenPredicate[0]->result_bitmap.size();
      for (int i = 0; i < len0; i++) {
        std::cout << childrenPredicate[0]->result_bitmap[i] << " ";
      }
      std::cout << std::endl;

      std::cout << "childrenPredicate[1] with result_bitmap.size() = "
                << childrenPredicate[1]->result_bitmap.size() << ":" << std::endl;
      auto len1 = std::min<::size_t>(100, childrenPredicate[1]->result_bitmap.size());
      // auto len1 = childrenPredicate[1]->result_bitmap.size();
      std::cout << "childrenPredicate[1]: " << std::endl;
      for (int i = 0; i < len1; i++) {
        std::cout << childrenPredicate[1]->result_bitmap[i] << " ";
      }
      std::cout << std::endl;

      thrust::transform(childrenPredicate[0]->result_bitmap.begin(), childrenPredicate[0]->result_bitmap.end(),
                        childrenPredicate[1]->result_bitmap.begin(), result_bitmap.begin(),
                        filter_binary_and<int, int>());
    } else if (mode == FILTER_OR) {
      childrenPredicate[0]->process(table, profiler);
      childrenPredicate[1]->process(table, profiler);
      thrust::transform(childrenPredicate[0]->result_bitmap.begin(), childrenPredicate[0]->result_bitmap.end(),
                        childrenPredicate[1]->result_bitmap.begin(), result_bitmap.begin(),
                        filter_binary_or<int, int>());
    }

  } else if (predColNum == 1) {
    Column *filter_column = table->columns[filterCol[0]];
    if (dataType == 0) {
      gpu_filter<long>((long *) filter_column->data_ptr, longFilterParams, profiler);
    } else if ((dataType == 2 && filter_column->type == LONG)) {
      longFilterParams[0] = intFilterParams[0];
      longFilterParams[1] = intFilterParams[1];
      gpu_filter<long>((long *) filter_column->data_ptr, longFilterParams, profiler);
    } else if (dataType == 1) {
      gpu_filter<double>((double *) filter_column->data_ptr, doubleFilterParams, profiler);
    } else if (dataType == 2) {
      gpu_filter<int32_t>((int32_t *) filter_column->data_ptr, intFilterParams, profiler);
    } else if (dataType == 3) {
      // Only consider the "=" predicate now.
      gpu_string_filter((char *) filter_column->data_ptr,
                        (int32_t *) filter_column->data_ptr_aux,
                        stringFilterParams,
                        profiler);
    } else if (dataType == 4 && mode == FILTER_NOT_NULL) { // not null
      switch (table->columns[filterCol[0]]->type) {
        case LONG: {
          gpu_filter<long>((long *) filter_column->data_ptr, longFilterParams, profiler);
          break;
        }
        case DOUBLE: {
          gpu_filter<double>((double *) filter_column->data_ptr, doubleFilterParams, profiler);
          break;
        }
        case INT: {
          gpu_filter<int32_t>((int32_t *) filter_column->data_ptr, intFilterParams, profiler);
          break;
        }
        case STRING: {
          gpu_string_filter_not_null((int32_t *) filter_column->data_ptr_aux, profiler);
          break;
        }
        case DEPEND: break;
      }
//      result_bitmap = thrust::device_vector<int>(row_num, (uint32_t)1);
    }
  } else if (predColNum == 2) {
    Column *filter_column_1 = table->columns[filterCol[0]];
    Column *filter_column_2 = table->columns[filterCol[1]];
    if (filter_column_1->type == LONG) {
      if (filter_column_2->type == LONG) {
        gpu_filter_two<long, long>((long *) filter_column_1->data_ptr, (long *) filter_column_2->data_ptr, profiler);
      } else if (filter_column_2->type == DOUBLE) {
        gpu_filter_two<long, double>((long *) filter_column_1->data_ptr,
                                     (double *) filter_column_2->data_ptr,
                                     profiler);
      } else if (filter_column_2->type == INT) {
        gpu_filter_two<int32_t, int32_t>((int32_t *) filter_column_1->data_ptr,
                                         (int32_t *) filter_column_2->data_ptr,
                                         profiler);
      }
    } else if (filter_column_1->type == DOUBLE) {
      if (filter_column_2->type == LONG) {
        gpu_filter_two<double, long>((double *) filter_column_1->data_ptr,
                                     (long *) filter_column_2->data_ptr,
                                     profiler);
      } else if (filter_column_2->type == DOUBLE) {
        gpu_filter_two<double, double>((double *) filter_column_1->data_ptr,
                                       (double *) filter_column_2->data_ptr,
                                       profiler);
      } else if (filter_column_2->type == INT) {
        gpu_filter_two<double, int32_t>((double *) filter_column_1->data_ptr,
                                        (int32_t *) filter_column_2->data_ptr,
                                        profiler);
      }
    } else if (filter_column_1->type == INT) {
      if (filter_column_2->type == LONG) {
        gpu_filter_two<int32_t, long>((int32_t *) filter_column_1->data_ptr,
                                      (long *) filter_column_2->data_ptr,
                                      profiler);
      } else if (filter_column_2->type == DOUBLE) {
        gpu_filter_two<int32_t, double>((int32_t *) filter_column_1->data_ptr,
                                        (double *) filter_column_2->data_ptr,
                                        profiler);
      } else if (filter_column_2->type == INT) {
        gpu_filter_two<int32_t, int32_t>((int32_t *) filter_column_1->data_ptr,
                                         (int32_t *) filter_column_2->data_ptr,
                                         profiler);
      }
    } else if (filter_column_1->type == STRING) {
      assert(filter_column_2->type == STRING);
      //TODO: Does not consider string col v.s. string col currently.
      std::cout << "GHive-CPP-ERROR [FilterPredicate-process]: Does not consider string col v.s. string col currently"
                << std::endl;
    }
  }
  std::cout << "GHive-CPP [FilterPredicate-process]: processing ends." << std::endl;
}

void FilterPredicate::cpu_process(Table *table) {
  std::cout << "GHive-CPP [FilterPredicate-process]: processing starts." << std::endl;
  std::cout << this->toString(0) << std::endl;
  row_num = table->row_num;

  if (predColNum == 0) { // and / or
    if (mode == FILTER_AND) {
      childrenPredicate[0]->cpu_process(table);
      childrenPredicate[1]->cpu_process(table);
      thrust::transform(childrenPredicate[0]->result_bitmap.begin(), childrenPredicate[0]->result_bitmap.end(),
                        childrenPredicate[1]->result_bitmap.begin(), result_bitmap.begin(),
                        filter_binary_and<int, int>());
    } else if (mode == FILTER_OR) {
      childrenPredicate[0]->cpu_process(table);
      childrenPredicate[1]->cpu_process(table);
      thrust::transform(childrenPredicate[0]->result_bitmap.begin(), childrenPredicate[0]->result_bitmap.end(),
                        childrenPredicate[1]->result_bitmap.begin(), result_bitmap.begin(),
                        filter_binary_or<int, int>());
    }

  } else if (predColNum == 1) {
    Column *filter_column = table->columns[filterCol[0]];
    if (dataType == 0) {
      cpu_filter<long>((long *) filter_column->data_ptr, longFilterParams);
    } else if ((dataType == 2 && filter_column->type == LONG)) {
      longFilterParams[0] = intFilterParams[0];
      longFilterParams[1] = intFilterParams[1];
      cpu_filter<long>((long *) filter_column->data_ptr, longFilterParams);
    } else if (dataType == 1) {
      cpu_filter<double>((double *) filter_column->data_ptr, doubleFilterParams);
    } else if (dataType == 2) {
      cpu_filter<int32_t>((int32_t *) filter_column->data_ptr, intFilterParams);
    } else if (dataType == 3) {
      // Only consider the "=" predicate now.
      cpu_string_filter((char *) filter_column->data_ptr, (int32_t *) filter_column->data_ptr_aux, stringFilterParams);
    } else if (dataType == 4) { // not null
      switch (table->columns[filterCol[0]]->type) {
        case LONG: {
          cpu_filter<long>((long *) filter_column->data_ptr, longFilterParams);
          break;
        }
        case DOUBLE: {
          cpu_filter<double>((double *) filter_column->data_ptr, doubleFilterParams);
          break;
        }
        case INT: {
          cpu_filter<int32_t>((int32_t *) filter_column->data_ptr, intFilterParams);
          break;
        }
        case STRING: {
          cpu_string_filter_not_null((int32_t *) filter_column->data_ptr_aux);
          break;
        }
        case DEPEND: break;
      }
    }
  } else if (predColNum == 2) {
    Column *filter_column_1 = table->columns[filterCol[0]];
    Column *filter_column_2 = table->columns[filterCol[1]];
    if (filter_column_1->type == LONG) {
      if (filter_column_2->type == LONG) {
        cpu_filter_two<long, long>((long *) filter_column_1->data_ptr, (long *) filter_column_2->data_ptr);
      } else if (filter_column_2->type == DOUBLE) {
        cpu_filter_two<long, double>((long *) filter_column_1->data_ptr, (double *) filter_column_2->data_ptr);
      } else if (filter_column_2->type == INT) {
        cpu_filter_two<int32_t, int32_t>((int32_t *) filter_column_1->data_ptr, (int32_t *) filter_column_2->data_ptr);
      }
    } else if (filter_column_1->type == DOUBLE) {
      if (filter_column_2->type == LONG) {
        cpu_filter_two<double, long>((double *) filter_column_1->data_ptr, (long *) filter_column_2->data_ptr);
      } else if (filter_column_2->type == DOUBLE) {
        cpu_filter_two<double, double>((double *) filter_column_1->data_ptr, (double *) filter_column_2->data_ptr);
      } else if (filter_column_2->type == INT) {
        cpu_filter_two<double, int32_t>((double *) filter_column_1->data_ptr, (int32_t *) filter_column_2->data_ptr);
      }
    } else if (filter_column_1->type == INT) {
      if (filter_column_2->type == LONG) {
        cpu_filter_two<int32_t, long>((int32_t *) filter_column_1->data_ptr, (long *) filter_column_2->data_ptr);
      } else if (filter_column_2->type == DOUBLE) {
        cpu_filter_two<int32_t, double>((int32_t *) filter_column_1->data_ptr, (double *) filter_column_2->data_ptr);
      } else if (filter_column_2->type == INT) {
        cpu_filter_two<int32_t, int32_t>((int32_t *) filter_column_1->data_ptr, (int32_t *) filter_column_2->data_ptr);
      }
    } else if (filter_column_1->type == STRING) {
      assert(filter_column_2->type == STRING);
      //TODO: Does not consider string col v.s. string col currently.
      std::cout << "GHive-CPP-ERROR [FilterPredicate-process]: Does not consider string col v.s. string col currently"
                << std::endl;
    }
  }
  std::cout << "GHive-CPP [FilterPredicate-process]: processing ends." << std::endl;
}

std::string FilterPredicate::toString() {
  return this->predicateLiteral;
}

std::string FilterPredicate::toString(int level) {
  std::string str;
  for (int i = 0; i < level; i++) {
    str += "  ";
  }
  str += "predicate: " + std::to_string(static_cast<int>(mode)) + "; ";
  if (mode == FILTER_AND || mode == FILTER_OR) {
    str += "filterCol: x;";
  } else {
    str += "filterCol: " + std::to_string(filterCol[0]) + ";";
  }
  str += "paramNum: " + std::to_string(paramNum) + ";";
  str += "params: ";
  if (paramNum == 0) {
    str += "x";
  } else if (dataType == 0) {
    for (int i = 0; i < paramNum; i++) {
      str += std::to_string(longFilterParams[i]) + " ";
    }
  } else if (dataType == 1) {
    for (int i = 0; i < paramNum; i++) {
      str += std::to_string(doubleFilterParams[i]) + " ";
    }
  } else if (dataType == 2) {
    for (int i = 0; i < paramNum; i++) {
      str += std::to_string(intFilterParams[i]) + " ";
    }
  } else if (dataType == 3) {
    for (int i = 0; i < paramNum; i++) {
      str += std::string(stringFilterParams[i]) + " ";
    }
  }
  str += ";\n";
  if (mode == FILTER_AND || mode == FILTER_OR) {
    if (childrenPredicate[0] != nullptr && childrenPredicate[1] != nullptr) {
      str += childrenPredicate[0]->toString(level + 1);
      str += childrenPredicate[1]->toString(level + 1);
    }
  }
  return str;
}
