#include <Operator/PTFPredicate.hpp>
#include <map>
#include <thrust/device_vector.h>
#include <regex>
#include <Operator/GroupByPredicate.hpp>
/*
DataFlow *PTFPredicate::execute(DataFlow *data_flow, Profiler &profiler){
  /*
  std::vector<uint32_t> sequence = data_flow->get_sequence();
  uint32_t row_num = data_flow->get_row_num();
  uint32_t partition_by_num = partition_by_col.size();
  uint32_t order_by_num =order_by_col.size();
  uint32_t window_function_num =window_functions.size();

  //string need one more col
  uint32_t partition_vec_size = partition_by_num ;
  for (unsigned int i : partition_by_col) {
    if (sequence[i] >= data_flow->longColNum + data_flow->doubleColNum + data_flow->intColNum) {
      partition_vec_size++;
    }
  }
  uint32_t order_by_vec_size = order_by_num ;
  for (unsigned int i : order_by_col) {
    if (sequence[i] >= data_flow->longColNum + data_flow->doubleColNum + data_flow->intColNum) {
      order_by_vec_size++;
    }
  }


  thrust::device_vector<const void *> d_key_vec(partition_vec_size+order_by_num);
  thrust::device_vector<const void *> d_partition_vec(partition_by_num);
  thrust::device_vector<int> d_keys_type(partition_vec_size+order_by_vec_size);

  //partition col
  int i=0;int l=0;
  cout<<"partition_by_num: "<<partition_by_num<<endl;
  for (; i < partition_by_num; i++, l++) {
    uint32_t ith_col = sequence[partition_by_col[i]];
    if (ith_col < data_flow->longColNum) {
      d_keys_type[i] = 0 ;
      std::cout << "partition long columns: " << std::endl;
      for (int x = 0; x < row_num; x ++) {
        std::cout << data_flow->longCols[ith_col][x] << " ";
        if (x > 1000) break;
      }
      cout<<endl;
      long *d_long_ptr;
      hipMalloc((void **) &d_long_ptr, row_num * sizeof(long));
      hipMemcpy(d_long_ptr, data_flow->longCols[ith_col], row_num * sizeof(long), hipMemcpyHostToDevice);
      d_key_vec[l] = d_long_ptr;
      d_partition_vec[l]=d_long_ptr;
    } else if (ith_col < data_flow->longColNum + data_flow->doubleColNum) {
      d_keys_type[i] = 1;
      uint32_t idx = ith_col - data_flow->longColNum;
      double *d_double_ptr;
      hipMalloc((void **) &d_double_ptr, row_num * sizeof(double));
      hipMemcpy(d_double_ptr, data_flow->doubleCols[idx], row_num * sizeof(double), hipMemcpyHostToDevice);
      d_key_vec[l] = d_double_ptr;
      d_partition_vec[l]=d_double_ptr;
    } else if (ith_col < data_flow->longColNum + data_flow->doubleColNum + data_flow->intColNum) {
      d_keys_type[i] = 2;
      uint32_t idx = ith_col - data_flow->longColNum - data_flow->doubleColNum;
      int *d_int_ptr;
      hipMalloc((void **) &d_int_ptr, row_num * sizeof(int));
      hipMemcpy(d_int_ptr, data_flow->intCols[idx], row_num * sizeof(int), hipMemcpyHostToDevice);
      d_key_vec[l] = d_int_ptr;
      d_partition_vec[l]=d_int_ptr;
    } else { // String
      d_keys_type[i] = 3;
      uint32_t idx = ith_col - data_flow->longColNum - data_flow->doubleColNum - data_flow->intColNum;
      char *d_str_ptr;
      int *d_str_idx_ptr;
      uint32_t size_char = 0;
      for (int x = 0; x < row_num; x++) {
        size_char = data_flow->strIdxCols[idx][2 * x + 1] > size_char ?
                    data_flow->strIdxCols[idx][2 * x + 1] : size_char;
      }
      hipMalloc((void **) &d_str_ptr, (size_char + 1) * sizeof(char));
      hipMemcpy(d_str_ptr, data_flow->strCols[idx], (size_char + 1) * sizeof(char), hipMemcpyHostToDevice);

      hipMalloc((void **) &d_str_idx_ptr, 2 * row_num * sizeof(int));
      hipMemcpy(d_str_idx_ptr, data_flow->strIdxCols[idx], 2 * row_num * sizeof(int), hipMemcpyHostToDevice);
      d_key_vec[l++] = d_str_ptr;
      d_key_vec[l] = d_str_idx_ptr;
      d_partition_vec[l]=d_str_ptr;
      d_partition_vec[l]=d_str_idx_ptr;
    }
  }
  //order_by col
  for (; i <partition_by_num+ order_by_num; i++, l++) {
    uint32_t ith_col = sequence[order_by_col[i]];
    if (ith_col < data_flow->longColNum) {
      d_keys_type[i] = 0 ;
      for (int x = 0; x < row_num; x ++) {
        std::cout << data_flow->longCols[ith_col][x] << " ";
        if (x > 1000) break;
      }
      long *d_long_ptr;
      hipMalloc((void **) &d_long_ptr, row_num * sizeof(long));
      hipMemcpy(d_long_ptr, data_flow->longCols[ith_col], row_num * sizeof(long), hipMemcpyHostToDevice);
      d_key_vec[l] = d_long_ptr;
    } else if (ith_col < data_flow->longColNum + data_flow->doubleColNum) {
      d_keys_type[i] = 1;
      uint32_t idx = ith_col - data_flow->longColNum;
      double *d_double_ptr;
      hipMalloc((void **) &d_double_ptr, row_num * sizeof(double));
      hipMemcpy(d_double_ptr, data_flow->doubleCols[idx], row_num * sizeof(double), hipMemcpyHostToDevice);
      d_key_vec[l] = d_double_ptr;
    } else if (ith_col < data_flow->longColNum + data_flow->doubleColNum + data_flow->intColNum) {
      d_keys_type[i] = 2;
      uint32_t idx = ith_col - data_flow->longColNum - data_flow->doubleColNum;
      int *d_int_ptr;
      hipMalloc((void **) &d_int_ptr, row_num * sizeof(int));
      hipMemcpy(d_int_ptr, data_flow->intCols[idx], row_num * sizeof(int), hipMemcpyHostToDevice);
      d_key_vec[l] = d_int_ptr;
    } else { // String
      d_keys_type[i] = 3;
      uint32_t idx = ith_col - data_flow->longColNum - data_flow->doubleColNum - data_flow->intColNum;
      char *d_str_ptr;
      int *d_str_idx_ptr;
      uint32_t size_char = 0;
      for (int x = 0; x < row_num; x++) {

        size_char = data_flow->strIdxCols[idx][2 * x + 1] > size_char ?
                    data_flow->strIdxCols[idx][2 * x + 1] : size_char;
      }
      hipMalloc((void **) &d_str_ptr, (size_char + 1) * sizeof(char));
      hipMemcpy(d_str_ptr, data_flow->strCols[idx], (size_char + 1) * sizeof(char), hipMemcpyHostToDevice);

      hipMalloc((void **) &d_str_idx_ptr, 2 * row_num * sizeof(int));
      hipMemcpy(d_str_idx_ptr, data_flow->strIdxCols[idx], 2 * row_num * sizeof(int), hipMemcpyHostToDevice);
      d_key_vec[l++] = d_str_ptr;
      d_key_vec[l] = d_str_idx_ptr;
    }
  }


  const void **d_keys_data_ptr = thrust::raw_pointer_cast(d_key_vec.data());
  const int *d_keys_type_ptr = thrust::raw_pointer_cast(d_keys_type.data());
  thrust::device_vector<int> d_result_idx(row_num);
  thrust::host_vector<int>h_result_idx(row_num);
  thrust::sequence(d_result_idx.begin(), d_result_idx.end());
  //TODO::CONSIDER ASC DEC!!!
  thrust::sort(d_result_idx.begin(), d_result_idx.end(), sort_comparator(d_keys_data_ptr, d_keys_type_ptr, order_by_num+partition_by_num));

  hipMemcpy(h_result_idx.data(), thrust::raw_pointer_cast(d_result_idx.data()),
             row_num * sizeof(int), hipMemcpyDeviceToHost);


  const void **d_partition_data_ptr = thrust::raw_pointer_cast(d_partition_vec.data());
  thrust::device_vector<int> d_result_keys_idx(row_num);
  thrust::host_vector<int> h_result_keys_idx;
  thrust::host_vector<void *> h_result_values(window_function_num);
  thrust::host_vector<uint32_t * >h_group_number(window_function_num);
  int values_type[window_function_num];
  thrust::host_vector<void *> window_value(window_function_num);

  uint32_t result_long_col_num = data_flow->longColNum;
  uint32_t result_double_col_num = data_flow->doubleColNum;
  uint32_t result_int_col_num = data_flow->intColNum;
  uint32_t result_str_col_num = data_flow->stringColNum;
  
  cout<<window_functions.size()<<endl;
  for (int j = 0; j < window_functions.size(); j++) {
    WindowFunction each_window = window_functions[j];
    uint32_t function_index = sequence[each_window.arguments];
    long result_size=0;
    switch (each_window.type) {
      case SUM: {
        thrust::device_vector<uint32_t> d_result_nums(row_num);
        //reduce for number
        auto cnt_end = thrust::reduce_by_key(d_result_idx.begin(), d_result_idx.end(),
                                                 thrust::constant_iterator<uint32_t>(1),
                                                 d_result_keys_idx.begin(), d_result_nums.begin(),
                                                 reducer_predicator(d_partition_data_ptr, d_keys_type_ptr,
                                                                    partition_by_num),
                                                 thrust::plus<uint32_t>());
        result_size = cnt_end.second - d_result_nums.begin();
        uint32_t *p_group_num = new uint32_t[result_size];
        hipMemcpy(p_group_num, thrust::raw_pointer_cast(d_result_nums.data()),
                   result_size * sizeof(uint32_t), hipMemcpyDeviceToHost);
        h_group_number[j] = p_group_num;

        if (function_index < data_flow->longColNum) {
          values_type[j] = 0;
          result_long_col_num++;
          thrust::device_vector<long> d_input_values(data_flow->longCols[function_index],
                                                     data_flow->longCols[function_index] + row_num);
          if (each_window.windowFrame == ROWS_PRECEDING_FOLLOWING) {
            //reduce for value
            thrust::device_vector<long> d_result_values(row_num);
            auto new_end = thrust::reduce_by_key(d_result_idx.begin(), d_result_idx.end(),
                                                       thrust::make_permutation_iterator(d_input_values.begin(),
                                                                                         d_result_idx.begin()),
                                                       d_result_keys_idx.begin(), d_result_values.begin(),
                                                       reducer_predicator(d_partition_data_ptr, d_keys_type_ptr,
                                                                          partition_by_num),
                                                       thrust::plus<long>());
            //h_result_keys_idx = thrust::host_vector<int>(d_result_keys_idx.begin(), new_end.first);
            result_size = new_end.second - d_result_values.begin();
            long *p_result = new long[result_size];
            hipMemcpy(p_result, thrust::raw_pointer_cast(d_result_values.data()),
                       result_size * sizeof(long), hipMemcpyDeviceToHost);
            h_result_values[j] = p_result;

            long *result_value = static_cast<long *>(h_result_values[j]);
            long *each_value = new long[row_num];
            uint32_t *each_num = h_group_number[j];
            for (int k = 0, m = 0, idx = 1; k < row_num & m < result_size; k++) {
              each_value[k] = result_value[m];
              idx++;
              if (idx > each_num[m]) {
                idx = 1;
                m++;
              }
            }
            window_value[j]=each_value;
          } else {
            long *each_value = new long[row_num];
            uint32_t *each_num = h_group_number[j];
            uint32_t idx = 0;
            for (int k = 0; k < result_size; k++) {
              thrust::inclusive_scan(thrust::make_permutation_iterator(d_input_values.begin(),
                                                                       d_result_idx.begin()) + idx,
                                     thrust::make_permutation_iterator(d_input_values.begin(),
                                                                       d_result_idx.begin()) + idx + each_num[k],
                                     each_value + idx, thrust::plus<long>());
              idx = each_num[k];
            }
            window_value[j]=each_value;
          }
        } 
        else if (function_index < data_flow->longColNum + data_flow->doubleColNum) {
          values_type[j] = 1;
          result_double_col_num++;
          uint32_t index = function_index - data_flow->longColNum;
          thrust::device_vector<double> d_input_values(data_flow->doubleCols[index],
                                                       data_flow->doubleCols[index] + row_num);

          if (each_window.windowFrame == ROWS_PRECEDING_FOLLOWING) {
            thrust::device_vector<double> d_result_values(row_num);
            //reduce for value
            auto new_end = thrust::reduce_by_key(d_result_idx.begin(), d_result_idx.end(),
                                                       thrust::make_permutation_iterator(d_input_values.begin(),
                                                                                         d_result_idx.begin()),
                                                       d_result_keys_idx.begin(), d_result_values.begin(),
                                                       reducer_predicator(d_partition_data_ptr, d_keys_type_ptr,
                                                                          partition_by_num),
                                                       thrust::plus<double>());
            //h_result_keys_idx = thrust::host_vector<int>(d_result_keys_idx.begin(), new_end.first);
            result_size = new_end.second - d_result_values.begin();
            double *p_result = new double[result_size];
            hipMemcpy(p_result, thrust::raw_pointer_cast(d_result_values.data()),
                       result_size * sizeof(double), hipMemcpyDeviceToHost);
            h_result_values[j] = p_result;

            double *result_value = static_cast<double *>(h_result_values[j]);
            double *each_value = new double[row_num];
            uint32_t *each_num = h_group_number[j];
            for (int k = 0, m = 0, idx = 1; k < row_num & m < result_size; k++) {
              each_value[k] = result_value[m];
              idx++;
              if (idx > each_num[m]) {
                idx = 1;
                m++;
              }
            }
            window_value[j]=each_value;
          } else {
            double *each_value = new double[row_num];
            uint32_t *each_num = h_group_number[j];
            uint32_t idx = 0;
            for (int k = 0; k < result_size; k++) {
              thrust::inclusive_scan(thrust::make_permutation_iterator(d_input_values.begin(),
                                                                       d_result_idx.begin()) + idx,
                                     thrust::make_permutation_iterator(d_input_values.begin(),
                                                                       d_result_idx.begin()) + idx + each_num[k],
                                     each_value + idx, thrust::plus<double>());
              idx = each_num[k];
            }
            window_value[j]=each_value;
          }
        } 
        else if (function_index < data_flow->longColNum + data_flow->doubleColNum + data_flow->intColNum) {
          values_type[j] = 2;
          result_int_col_num++;
          uint32_t index = function_index - data_flow->longColNum - data_flow->doubleColNum;
          thrust::device_vector<int> d_input_values(data_flow->intCols[index], data_flow->intCols[index] + row_num);

          if (each_window.windowFrame == ROWS_PRECEDING_FOLLOWING) {
            thrust::device_vector<int> d_result_values(row_num);
            //reduce for value
            auto new_end = thrust::reduce_by_key(d_result_idx.begin(), d_result_idx.end(),
                                                       thrust::make_permutation_iterator(d_input_values.begin(),
                                                                                         d_result_idx.begin()),
                                                       d_result_keys_idx.begin(), d_result_values.begin(),
                                                       reducer_predicator(d_partition_data_ptr, d_keys_type_ptr,
                                                                          partition_by_num),
                                                       thrust::plus<int>());
            //h_result_keys_idx = thrust::host_vector<int>(d_result_keys_idx.begin(), new_end.first);
            result_size = new_end.second - d_result_values.begin();
            int32_t *p_result = new int32_t[result_size];
            hipMemcpy(p_result, thrust::raw_pointer_cast(d_result_values.data()),
                       result_size * sizeof(int32_t), hipMemcpyDeviceToHost);
            h_result_values[j] = p_result;

            int *result_value = static_cast<int *>(h_result_values[j]);
            int *each_value = new int[row_num];
            uint32_t *each_num = h_group_number[j];
            for (int k = 0, m = 0, idx = 1; k < row_num & m < result_size; k++) {
              each_value[k] = result_value[m];
              idx++;
              if (idx > each_num[m]) {
                idx = 1;
                m++;
              }
            }
            cout << "debug_point_0" << endl;
            for (int z = 0; z < row_num; z++) {
              cout << each_value[z] << endl;
            }
            window_value[j] = each_value;

          } else {
            int *each_value = new int[row_num];
            uint32_t *each_num = h_group_number[j];
            uint32_t idx = 0;
            for (int k = 0; k < result_size; k++) {
              thrust::inclusive_scan(thrust::make_permutation_iterator(d_input_values.begin(),
                                                                       d_result_idx.begin()) + idx,
                                     thrust::make_permutation_iterator(d_input_values.begin(),
                                                                       d_result_idx.begin()) + idx + each_num[k],
                                     each_value + idx, thrust::plus<int32_t>());
              idx = each_num[k];
            }
            window_value[j] = each_value;
          }
        } 
        else {
          result_str_col_num++;
        }
        break;
      }
      case MAX: {
        thrust::device_vector<uint32_t> d_result_nums(row_num);
        //reduce for number
        auto cnt_end = thrust::reduce_by_key(d_result_idx.begin(), d_result_idx.end(),
                                                 thrust::constant_iterator<uint32_t>(1),
                                                 d_result_keys_idx.begin(), d_result_nums.begin(),
                                                 reducer_predicator(d_partition_data_ptr, d_keys_type_ptr,
                                                                    partition_by_num),
                                                 thrust::plus<uint32_t>());
        result_size = cnt_end.second - d_result_nums.begin();
        uint32_t *p_group_num = new uint32_t[result_size];
        hipMemcpy(p_group_num, thrust::raw_pointer_cast(d_result_nums.data()),
                   result_size * sizeof(uint32_t), hipMemcpyDeviceToHost);
        h_group_number[j] = p_group_num;

        if (function_index < data_flow->longColNum) {
          values_type[j] = 0;
          result_long_col_num++;
          thrust::device_vector<long> d_input_values(data_flow->longCols[function_index],
                                                     data_flow->longCols[function_index] + row_num);
          if (each_window.windowFrame == ROWS_PRECEDING_FOLLOWING) {
            //reduce for value
            thrust::device_vector<long> d_result_values(row_num);
            auto new_end = thrust::reduce_by_key(d_result_idx.begin(), d_result_idx.end(),
                                                       thrust::make_permutation_iterator(d_input_values.begin(),
                                                                                         d_result_idx.begin()),
                                                       d_result_keys_idx.begin(), d_result_values.begin(),
                                                       reducer_predicator(d_partition_data_ptr, d_keys_type_ptr,
                                                                          partition_by_num),
                                                       thrust::maximum<long>());
            //h_result_keys_idx = thrust::host_vector<int>(d_result_keys_idx.begin(), new_end.first);
            result_size = new_end.second - d_result_values.begin();
            long *p_result = new long[result_size];
            hipMemcpy(p_result, thrust::raw_pointer_cast(d_result_values.data()),
                       result_size * sizeof(long), hipMemcpyDeviceToHost);
            h_result_values[j] = p_result;

            long *result_value = static_cast<long *>(h_result_values[j]);
            long *each_value = new long[row_num];
            uint32_t *each_num = h_group_number[j];
            for (int k = 0, m = 0, idx = 1; k < row_num & m < result_size; k++) {
              each_value[k] = result_value[m];
              idx++;
              if (idx > each_num[m]) {
                idx = 1;
                m++;
              }
            }
            window_value[j]=each_value;
          } else {
            long *each_value = new long[row_num];
            uint32_t *each_num = h_group_number[j];
            uint32_t idx = 0;
            for (int k = 0; k < result_size; k++) {
              thrust::inclusive_scan(thrust::make_permutation_iterator(d_input_values.begin(),
                                                                       d_result_idx.begin()) + idx,
                                     thrust::make_permutation_iterator(d_input_values.begin(),
                                                                       d_result_idx.begin()) + idx + each_num[k],
                                     each_value + idx, thrust::maximum<long>());
              idx = each_num[k];
            }
            window_value[j]=each_value;
          }
        }
        else if (function_index < data_flow->longColNum + data_flow->doubleColNum) {
          values_type[j] = 1;
          result_double_col_num++;
          uint32_t index = function_index - data_flow->longColNum;
          thrust::device_vector<double> d_input_values(data_flow->doubleCols[index],
                                                       data_flow->doubleCols[index] + row_num);

          if (each_window.windowFrame == ROWS_PRECEDING_FOLLOWING) {
            thrust::device_vector<double> d_result_values(row_num);
            //reduce for value
            auto new_end = thrust::reduce_by_key(d_result_idx.begin(), d_result_idx.end(),
                                                       thrust::make_permutation_iterator(d_input_values.begin(),
                                                                                         d_result_idx.begin()),
                                                       d_result_keys_idx.begin(), d_result_values.begin(),
                                                       reducer_predicator(d_partition_data_ptr, d_keys_type_ptr,
                                                                          partition_by_num),
                                                       thrust::plus<double>());
            //h_result_keys_idx = thrust::host_vector<int>(d_result_keys_idx.begin(), new_end.first);
            result_size = new_end.second - d_result_values.begin();
            double *p_result = new double[result_size];
            hipMemcpy(p_result, thrust::raw_pointer_cast(d_result_values.data()),
                       result_size * sizeof(double), hipMemcpyDeviceToHost);
            h_result_values[j] = p_result;

            double *result_value = static_cast<double *>(h_result_values[j]);
            double *each_value = new double[row_num];
            uint32_t *each_num = h_group_number[j];
            for (int k = 0, m = 0, idx = 1; k < row_num & m < result_size; k++) {
              each_value[k] = result_value[m];
              idx++;
              if (idx > each_num[m]) {
                idx = 1;
                m++;
              }
            }
            window_value[j]=each_value;
          } else {
            double *each_value = new double[row_num];
            uint32_t *each_num = h_group_number[j];
            uint32_t idx = 0;
            for (int k = 0; k < result_size; k++) {
              thrust::inclusive_scan(thrust::make_permutation_iterator(d_input_values.begin(),
                                                                       d_result_idx.begin()) + idx,
                                     thrust::make_permutation_iterator(d_input_values.begin(),
                                                                       d_result_idx.begin()) + idx + each_num[k],
                                     each_value + idx, thrust::plus<double>());
              idx = each_num[k];
            }
            window_value[j]=each_value;
          }
        }
        else if (function_index < data_flow->longColNum + data_flow->doubleColNum + data_flow->intColNum) {
          values_type[j] = 2;
          result_int_col_num++;
          uint32_t index = function_index - data_flow->longColNum - data_flow->doubleColNum;
          thrust::device_vector<int> d_input_values(data_flow->intCols[index], data_flow->intCols[index] + row_num);

          if (each_window.windowFrame == ROWS_PRECEDING_FOLLOWING) {
            thrust::device_vector<int> d_result_values(row_num);
            //reduce for value
            auto new_end = thrust::reduce_by_key(d_result_idx.begin(), d_result_idx.end(),
                                                       thrust::make_permutation_iterator(d_input_values.begin(),
                                                                                         d_result_idx.begin()),
                                                       d_result_keys_idx.begin(), d_result_values.begin(),
                                                       reducer_predicator(d_partition_data_ptr, d_keys_type_ptr,
                                                                          partition_by_num),
                                                       thrust::plus<int32_t>());
            h_result_keys_idx = thrust::host_vector<int>(d_result_keys_idx.begin(), new_end.first);
            result_size = new_end.second - d_result_values.begin();
            int32_t *p_result = new int32_t[result_size];
            hipMemcpy(p_result, thrust::raw_pointer_cast(d_result_values.data()),
                       result_size * sizeof(int32_t), hipMemcpyDeviceToHost);
            h_result_values[j] = p_result;

            int *result_value = static_cast<int *>(h_result_values[j]);
            int *each_value = new int[row_num];
            uint32_t *each_num = h_group_number[j];
            for (int k = 0, m = 0, idx = 1; k < row_num & m < result_size; k++) {
              each_value[k] = result_value[m];
              idx++;
              if (idx > each_num[m]) {
                idx = 1;
                m++;
              }
            }
            cout << "debug_point_0" << endl;
            for (int z = 0; z < row_num; z++) {
              cout << each_value[z] << endl;
            }
            window_value[j] = each_value;

          } else {
            int *each_value = new int[row_num];
            uint32_t *each_num = h_group_number[j];
            uint32_t idx = 0;
            for (int k = 0; k < result_size; k++) {
              thrust::inclusive_scan(thrust::make_permutation_iterator(d_input_values.begin(),
                                                                       d_result_idx.begin()) + idx,
                                     thrust::make_permutation_iterator(d_input_values.begin(),
                                                                       d_result_idx.begin()) + idx + each_num[k],
                                     each_value + idx, thrust::plus<int32_t>());
              idx = each_num[k];
            }
            window_value[j] = each_value;
          }
        } 
        else {
          result_str_col_num++;
        }
        break;
      }
      case AVG: {
        values_type[j] = 1; // The result type of AVG is double.
        result_double_col_num++;
        thrust::device_vector<int> d_result_cnt(row_num);
        auto cnt_end = thrust::reduce_by_key(d_result_idx.begin(), d_result_idx.end(),
                                             thrust::constant_iterator<int>(1),
                                             d_result_keys_idx.begin(), d_result_cnt.begin(),
                                             reducer_predicator(d_partition_data_ptr, d_keys_type_ptr,
                                                                 partition_by_num),
                                             thrust::plus<int>());
        result_size = cnt_end.second - d_result_cnt.begin();
        uint32_t *p_group_num = new uint32_t[result_size];
        hipMemcpy(p_group_num, thrust::raw_pointer_cast(d_result_cnt.data()),
                   result_size * sizeof(uint32_t), hipMemcpyDeviceToHost);
        h_group_number[j] = p_group_num;
        thrust::device_vector<double> d_avg_result(row_num);

        if (function_index < data_flow->longColNum) {
          thrust::device_vector<long> d_input_values(data_flow->longCols[function_index],
                                                     data_flow->longCols[function_index] + row_num);
          auto new_end = thrust::reduce_by_key(d_result_idx.begin(), d_result_idx.end(),
                                               thrust::make_permutation_iterator(d_input_values.begin(),
                                                                                 d_result_idx.begin()),
                                               d_result_keys_idx.begin(), d_avg_result.begin(),
                                               reducer_predicator(d_partition_data_ptr, d_keys_type_ptr,
                                                                   partition_by_num),
                                               thrust::plus<long>());
          h_result_keys_idx = thrust::host_vector<int>(d_result_keys_idx.begin(), new_end.first);

          thrust::transform(d_avg_result.begin(), new_end.second, d_result_cnt.begin(),
                            d_avg_result.begin(), divides<double>());
          result_size = cnt_end.second - d_result_cnt.begin();
          double *p_result = new double[result_size];
          hipMemcpy(p_result, thrust::raw_pointer_cast(d_avg_result.data()),
                     result_size * sizeof(double), hipMemcpyDeviceToHost);
          h_result_values[j] = p_result;
          
          long * result_value = static_cast<long *>(h_result_values[j]);
          long * each_value = new long [row_num];
          uint32_t * each_num = h_group_number[j];
          for (int k=0,m=0,idx=1;k<row_num&l<result_size;k++){
            each_value[k]=result_value[m];
            idx++;
            if (idx>each_num[m]){
              idx=1;
              m++;
            }
          }
          window_value[j]=each_value;
        }
        else if (function_index < data_flow->longColNum + data_flow->doubleColNum) {
          uint32_t index = function_index - data_flow->longColNum;
          thrust::device_vector<double> d_input_values(data_flow->doubleCols[index],
                                                       data_flow->doubleCols[index] + row_num);
          auto new_end = thrust::reduce_by_key(d_result_idx.begin(), d_result_idx.end(),
                                               thrust::make_permutation_iterator(d_input_values.begin(),
                                                                                 d_result_idx.begin()),
                                               d_result_keys_idx.begin(), d_avg_result.begin(),
                                               reducer_predicator(d_partition_data_ptr, d_keys_type_ptr,
                                                                  order_by_num + partition_by_num),
                                               thrust::plus<double>());
          h_result_keys_idx = thrust::host_vector<int>(d_result_keys_idx.begin(), new_end.first);

          thrust::transform(d_avg_result.begin(), new_end.second, d_result_cnt.begin(),
                            d_avg_result.begin(), divides<double>());

          result_size = cnt_end.second - d_result_cnt.begin();
          double *p_result = new double[result_size];
          hipMemcpy(p_result, thrust::raw_pointer_cast(d_avg_result.data()),
                     result_size * sizeof(double), hipMemcpyDeviceToHost);
          h_result_values[j] = p_result;

          double * result_value = static_cast<double *>(h_result_values[j]);
          double * each_value = new double [row_num];
          uint32_t * each_num = h_group_number[j];
          for (int k=0,m=0,idx=1;k<row_num&l<result_size;k++){
            each_value[k]=result_value[m];
            idx++;
            if (idx>each_num[m]){
              idx=1;
              m++;
            }
          }
          window_value[j]=each_value;
        }
        else if (function_index < data_flow->longColNum + data_flow->doubleColNum + data_flow->intColNum) {
          uint32_t index = function_index - data_flow->longColNum - data_flow->doubleColNum;
          thrust::device_vector<int> d_input_values(data_flow->intCols[index], data_flow->intCols[index] + row_num);
          auto new_end = thrust::reduce_by_key(d_result_idx.begin(), d_result_idx.end(),
                                               thrust::make_permutation_iterator(d_input_values.begin(),
                                                                                 d_result_idx.begin()),
                                               d_result_keys_idx.begin(), d_avg_result.begin(),
                                               reducer_predicator(d_partition_data_ptr, d_keys_type_ptr,
                                                                  order_by_num + partition_by_num),
                                               thrust::plus<int>());
          h_result_keys_idx = thrust::host_vector<int>(d_result_keys_idx.begin(), new_end.first);

          thrust::transform(d_avg_result.begin(), new_end.second, d_result_cnt.begin(),
                            d_avg_result.begin(), divides<double>());

          result_size = cnt_end.second - d_result_cnt.begin();
          double *p_result = new double[result_size];
          hipMemcpy(p_result, thrust::raw_pointer_cast(d_avg_result.data()),
                     result_size * sizeof(double), hipMemcpyDeviceToHost);
          h_result_values[j] = p_result;

          double * result_value = static_cast<double *>(h_result_values[j]);
          double * each_value = new double [row_num];
          uint32_t * each_num = h_group_number[j];
          for (int k=0,m=0,idx=1;k<row_num&l<result_size;k++){
            each_value[k]=result_value[m];
            idx++;
            if (idx>each_num[m]){
              idx=1;
              m++;
            }
          }
          window_value[j]=each_value;
        }
        else {
        }
        break;
      }
      case RANK:{
        //always int
        values_type[j] = 2;
        result_int_col_num++;
        thrust::device_vector<uint32_t> d_result_nums(row_num);
        auto cnt_end = thrust::reduce_by_key(d_result_idx.begin(), d_result_idx.end(),
                                             thrust::constant_iterator<uint32_t>(1),
                                             d_result_keys_idx.begin(), d_result_nums.begin(),
                                             reducer_predicator(d_partition_data_ptr, d_keys_type_ptr,
                                                                order_by_num + partition_by_num),
                                             thrust::plus<uint32_t>());
        result_size = cnt_end.second - d_result_nums.begin();
        uint32_t *p_group_num = new uint32_t[result_size];
        hipMemcpy(p_group_num, thrust::raw_pointer_cast(d_result_nums.data()),
                   result_size * sizeof(uint32_t), hipMemcpyDeviceToHost);
        h_group_number[j] = p_group_num;
        int32_t  *each_value = new int32_t [row_num];
        uint32_t *each_num = h_group_number[j];
        uint32_t idx = 0;
        for (int k = 0; k < result_size; k++) {
          thrust::inclusive_scan(thrust::constant_iterator<uint32_t>(1),
                                 thrust::constant_iterator<uint32_t>(1) + each_num[k],
                                 each_value + idx, thrust::plus<uint32_t>());
          idx += each_num[k];
        }
        break;
      }
      case UNKNOWN:
        break;
    }

  }


  std::vector<uint32_t> seq;
  DataFlow *result_df = new DataFlow(result_long_col_num, result_double_col_num, result_int_col_num,
                                     result_str_col_num, row_num);
  result_df->initCols();


  
  std::cout << "result_long_col_num: " << result_long_col_num << std::endl;
  std::cout << "result_double_col_num: " << result_double_col_num << std::endl;
  std::cout << "result_int_col_num: " << result_int_col_num << std::endl;
  std::cout << "result_str_col_num: " << result_str_col_num << std::endl;

  uint32_t tmp_long_idx = 0;
  uint32_t tmp_double_idx = 0;
  uint32_t tmp_int_idx = 0;
  uint32_t tmp_string_idx= 0;
  for (int n = 0; n < data_flow->longColNum + data_flow->doubleColNum + data_flow->intColNum + data_flow->stringColNum; n++) {
    if (n < data_flow->longColNum){
      seq.push_back(tmp_long_idx++);
      std::cout << "FILE: " << __FILE__ << ", LINE: " << __LINE__ << std::endl;
      for(int p=0;p<row_num;p++){
        result_df->longCols[n][p]=data_flow->longCols[n][h_result_idx[p]];
      }
    }else if(n < data_flow->longColNum + data_flow->doubleColNum){
      seq.push_back(result_long_col_num+tmp_double_idx++);
      std::cout << "FILE: " << __FILE__ << ", LINE: " << __LINE__ << std::endl;
      for (int p=0;p<row_num;p++){
        result_df->doubleCols[n-data_flow->longColNum][p]=data_flow->doubleCols[n-data_flow->longColNum][h_result_idx[p]];
      }
    }else if(n < data_flow->longColNum + data_flow->doubleColNum + data_flow->intColNum) {
      seq.push_back(result_long_col_num+result_double_col_num+tmp_int_idx++);
      std::cout << "FILE: " << __FILE__ << ", LINE: " << __LINE__ << std::endl;
      for (int p = 0; p < row_num; p++) {
        result_df->intCols[n - data_flow->longColNum - data_flow->doubleColNum][p] =
            data_flow->intCols[n - data_flow->longColNum - data_flow->doubleColNum][h_result_idx[p]];
      }
    }else{
      seq.push_back(result_long_col_num+result_double_col_num+result_int_col_num+tmp_string_idx++);
      int32_t position=0;
      for (int p = 0; p < row_num; p++) {
        int length=data_flow->strIdxCols[n- data_flow->longColNum - data_flow->doubleColNum-data_flow->intColNum][2 * h_result_idx[p] + 1] - data_flow->strIdxCols[n- data_flow->longColNum - data_flow->doubleColNum-data_flow->intColNum][2 * h_result_idx[p]];
        string content=std::string(data_flow->strCols[n- data_flow->longColNum - data_flow->doubleColNum-data_flow->intColNum] + data_flow->strIdxCols[n- data_flow->longColNum - data_flow->doubleColNum-data_flow->intColNum][2 * h_result_idx[p]],
                                   length);
        cout<<content<<endl;
        result_df->strIdxCols[n - data_flow->longColNum - data_flow->doubleColNum - data_flow->intColNum][p*2]=position;
        result_df->strIdxCols[n - data_flow->longColNum - data_flow->doubleColNum - data_flow->intColNum][p*2+1]=position+length;
        for (int q=0;q<length;q++){
          result_df->strCols[n - data_flow->longColNum - data_flow->doubleColNum - data_flow->intColNum][position+q]=
              content[q];
        }
        position+=length;
      }
    }
  }

  for (int s = 0; s < window_function_num; s++) {
    if (values_type[s] == 0) {
      uint32_t value_idx = tmp_long_idx ++;
      seq.push_back(value_idx);
      result_df->longCols[value_idx] = (long *)window_value[s];
    } else if (values_type[s] == 1) {
      uint32_t value_idx = tmp_double_idx ++;
      seq.push_back(result_long_col_num + value_idx);
      result_df->doubleCols[value_idx] = (double *)window_value[s];
    } else if (values_type[s] == 2) {
      uint32_t value_idx = tmp_int_idx ++;
      seq.push_back(result_long_col_num + result_double_col_num + value_idx);
      result_df->intCols[value_idx] = (int32_t *)window_value[s];
      for(int x=0;x<row_num;x++){
        cout<<((int32_t*)window_value[s])[x];
      }
    } else {

    }
  }
  result_df->set_sequence(seq);
  result_df->keyNum =data_flow->keyNum;
  return result_df;
}
*/
Table *PTFPredicate::execute(Table *input_tbl, Profiler &profiler) {

  uint32_t row_num = input_tbl->row_num;
  uint32_t partition_by_num = partition_by_col.size();
  uint32_t order_by_num =order_by_col.size();
  uint32_t window_function_num =window_functions.size();

  //string need one more col
  uint32_t partition_vec_size = partition_by_num ;
  for (unsigned int i : partition_by_col) {
    if (input_tbl->columns[i]->type==STRING) {
      partition_vec_size++;
    }
  }
  uint32_t order_by_vec_size = order_by_num ;
  for (unsigned int i : order_by_col) {
    if (input_tbl->columns[i]->type==STRING) {
      order_by_vec_size++;
    }
  }


  thrust::device_vector<const void *> d_key_vec(partition_vec_size+order_by_num);
  thrust::device_vector<const void *> d_partition_vec(partition_by_num);
  thrust::device_vector<int> d_keys_type(partition_vec_size+order_by_vec_size);

  //partition col
  int i=0;int l=0;
  std::vector<SortOrder> sort_order(partition_by_num+order_by_num);
  cout<<"partition_by_num: "<<partition_by_num<<endl;
  for (; i < partition_by_num; i++, l++) {
    sort_order.push_back(SortOrder::ASC);
    if (input_tbl->columns[i]->type==LONG) {
      d_keys_type[i] = 0 ;
      std::cout << "partition long columns: " << std::endl;
      long *d_long_ptr;
      hipMalloc((void **) &d_long_ptr, row_num * sizeof(long));
      hipMemcpy(d_long_ptr, input_tbl->columns[i]->data_ptr, row_num * sizeof(long), hipMemcpyHostToDevice);
      d_key_vec[l] = d_long_ptr;
      d_partition_vec[l]=d_long_ptr;
    } else if (input_tbl->columns[i]->type==DOUBLE) {
      d_keys_type[i] = 1;
      double *d_double_ptr;
      hipMalloc((void **) &d_double_ptr, row_num * sizeof(double));
      hipMemcpy(d_double_ptr, input_tbl->columns[i]->data_ptr, row_num * sizeof(double), hipMemcpyHostToDevice);
      d_key_vec[l] = d_double_ptr;
      d_partition_vec[l]=d_double_ptr;
    } else if (input_tbl->columns[i]->type==INT) {
      d_keys_type[i] = 2;
      int *d_int_ptr;
      hipMalloc((void **) &d_int_ptr, row_num * sizeof(int));
      hipMemcpy(d_int_ptr, input_tbl->columns[i]->data_ptr, row_num * sizeof(int), hipMemcpyHostToDevice);
      d_key_vec[l] = d_int_ptr;
      d_partition_vec[l]=d_int_ptr;
    } else { // String
      d_keys_type[i] = 3;
      char *d_str_ptr;
      int *d_str_idx_ptr;
      uint32_t size_char = 0;
      for (int x = 0; x < row_num; x++) {
        size_char = ((int32_t *)(input_tbl->columns[i])->data_ptr_aux)[2 * x + 1] > size_char ?
                    ((int32_t *)(input_tbl->columns[i])->data_ptr_aux)[2 * x + 1] : size_char;
      }
      hipMalloc((void **) &d_str_ptr, size_char * sizeof(char));
      hipMemcpy(d_str_ptr, input_tbl->columns[i]->data_ptr, size_char * sizeof(char), hipMemcpyHostToDevice);
      hipMalloc((void **) &d_str_idx_ptr, 2 * row_num * sizeof(int));
      hipMemcpy(d_str_idx_ptr, input_tbl->columns[i]->data_ptr, 2 * row_num * sizeof(int), hipMemcpyHostToDevice);
      d_key_vec[l++] = d_str_ptr;
      d_key_vec[l] = d_str_idx_ptr;
      d_partition_vec[l]=d_str_ptr;
      d_partition_vec[l]=d_str_idx_ptr;
    }
  }
  //order_by col
  for (; i <partition_by_num+ order_by_num; i++, l++) {
    sort_order.push_back(asc[i]);
    if (input_tbl->columns[i]->type==LONG) {
      d_keys_type[i] = 0 ;
      long *d_long_ptr;
      hipMalloc((void **) &d_long_ptr, row_num * sizeof(long));
      hipMemcpy(d_long_ptr, input_tbl->columns[i]->data_ptr, row_num * sizeof(long), hipMemcpyHostToDevice);
      d_key_vec[l] = d_long_ptr;
    } else if (input_tbl->columns[i]->type==DOUBLE) {
      d_keys_type[i] = 1;
      double *d_double_ptr;
      hipMalloc((void **) &d_double_ptr, row_num * sizeof(double));
      hipMemcpy(d_double_ptr, input_tbl->columns[i]->data_ptr, row_num * sizeof(double), hipMemcpyHostToDevice);
      d_key_vec[l] = d_double_ptr;
    } else if (input_tbl->columns[i]->type==INT) {
      d_keys_type[i] = 2;
      int *d_int_ptr;
      hipMalloc((void **) &d_int_ptr, row_num * sizeof(int));
      hipMemcpy(d_int_ptr, input_tbl->columns[i]->data_ptr, row_num * sizeof(int), hipMemcpyHostToDevice);
      d_key_vec[l] = d_int_ptr;
    } else { // String
      d_keys_type[i] = 3;
      char *d_str_ptr;
      int *d_str_idx_ptr;
      uint32_t size_char = 0;
      for (int x = 0; x < row_num; x++) {
        size_char = ((int32_t *) (input_tbl->columns[i])->data_ptr_aux)[2 * x + 1] > size_char ?
                    ((int32_t *) (input_tbl->columns[i])->data_ptr_aux)[2 * x + 1] : size_char;
      }
      hipMalloc((void **) &d_str_ptr, size_char * sizeof(char ));
      hipMemcpy(d_str_ptr, input_tbl->columns[i]->data_ptr, size_char * sizeof(char), hipMemcpyHostToDevice);
      hipMalloc((void **) &d_str_idx_ptr, 2 * row_num * sizeof(int));
      hipMemcpy(d_str_idx_ptr, input_tbl->columns[i]->data_ptr, 2 * row_num * sizeof(int), hipMemcpyHostToDevice);

      d_key_vec[l++] = d_str_ptr;
      d_key_vec[l] = d_str_idx_ptr;
    }
  }


  const void **d_keys_data_ptr = thrust::raw_pointer_cast(d_key_vec.data());
  const int *d_keys_type_ptr = thrust::raw_pointer_cast(d_keys_type.data());
  thrust::device_vector<uint> d_result_idx(row_num);
  thrust::host_vector<int>h_result_idx(row_num);
  thrust::sequence(d_result_idx.begin(), d_result_idx.end());
  //TODO::CONSIDER ASC DEC!!! solved!
  //TODO::null first or not?
  thrust::sort(d_result_idx.begin(), d_result_idx.end(), sort_comparator(d_keys_data_ptr, d_keys_type_ptr, order_by_num+partition_by_num,sort_order.data()));

  hipMemcpy(h_result_idx.data(), thrust::raw_pointer_cast(d_result_idx.data()),
             row_num * sizeof(int), hipMemcpyDeviceToHost);


  const void **d_partition_data_ptr = thrust::raw_pointer_cast(d_partition_vec.data());
  thrust::device_vector<int> d_result_keys_idx(row_num);
  thrust::host_vector<int> h_result_keys_idx;
  thrust::host_vector<void *> h_result_values(window_function_num);
  thrust::host_vector<uint32_t * >h_group_number(window_function_num);
  struct reducer_predicator rp(d_partition_data_ptr, d_keys_type_ptr, partition_by_num);

  std::vector<Column *> result_columns;

  for (int j = 0; j < window_functions.size(); j++) {

    WindowFunction each_window = window_functions[j];
    Column *column = input_tbl->columns[each_window.arguments];

    switch (each_window.type) {
      case SUM: {
        switch (column->type) {
          case LONG: {
            result_columns[i] = aggregation_sum<long>(d_result_idx, input_tbl->columns[each_window.arguments], row_num,
                                                      rp, each_window.window_frame);
            break;
          }
          case DOUBLE: {
            result_columns[i] = aggregation_sum<double>(d_result_idx, input_tbl->columns[each_window.arguments],
                                                        row_num, rp, each_window.window_frame);
            break;
          }
          case INT: {
            result_columns[i] = aggregation_sum<int>(d_result_idx, input_tbl->columns[each_window.arguments], row_num,
                                                     rp, each_window.window_frame);
            break;
          }
        }
        break;
      }

      case MAX: {
        switch (column->type) {
          case LONG: {
            result_columns[i] = aggregation_max<long>(d_result_idx, input_tbl->columns[each_window.arguments], row_num,
                                                      rp, each_window.window_frame);
            break;
          }
          case DOUBLE: {
            result_columns[i] = aggregation_max<double>(d_result_idx, input_tbl->columns[each_window.arguments],
                                                        row_num, rp, each_window.window_frame);
            break;
          }
          case INT: {
            result_columns[i] = aggregation_max<int>(d_result_idx, input_tbl->columns[each_window.arguments], row_num,
                                                     rp, each_window.window_frame);
            break;
          }
        }
        break;
      }
      case AVG: {
        switch (column->type) {
          case LONG: {
            result_columns[i] = aggregation_avg<long>(d_result_idx, input_tbl->columns[each_window.arguments], row_num,
                                                      rp, each_window.window_frame);
            break;
          }
          case DOUBLE: {
            result_columns[i] = aggregation_avg<double>(d_result_idx, input_tbl->columns[each_window.arguments],
                                                        row_num, rp, each_window.window_frame);
            break;
          }
          case INT: {
            result_columns[i] = aggregation_avg<int>(d_result_idx, input_tbl->columns[each_window.arguments], row_num,
                                                     rp, each_window.window_frame);
            break;
          }
        }
        break;
      }
      //todo:RANK depends on sort
      case RANK: {
        switch (column->type) {
          case LONG: {
            result_columns[i] = aggregation_rank<long>(d_result_idx, row_num,
                                                      rp, each_window.window_frame);
            break;
          }
          case DOUBLE: {
            result_columns[i] = aggregation_rank<double>(d_result_idx,
                                                        row_num, rp, each_window.window_frame);
            break;
          }
          case INT: {
            result_columns[i] = aggregation_rank<int>(d_result_idx, row_num,
                                                     rp, each_window.window_frame);
            break;
          }
        }
        break;
      }
    }
  }

  Table* out_tbl= new Table();
  out_tbl->row_num=input_tbl->row_num;
  for (auto & column : input_tbl->columns) {
    out_tbl->columns.push_back(column);
  }

  for (int s = 0; s < window_function_num; s++) {
    out_tbl->columns.push_back(result_columns[s]);
  }
  return out_tbl;

}


template<typename T>
Column *PTFPredicate::aggregation_sum(thrust::device_vector<uint32_t> &d_result_idx,
                                          Column *column, uint32_t row_num, struct reducer_predicator &rp,windowFrame window_frame) {
  thrust::device_vector<int> d_result_keys_idx(row_num);
  thrust::device_vector<uint32_t> d_result_nums(row_num);
  //reduce for number
  auto cnt_end = thrust::reduce_by_key(d_result_idx.begin(), d_result_idx.end(),
                                       thrust::constant_iterator<uint32_t>(1),
                                       d_result_keys_idx.begin(), d_result_nums.begin(),
                                       rp,
                                       thrust::plus<uint32_t>());
  uint result_size = cnt_end.second - d_result_nums.begin();
  uint32_t *p_group_num = new uint32_t[result_size];
  hipMemcpy(p_group_num, thrust::raw_pointer_cast(d_result_nums.data()),
             result_size * sizeof(uint32_t), hipMemcpyDeviceToHost);
  thrust::device_vector<T> d_input_values((T *) column->data_ptr,
                                             (T *)column->data_ptr + row_num);
  if (window_frame == ROWS_PRECEDING_FOLLOWING) {
    //reduce for value
    thrust::device_vector<T> d_result_values(row_num);
    auto new_end = thrust::reduce_by_key(d_result_idx.begin(), d_result_idx.end(),
                                         thrust::make_permutation_iterator(d_input_values.begin(),
                                                                           d_result_idx.begin()),
                                         d_result_keys_idx.begin(), d_result_values.begin(),
                                         rp,
                                         newPlus<T>());
    result_size = new_end.second - d_result_values.begin();
    T *p_result = new T[result_size];
    hipMemcpy(p_result, thrust::raw_pointer_cast(d_result_values.data()),
               result_size * sizeof(T), hipMemcpyDeviceToHost);
    T *result_value = static_cast<T *>(p_result);
    T *each_value = new T[row_num];
    uint32_t *each_num = p_group_num;
    for (int k = 0, m = 0, idx = 1; k < row_num & m < result_size; k++) {
      each_value[k] = result_value[m];
      idx++;
      if (idx > each_num[m]) {
        idx = 1;
        m++;
      }
    }
    return new Column(column->type, row_num, each_value);
  } else {
    T *each_value = new T[row_num];
    uint32_t *each_num = p_group_num;
    uint32_t idx = 0;
    for (int k = 0; k < result_size; k++) {
      thrust::inclusive_scan(thrust::make_permutation_iterator(d_input_values.begin(),
                                                               d_result_idx.begin()) + idx,
                             thrust::make_permutation_iterator(d_input_values.begin(),
                                                               d_result_idx.begin()) + idx + each_num[k],
                             each_value + idx, newPlus<T>());
      idx = each_num[k];
    }
    return new Column(column->type, row_num, each_value);
  }
}


template<typename T>
Column *PTFPredicate::aggregation_max(thrust::device_vector<uint32_t> &d_result_idx,
                                          Column *column, uint32_t row_num, struct reducer_predicator &rp,windowFrame window_frame) {

  thrust::device_vector<int> d_result_keys_idx(row_num);
  thrust::device_vector<uint32_t> d_result_nums(row_num);
  //reduce for number
  auto cnt_end = thrust::reduce_by_key(d_result_idx.begin(), d_result_idx.end(),
                                       thrust::constant_iterator<uint32_t>(1),
                                       d_result_keys_idx.begin(), d_result_nums.begin(),
                                       rp,
                                       thrust::plus<uint32_t>());
  uint result_size = cnt_end.second - d_result_nums.begin();
  uint32_t *p_group_num = new uint32_t[result_size];
  hipMemcpy(p_group_num, thrust::raw_pointer_cast(d_result_nums.data()),
             result_size * sizeof(uint32_t), hipMemcpyDeviceToHost);
  thrust::device_vector<T> d_input_values((T *) column->data_ptr,
                                          (T *)column->data_ptr + row_num);
  if (window_frame == ROWS_PRECEDING_FOLLOWING) {
    //reduce for value
    thrust::device_vector<T> d_result_values(row_num);
    auto new_end = thrust::reduce_by_key(d_result_idx.begin(), d_result_idx.end(),
                                         thrust::make_permutation_iterator(d_input_values.begin(),
                                                                           d_result_idx.begin()),
                                         d_result_keys_idx.begin(), d_result_values.begin(),
                                         rp,
                                         newMaximum<T>());
    result_size = new_end.second - d_result_values.begin();
    T *p_result = new T[result_size];
    hipMemcpy(p_result, thrust::raw_pointer_cast(d_result_values.data()),
               result_size * sizeof(T), hipMemcpyDeviceToHost);
    T *result_value = static_cast<T *>(p_result);
    T *each_value = new T[row_num];
    uint32_t *each_num = p_group_num;
    for (int k = 0, m = 0, idx = 1; k < row_num & m < result_size; k++) {
      each_value[k] = result_value[m];
      idx++;
      if (idx > each_num[m]) {
        idx = 1;
        m++;
      }
    }
    return new Column(column->type, row_num, each_value);
  } else {
    T *each_value = new T[row_num];
    uint32_t *each_num = p_group_num;
    uint32_t idx = 0;
    for (int k = 0; k < result_size; k++) {
      thrust::inclusive_scan(thrust::make_permutation_iterator(d_input_values.begin(),
                                                               d_result_idx.begin()) + idx,
                             thrust::make_permutation_iterator(d_input_values.begin(),
                                                               d_result_idx.begin()) + idx + each_num[k],
                             each_value + idx, newMaximum<T>());
      idx = each_num[k];
    }
    return new Column(column->type, row_num, each_value);
  }

}

template<typename T>
Column *PTFPredicate::aggregation_avg(thrust::device_vector<uint32_t> &d_result_idx,
                                          Column *column, uint32_t row_num, struct reducer_predicator &rp,windowFrame window_frame) {
  thrust::device_vector<int> d_result_keys_idx(row_num);
  thrust::device_vector<int> d_result_cnt(row_num);
  thrust::device_vector<T> d_input_values((T*)(column)->data_ptr,
                                          (T*)(column)->data_ptr + row_num);
  auto cnt_end = thrust::reduce_by_key(d_result_idx.begin(), d_result_idx.end(),
                                       thrust::make_permutation_iterator(d_input_values.begin(),
                                                                         d_result_idx.begin()),
                                       d_result_keys_idx.begin(), d_result_cnt.begin(),
                                       rp,countPlus<int>());
  uint32_t result_size = cnt_end.second - d_result_cnt.begin();
  uint32_t *p_group_num = new uint32_t[result_size];
  hipMemcpy(p_group_num, thrust::raw_pointer_cast(d_result_cnt.data()),
             result_size * sizeof(uint32_t), hipMemcpyDeviceToHost);

  thrust::device_vector<double> d_avg_result(row_num);
    auto new_end = thrust::reduce_by_key(d_result_idx.begin(), d_result_idx.end(),
                                         thrust::make_permutation_iterator(d_input_values.begin(),
                                                                           d_result_idx.begin()),
                                         d_result_keys_idx.begin(), d_avg_result.begin(),
                                         rp,
                                         newPlus<T>());

    thrust::transform(d_avg_result.begin(), new_end.second, d_result_cnt.begin(),
                      d_avg_result.begin(), divides<double>());
    result_size = cnt_end.second - d_result_cnt.begin();
    double *p_result = new double[result_size];
    hipMemcpy(p_result, thrust::raw_pointer_cast(d_avg_result.data()),
               result_size * sizeof(double), hipMemcpyDeviceToHost);

    double * result_value = static_cast<double *>(p_result);
    double * each_value = new double [row_num];
    uint32_t * each_num = p_group_num;
    for (int k=0,m=0,idx=1;k<row_num&m<result_size;k++){
      each_value[k]=result_value[m];
      idx++;
      if (idx>each_num[m]){
        idx=1;
        m++;
      }
    }
  return new Column(DOUBLE,row_num,each_value);
  }

template <typename T>
Column *PTFPredicate::aggregation_rank(thrust::device_vector<uint32_t> &d_result_idx,
                                          uint32_t row_num, struct reducer_predicator &rp,windowFrame window_frame) {
  thrust::device_vector<uint32_t> d_result_nums(row_num);
  thrust::device_vector<int> d_result_keys_idx(row_num);
  auto cnt_end = thrust::reduce_by_key(d_result_idx.begin(), d_result_idx.end(),
                                       thrust::constant_iterator<uint32_t>(1),
                                       d_result_keys_idx.begin(), d_result_nums.begin(),
                                       rp,
                                       thrust::plus<uint32_t>());
  uint32_t result_size = cnt_end.second - d_result_nums.begin();
  uint32_t *p_group_num = new uint32_t[result_size];
  hipMemcpy(p_group_num, thrust::raw_pointer_cast(d_result_nums.data()),
             result_size * sizeof(uint32_t), hipMemcpyDeviceToHost);
  int32_t  *each_value = new int32_t [row_num];
  uint32_t *each_num = p_group_num;
  uint32_t idx = 0;
  for (int k = 0; k < result_size; k++) {
    thrust::inclusive_scan(thrust::constant_iterator<uint32_t>(1),
                           thrust::constant_iterator<uint32_t>(1) + each_num[k],
                           each_value + idx, thrust::plus<uint32_t>());
    idx += each_num[k];
  }
  return new Column(INT,row_num,each_value);
}

