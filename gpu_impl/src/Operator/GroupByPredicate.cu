#include <iostream>
#include <unordered_map>
#include <Profile/Profiler.hpp>
#include <DataFlow/Table.hpp>
#include <Util/Util.hpp>
#include <Profile/SortGroupByProfiler.hpp>
//#include "Util/Util.hpp"
#include "Operator/GroupByPredicate.hpp"
#include "Operator/GroupByAggregation.cuh"
#include "Operator/GroupByOperator.cuh"
using namespace std;

bool operator==(SortOrder order, char val) {
  return val == static_cast<char> (order);
}

GroupByPredicate::GroupByPredicate() {}

Table *GroupByPredicate::gpu_execute(Table *input_tbl, SortGroupByProfiler &profiler) {

  std::cout << "GHive-CPP [GroupByPredicate-execute]: gpu_execute starts executing" << std::endl;
  std::cout << "GHive-CPP [GroupByPredicate-execute]: keys: " << vector_to_string<uint32_t>(keys) << std::endl;
  uint32_t row_num = input_tbl->row_num;
  uint32_t key_num = keys.size();//the number of group by columns.
  uint32_t value_num = aggregation_descs.size();

  uint32_t key_vec_size = key_num;
  for (uint32_t key: keys) {
    if (input_tbl->columns[key]->type == STRING) {
      key_vec_size++;
    }
  }
  std::cout << "GHive-CPP [GroupByPredicate-execute]: Collect data and prepare sorting" << std::endl;
  profiler.start_gpu_alloc();
  profiler.start_pci_host2device();
  thrust::device_vector<const void *> d_keys_vec(key_vec_size);
  thrust::device_vector<int> d_keys_type(key_num);
  profiler.end_gpu_alloc();
  for (uint32_t i = 0, l = 0; i < keys.size(); i++, l++) {
    Column *column = input_tbl->columns[keys[i]];
    switch (column->type) {
      case LONG: {
        d_keys_type[i] = 0;
        long *d_long_ptr;
        hipMalloc((void **) &d_long_ptr, row_num * sizeof(long));
        hipMemcpy(d_long_ptr, column->data_ptr, row_num * sizeof(long), hipMemcpyHostToDevice);
        d_keys_vec[l] = d_long_ptr;
        break;
      }
      case DOUBLE: {
        d_keys_type[i] = 1;
        double *d_double_ptr;
        hipMalloc((void **) &d_double_ptr, row_num * sizeof(double));
        hipMemcpy(d_double_ptr, column->data_ptr, row_num * sizeof(double), hipMemcpyHostToDevice);
        d_keys_vec[l] = d_double_ptr;
        break;
      }
      case INT: {
        d_keys_type[i] = 2;
        int32_t *d_int_ptr;
        hipMalloc((void **) &d_int_ptr, row_num * sizeof(int32_t));
        hipMemcpy(d_int_ptr, column->data_ptr, row_num * sizeof(int32_t), hipMemcpyHostToDevice);
        d_keys_vec[l] = d_int_ptr;
        break;
      }
      case STRING: {
        d_keys_type[i] = 3;
        char *d_str_ptr = nullptr;
        int32_t *d_str_idx_ptr = nullptr;
        int32_t size_char = 0;
        for (int x = 0; x < row_num; x++) {
          // std::cout << "x = " << x << std::endl;
          size_char = ((int32_t *) column->data_ptr_aux)[2 * x] > size_char ?
                      ((int32_t *) column->data_ptr_aux)[2 * x] : size_char;
          // std::cout << "x = " << x << std::endl;
          size_char = ((int32_t *) column->data_ptr_aux)[2 * x + 1] > size_char ?
                      ((int32_t *) column->data_ptr_aux)[2 * x + 1] : size_char;
        }
        hipMalloc((void **) &d_str_ptr, size_char * sizeof(char));
        hipMemcpy(d_str_ptr, column->data_ptr, size_char * sizeof(char), hipMemcpyHostToDevice);
        hipMalloc((void **) &d_str_idx_ptr, 2 * row_num * sizeof(int32_t));
        hipMemcpy(d_str_idx_ptr, column->data_ptr_aux, 2 * row_num * sizeof(int32_t), hipMemcpyHostToDevice);
        d_keys_vec[l++] = d_str_ptr;
        d_keys_vec[l] = d_str_idx_ptr;
        break;
      }
      case DEPEND: {
        break;
      }
    }
  }


  profiler.start_gpu_alloc();
  const void **d_keys_data_ptr = thrust::raw_pointer_cast(d_keys_vec.data());
  const int *d_keys_type_ptr = thrust::raw_pointer_cast(d_keys_type.data());
  struct reducer_predicator rp(d_keys_data_ptr, d_keys_type_ptr, key_num);

  thrust::device_vector<uint32_t> d_result_idx(row_num);
  thrust::sequence(d_result_idx.begin(), d_result_idx.end());
  profiler.end_gpu_alloc();
  profiler.end_pci_host2device();
  profiler.start_sort();
  profiler.start_gpu_exec();
  thrust::sort(d_result_idx.begin(), d_result_idx.end(), sort_comparator(d_keys_data_ptr, d_keys_type_ptr, key_num));
  profiler.end_gpu_exec();
  profiler.end_sort();

  std::cout << "GHive-CPP [GroupByPredicate-execute]: Finish gpu sorting" << std::endl;

  thrust::host_vector<uint32_t> h_result_keys_idx;

  std::vector<Column *> result_columns;

  if (aggregation_descs.size() == 0) {
    no_aggregation(d_result_idx, h_result_keys_idx, rp, profiler);
  }
  for (int i = 0; i < aggregation_descs.size(); i++) {
    AggregationDesc desc = aggregation_descs[i];
    std::cout << "GHive-CPP [GroupByPredicate-execute]: desc_index: " << desc.index << std::endl;
    Column *column = input_tbl->columns[desc.index];
    switch (desc.type) {
      case SUM: {
        switch (column->type) {
          case LONG: {
            result_columns.push_back(aggregation_sum<long>(d_result_idx,
                                                           h_result_keys_idx,
                                                           column,
                                                           row_num,
                                                           rp,
                                                           profiler));
            break;
          }
          case DOUBLE: {
            result_columns.push_back(aggregation_sum<double>(d_result_idx,
                                                             h_result_keys_idx,
                                                             column,
                                                             row_num,
                                                             rp,
                                                             profiler));
            break;
          }
          case INT: {
            result_columns.push_back(aggregation_sum<int32_t>(d_result_idx,
                                                              h_result_keys_idx,
                                                              column,
                                                              row_num,
                                                              rp,
                                                              profiler));
            break;
          }
          case STRING:
          case DEPEND:
            std::cout << "GHive-CPP-ERROR [GroupByPredicate-execute]: Unsupported type for SUM operation"
                      << column->type << std::endl;
            break;
        }
        break;
      }
      case MAX: {
        switch (column->type) {
          case LONG: {
            result_columns.push_back(aggregation_max<long>(d_result_idx,
                                                           h_result_keys_idx,
                                                           column,
                                                           row_num,
                                                           rp,
                                                           profiler));
            break;
          }
          case DOUBLE: {
            result_columns.push_back(aggregation_max<double>(d_result_idx,
                                                             h_result_keys_idx,
                                                             column,
                                                             row_num,
                                                             rp,
                                                             profiler));
            break;
          }
          case INT: {
            result_columns.push_back(aggregation_max<int32_t>(d_result_idx,
                                                              h_result_keys_idx,
                                                              column,
                                                              row_num,
                                                              rp,
                                                              profiler));
            break;
          }
          case STRING:
          case DEPEND:
            std::cout << "GHive-CPP-ERROR [GroupByPredicate-execute]: Unsupported type for MAX operation"
                      << column->type << std::endl;
            break;
        }
        break;
      }
      case MIN: {
        switch (column->type) {
          case LONG: {
            result_columns.push_back(aggregation_min<long>(d_result_idx,
                                                           h_result_keys_idx,
                                                           column,
                                                           row_num,
                                                           rp,
                                                           profiler));
            break;
          }
          case DOUBLE: {
            result_columns.push_back(aggregation_min<double>(d_result_idx,
                                                             h_result_keys_idx,
                                                             column,
                                                             row_num,
                                                             rp,
                                                             profiler));
            break;
          }
          case INT: {
            result_columns.push_back(aggregation_min<int32_t>(d_result_idx,
                                                              h_result_keys_idx,
                                                              column,
                                                              row_num,
                                                              rp,
                                                              profiler));
            break;
          }
          case STRING:
          case DEPEND:
            std::cout << "GHive-CPP-ERROR [GroupByPredicate-execute]: Unsupported type for MIN operation"
                      << column->type << std::endl;
            break;
        }
        break;
      }
      case AVG: {
        switch (column->type) {
          case LONG: {
            result_columns.push_back(aggregation_avg<long>(d_result_idx,
                                                           h_result_keys_idx,
                                                           column,
                                                           row_num,
                                                           rp,
                                                           profiler));
            break;
          }
          case DOUBLE: {
            result_columns.push_back(aggregation_avg<double>(d_result_idx,
                                                             h_result_keys_idx,
                                                             column,
                                                             row_num,
                                                             rp,
                                                             profiler));
            break;
          }
          case INT: {
            result_columns.push_back(aggregation_avg<int32_t>(d_result_idx,
                                                              h_result_keys_idx,
                                                              column,
                                                              row_num,
                                                              rp,
                                                              profiler));
            break;
          }
          case STRING:
          case DEPEND:
            std::cout << "GHive-CPP-ERROR [GroupByPredicate-execute]: Unsupported type for MIN operation"
                      << column->type << std::endl;
            break;
        }
        break;
      }
      case CNT: {
        result_columns.push_back(aggregation_cnt(d_result_idx, h_result_keys_idx, row_num, rp, profiler));
        break;
      }
      case RANK: {
        break;
      }
      case UNKNOWN: {
        break;
      }
    }
  }
  profiler.start_data_recover();
  for (int32_t i = keys.size() - 1; i >= 0; i--) {
    Column *key_column = input_tbl->columns[keys[i]];
    switch (key_column->type) {
      case LONG: {
        long *original_col = (long *) key_column->data_ptr;
        uint32_t new_size = h_result_keys_idx.size();
        long *new_col = new long[new_size];
        for (uint32_t j = 0; j < new_size; j++) {
          new_col[j] = original_col[h_result_keys_idx[j]];
        }
        Column *result_col = new Column(LONG, new_size, new_col);
        result_columns.insert(result_columns.begin(), result_col);
        break;
      }
      case DOUBLE: {
        double *original_col = (double *) key_column->data_ptr;
        uint32_t new_size = h_result_keys_idx.size();
        double *new_col = new double[new_size];
        for (uint32_t j = 0; j < new_size; j++) {
          new_col[j] = original_col[h_result_keys_idx[j]];
        }
        Column *result_col = new Column(DOUBLE, new_size, new_col);
        result_columns.insert(result_columns.begin(), result_col);
        break;
      }
      case INT: {
        int32_t *original_col = (int32_t *) key_column->data_ptr;
        uint32_t new_size = h_result_keys_idx.size();
        int32_t *new_col = new int32_t[new_size];
        for (uint32_t j = 0; j < new_size; j++) {
          new_col[j] = original_col[h_result_keys_idx[j]];
        }
        Column *result_col = new Column(INT, new_size, new_col);
        result_columns.insert(result_columns.begin(), result_col);
        break;
      }
      case STRING: {
        int32_t *str_idx_col = (int32_t *) key_column->data_ptr_aux;
        uint32_t new_size = h_result_keys_idx.size();
        uint32_t char_size = 0;
        for (uint32_t j = 0; j < new_size; j++) {
          uint32_t idx = h_result_keys_idx[j];
          if (str_idx_col[2 * idx] >= 0) {
            char_size += str_idx_col[2 * idx + 1] - str_idx_col[2 * idx];
          }
        }
        char *str_col_old = (char *) key_column->data_ptr;
        char *str_col_new = new char[char_size];
        int32_t *str_idx_col_new = new int32_t[new_size * 2];
        uint32_t char_idx = 0;
        for (uint32_t j = 0; j < new_size; j++) {
          uint32_t idx = h_result_keys_idx[j];
          if (str_idx_col[2 * idx] >= 0) {
            uint32_t str_len = str_idx_col[2 * idx + 1] - str_idx_col[2 * idx];
            memcpy(str_col_new + char_idx, str_col_old + str_idx_col[2 * idx], str_len * sizeof(char));
            str_idx_col_new[2 * j] = char_idx;
            char_idx += str_len;
            str_idx_col_new[2 * j + 1] = char_idx;
          }
        }
        Column *column = new Column(STRING, new_size, str_col_new, str_idx_col_new, char_size);
        result_columns.insert(result_columns.begin(), column);
      }
      case DEPEND:break;
    }
  }
  Table *result_tbl = new Table();
  result_tbl->columns = result_columns;
  if (key_num != 0) {
    result_tbl->row_num = h_result_keys_idx.size();
  } else {
    result_tbl->row_num = 1;
  }
  profiler.end_data_recover();
  return result_tbl;
}

Table *GroupByPredicate::cpu_execute(Table *input_tbl, SortGroupByProfiler &profiler) {
  profiler.start_sort();
  uint32_t row_num = input_tbl->row_num;
  uint32_t key_num = keys.size();//the number of group by columns.
  uint32_t value_num = aggregation_descs.size();

  uint32_t key_vec_size = key_num;
  for (uint32_t key: keys) {
    if (input_tbl->columns[key]->type == STRING) {
      key_vec_size++;
    }
  }

  thrust::host_vector<const void *> h_keys_vec(key_vec_size);
  thrust::host_vector<int> h_keys_type(key_num);
  for (uint32_t i = 0, l = 0; i < keys.size(); i++, l++) {
    Column *column = input_tbl->columns[keys[i]];
    switch (column->type) {
      case LONG: {
        h_keys_type[i] = 0;
        h_keys_vec[l] = column->data_ptr;
        break;
      }
      case DOUBLE: {
        h_keys_type[i] = 1;
        h_keys_vec[l] = column->data_ptr;
        break;
      }
      case INT: {
        h_keys_type[i] = 2;
        h_keys_vec[l] = column->data_ptr;
        break;
      }
      case STRING: {
        h_keys_type[i] = 3;
        h_keys_vec[l++] = column->data_ptr;
        h_keys_vec[l] = column->data_ptr_aux;
        break;
      }
      case DEPEND: {
        break;
      }
    }
  }

  const void **h_keys_data_ptr = h_keys_vec.data();
  const int *h_keys_type_ptr = h_keys_type.data();

  thrust::host_vector<uint32_t> result_sorted_idx(row_num);
  thrust::sequence(result_sorted_idx.begin(), result_sorted_idx.end());
  thrust::sort(result_sorted_idx.begin(), result_sorted_idx.end(),
               sort_comparator(h_keys_data_ptr, h_keys_type_ptr, key_num));
  std::cout << "GHive-CPP [GroupByPredicate-execute]: Finish cpu sorting" << std::endl;
  profiler.end_sort();
  profiler.start_agg();
  thrust::host_vector<uint32_t> h_result_keys_idx;

  struct reducer_predicator rp(h_keys_data_ptr, h_keys_type_ptr, key_num);
  std::vector<Column *> result_columns;

  for (int32_t i = 0; i < aggregation_descs.size(); i++) {
    AggregationDesc desc = aggregation_descs[i];
    std::cout << "GHive-CPP [GroupByPredicate-execute]: desc_index: " << desc.index << std::endl;
    Column *column = input_tbl->columns[desc.index];
    switch (desc.type) {
      case SUM: {
        switch (column->type) {
          case LONG: {
            result_columns.push_back(aggregation_sum_cpu<long>(result_sorted_idx,
                                                               h_result_keys_idx,
                                                               column,
                                                               row_num,
                                                               rp));
            break;
          }
          case DOUBLE: {
            result_columns.push_back(aggregation_sum_cpu<double>(result_sorted_idx,
                                                                 h_result_keys_idx,
                                                                 column,
                                                                 row_num,
                                                                 rp));
            break;
          }
          case INT: {
            result_columns.push_back(aggregation_sum_cpu<int32_t>(result_sorted_idx,
                                                                  h_result_keys_idx,
                                                                  column,
                                                                  row_num,
                                                                  rp));
            break;
          }
          case STRING:
          case DEPEND:
            std::cout << "GHive-CPP-ERROR [GroupByPredicate-execute]: Unsupported type for SUM operation"
                      << column->type << std::endl;
            break;
        }
        break;
      }
      case MAX: {
        break;
      }
      case MIN: {
        break;
      }
      case AVG: {
        break;
      }
      case CNT: {
        break;
      }
      case RANK: {
        break;
      }
      case UNKNOWN: {
        break;
      }
    }

  }
  profiler.end_agg();

  return nullptr;
}

Column *GroupByPredicate::no_aggregation(thrust::device_vector<uint32_t> &d_result_idx,
                                         thrust::host_vector<uint32_t> &result_key_idx,
                                         struct reducer_predicator &rp, SortGroupByProfiler &profiler) {

  std::cout << "GHive-CPP [GroupByPredicate-no_aggregation]" << std::endl;

  profiler.start_agg();
  profiler.start_gpu_exec();
  auto new_end = thrust::unique(d_result_idx.begin(), d_result_idx.end(), rp);
  profiler.end_gpu_exec();
  profiler.start_agg();

  profiler.start_pci_device2host();
  result_key_idx = thrust::host_vector<int32_t>(d_result_idx.begin(), new_end);
  profiler.end_pci_device2host();

  return nullptr;

}

template<typename T>
Column *GroupByPredicate::aggregation_sum(thrust::device_vector<uint32_t> &d_result_idx,
                                          thrust::host_vector<uint32_t> &result_key_idx,
                                          Column *column, uint32_t row_num,
                                          struct reducer_predicator &rp, SortGroupByProfiler &profiler) {
  std::cout << "GHive-CPP [GroupByPredicate-aggregation_sum]" << std::endl;
  T *p_data = (T *) column->data_ptr;
  profiler.start_pci_host2device();
  thrust::device_vector<T> d_input_values(p_data, p_data + row_num);
  profiler.end_pci_host2device();
  if (keys.size() != 0) {
    profiler.start_gpu_alloc();
    thrust::device_vector<int> d_result_keys_idx(row_num);
    thrust::device_vector<T> d_result_values(row_num);
    profiler.end_gpu_alloc();
    profiler.start_agg();
    profiler.start_gpu_exec();
    auto new_end = thrust::reduce_by_key(d_result_idx.begin(), d_result_idx.end(),
                                         thrust::make_permutation_iterator(d_input_values.begin(),
                                                                           d_result_idx.begin()),
                                         d_result_keys_idx.begin(), d_result_values.begin(), rp,
                                         agg_plus<T>());
    profiler.end_gpu_exec();
    profiler.end_agg();
    profiler.start_data_recover();
    profiler.start_cpu_alloc();
    result_key_idx = thrust::host_vector<int32_t>(d_result_keys_idx.begin(), new_end.first);
    uint32_t result_size = new_end.second - d_result_values.begin();
    T *p_result = new T[result_size];
    profiler.end_cpu_alloc();
    profiler.end_data_recover();
    profiler.start_pci_device2host();
    hipMemcpy(p_result, thrust::raw_pointer_cast(d_result_values.data()),
               result_size * sizeof(T), hipMemcpyDeviceToHost);
    profiler.end_pci_device2host();
    return new Column(column->type, result_size, p_result);
  } else {
    T *p_result = new T[1];
    profiler.start_agg();
    profiler.start_gpu_exec();
    p_result[0] = thrust::reduce(d_input_values.begin(), d_input_values.end(), (T) 0.0, agg_plus<T>());
    profiler.end_gpu_exec();
    profiler.end_agg();
    return new Column(column->type, 1, p_result);
  }
}

template<typename T>
Column *GroupByPredicate::aggregation_max(thrust::device_vector<uint32_t> &d_result_idx,
                                          thrust::host_vector<uint32_t> &result_key_idx,
                                          Column *column,
                                          uint32_t row_num,
                                          struct reducer_predicator &rp,
                                          SortGroupByProfiler &profiler) {
  T *p_data = (T *) column->data_ptr;
  profiler.start_pci_host2device();
  thrust::device_vector<T> d_input_values(p_data, p_data + row_num);
  profiler.end_pci_host2device();
  if (keys.size() != 0) {
    profiler.start_gpu_alloc();
    thrust::device_vector<int> d_result_keys_idx(row_num);
    thrust::device_vector<T> d_result_values(row_num);
    profiler.end_gpu_alloc();
    profiler.start_agg();
    auto new_end = thrust::reduce_by_key(d_result_idx.begin(), d_result_idx.end(),
                                         thrust::make_permutation_iterator(d_input_values.begin(),
                                                                           d_result_idx.begin()),
                                         d_result_keys_idx.begin(), d_result_values.begin(), rp,
                                         agg_maximum<T>());
    profiler.end_agg();
    profiler.start_cpu_alloc();
    result_key_idx = thrust::host_vector<int32_t>(d_result_keys_idx.begin(), new_end.first);
    uint32_t result_size = new_end.second - d_result_values.begin();
    T *p_result = new T[result_size];
    profiler.end_cpu_alloc();
    profiler.start_pci_device2host();
    hipMemcpy(p_result, thrust::raw_pointer_cast(d_result_values.data()),
               result_size * sizeof(T), hipMemcpyDeviceToHost);
    profiler.end_pci_device2host();
    return new Column(column->type, result_size, p_result);
  } else {
    T *p_result = new T[1];
    profiler.start_agg();
    p_result[0] = thrust::reduce(d_input_values.begin(), d_input_values.end(), 0, agg_maximum<T>());
    profiler.end_agg();
    return new Column(column->type, 1, p_result);
  }
}

template<typename T>
Column *GroupByPredicate::aggregation_min(thrust::device_vector<uint32_t> &d_result_idx,
                                          thrust::host_vector<uint32_t> &result_key_idx,
                                          Column *column, uint32_t row_num,
                                          struct reducer_predicator &rp, SortGroupByProfiler &profiler) {
  T *p_data = (T *) column->data_ptr;
  profiler.start_pci_host2device();
  thrust::device_vector<T> d_input_values(p_data, p_data + row_num);
  profiler.end_pci_host2device();
  if (keys.size() != 0) {
    profiler.start_gpu_alloc();
    thrust::device_vector<int> d_result_keys_idx(row_num);
    thrust::device_vector<T> d_result_values(row_num);
    profiler.end_gpu_alloc();
    profiler.start_agg();
    auto new_end = thrust::reduce_by_key(d_result_idx.begin(), d_result_idx.end(),
                                         thrust::make_permutation_iterator(d_input_values.begin(),
                                                                           d_result_idx.begin()),
                                         d_result_keys_idx.begin(), d_result_values.begin(), rp,
                                         agg_minimum<T>());
    profiler.end_agg();
    profiler.start_cpu_alloc();
    result_key_idx = thrust::host_vector<int32_t>(d_result_keys_idx.begin(), new_end.first);
    uint32_t result_size = new_end.second - d_result_values.begin();
    T *p_result = new T[result_size];
    profiler.end_cpu_alloc();
    profiler.start_pci_device2host();
    hipMemcpy(p_result, thrust::raw_pointer_cast(d_result_values.data()),
               result_size * sizeof(T), hipMemcpyDeviceToHost);
    profiler.end_pci_device2host();
    return new Column(column->type, result_size, p_result);
  } else {
    T *p_result = new T[1];
    p_result[0] = thrust::reduce(d_input_values.begin(), d_input_values.end(), 0, agg_minimum<T>());
    return new Column(column->type, 1, p_result);
  }
}

template<typename T>
Column *GroupByPredicate::aggregation_avg(thrust::device_vector<uint32_t> &d_result_idx,
                                          thrust::host_vector<uint32_t> &result_key_idx,
                                          Column *column,
                                          uint32_t row_num,
                                          struct reducer_predicator &rp,
                                          SortGroupByProfiler &profiler) {
  T *p_data = (T *) column->data_ptr;
  profiler.start_pci_host2device();
  thrust::device_vector<T> d_input_values(p_data, p_data + row_num);
  profiler.end_pci_host2device();
  if (keys.size() != 0) {
    profiler.start_gpu_alloc();
    thrust::device_vector<int32_t> d_result_cnt(row_num);
    thrust::device_vector<int> d_result_keys_idx(row_num);
    profiler.end_gpu_alloc();
    profiler.start_agg();
    auto cnt_end = thrust::reduce_by_key(d_result_idx.begin(), d_result_idx.end(),
                                         thrust::make_permutation_iterator(d_input_values.begin(),
                                                                           d_result_idx.begin()),
                                         d_result_keys_idx.begin(), d_result_cnt.begin(),
                                         rp, agg_plus<int32_t>());
    profiler.end_agg();
    profiler.start_gpu_alloc();
    thrust::device_vector<double> d_avg_result(row_num);
    profiler.end_gpu_alloc();
    profiler.start_agg();
    auto new_end = thrust::reduce_by_key(d_result_idx.begin(), d_result_idx.end(),
                                         thrust::make_permutation_iterator(d_input_values.begin(),
                                                                           d_result_idx.begin()),
                                         d_result_keys_idx.begin(), d_avg_result.begin(),
                                         rp, agg_plus<T>());
    thrust::transform(d_avg_result.begin(), new_end.second, d_result_cnt.begin(),
                      d_avg_result.begin(), divides<double>());
    profiler.end_agg();
    profiler.start_cpu_alloc();
    result_key_idx = thrust::host_vector<int32_t>(d_result_keys_idx.begin(), new_end.first);
    uint32_t result_size = cnt_end.second - d_result_cnt.begin();
    profiler.end_cpu_alloc();
    profiler.start_pci_device2host();
    double *p_result = new double[result_size];
    hipMemcpy(p_result, thrust::raw_pointer_cast(d_avg_result.data()),
               result_size * sizeof(double), hipMemcpyDeviceToHost);
    profiler.end_pci_device2host();
    return new Column(column->type, result_size, p_result);
  } else {
    T *p_result = new T[1];
    profiler.start_agg();
    T sum = thrust::reduce(d_input_values.begin(), d_input_values.end(), 0, agg_plus<T>());
    T cnt = thrust::reduce(d_input_values.begin(), d_input_values.end(), 0, agg_cnt<T>());
    profiler.end_agg();
    p_result[0] /= sum / cnt;
    return new Column(column->type, 1, p_result);
  }
}

Column *GroupByPredicate::aggregation_cnt(thrust::device_vector<uint32_t> &d_result_idx,
                                          thrust::host_vector<uint32_t> &result_key_idx,
                                          uint32_t row_num,
                                          struct reducer_predicator &rp,
                                          SortGroupByProfiler &profiler) {
  if (keys.size() != 0) {
    profiler.start_gpu_alloc();
    thrust::device_vector<int> d_result_keys_idx(row_num);
    thrust::device_vector<int32_t> d_result_values(row_num);
    profiler.end_gpu_alloc();
    profiler.start_agg();
    auto
        new_end = thrust::reduce_by_key(d_result_idx.begin(), d_result_idx.end(), thrust::constant_iterator<int32_t>(1),
                                        d_result_keys_idx.begin(), d_result_values.begin(), rp,
                                        thrust::plus<int32_t>());
    profiler.end_agg();
    profiler.start_cpu_alloc();
    result_key_idx = thrust::host_vector<int32_t>(d_result_keys_idx.begin(), new_end.first);
    uint32_t result_size = new_end.second - d_result_values.begin();
    int32_t *p_result = new int32_t[result_size];
    profiler.end_cpu_alloc();
    profiler.start_pci_device2host();
    hipMemcpy(p_result, thrust::raw_pointer_cast(d_result_values.data()),
               result_size * sizeof(int32_t), hipMemcpyDeviceToHost);
    profiler.end_pci_device2host();
    return new Column(INT, result_size, p_result);
  } else {
    int32_t *p_result = new int32_t[1];
    p_result[0] = row_num;
    return new Column(INT, 1, p_result);
  }
}

template<typename T>
Column *GroupByPredicate::aggregation_sum_cpu(thrust::host_vector<uint32_t> &d_result_idx,
                                              thrust::host_vector<uint32_t> &result_key_idx,
                                              Column *column,
                                              uint32_t row_num,
                                              reducer_predicator &rp) {
  std::cout << "GHive-CPP [GroupByPredicate-aggregation_sum]" << std::endl;
  T *p_data = (T *) column->data_ptr;
  if (keys.size() != 0) {
    thrust::host_vector<int> h_result_keys_idx(row_num);
    T *result_values = new T[row_num];
    auto new_end = thrust::reduce_by_key(d_result_idx.begin(), d_result_idx.end(),
                                         thrust::make_permutation_iterator(p_data,
                                                                           d_result_idx.begin()),
                                         h_result_keys_idx.begin(), result_values, rp,
                                         agg_plus<T>());
    result_key_idx = thrust::host_vector<int32_t>(h_result_keys_idx.begin(), new_end.first);
    uint32_t result_size = new_end.second - result_values;

    return new Column(column->type, result_size, result_values);
  } else {
    T *p_result = new T[1];
    p_result[0] = thrust::reduce(p_data, p_data + row_num, 0, agg_plus<T>());
    return new Column(column->type, 1, p_result);
  }

}


