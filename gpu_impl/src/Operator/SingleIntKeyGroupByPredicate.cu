#include <iostream>
#include <unordered_map>
#include <Profile/Profiler.hpp>
#include <DataFlow/Table.hpp>
#include <Util/Util.hpp>
#include <Profile/SortGroupByProfiler.hpp>
//#include "Util/Util.hpp"
#include "Operator/SingleIntKeyGroupByPredicate.hpp"
#include "Operator/GroupByAggregation.cuh"
#include "Operator/GroupByOperator.cuh"
#include "Operator/GroupByPredicate.hpp"
using namespace std;

SingleIntKeyGroupByPredicate::SingleIntKeyGroupByPredicate() {}

Table *SingleIntKeyGroupByPredicate::gpu_execute(Table *input_tbl, SortGroupByProfiler &profiler) {

  std::cout << "GHive-CPP [GroupByPredicate-execute]: gpu_execute starts executing" << std::endl;
  std::cout << "GHive-CPP [GroupByPredicate-execute]: keys: " << vector_to_string<uint32_t>(keys) << std::endl;
  uint32_t row_num = input_tbl->row_num;
  uint32_t key_num = keys.size();//the number of group by columns.
  assert(key_num == 1);
  assert(input_tbl->columns[keys[0]]->type == INT);
  uint32_t value_num = aggregation_descs.size();



  std::cout << "GHive-CPP [GroupByPredicate-execute]: Collect data and prepare sorting" << std::endl;
  profiler.start_gpu_alloc();
  profiler.start_pci_host2device();

  int32_t *d_int_ptr;
  hipMalloc((void **) &d_int_ptr, row_num * sizeof(int32_t));
  hipMemcpy(d_int_ptr, input_tbl->columns[keys[0]]->data_ptr, row_num * sizeof(int32_t), hipMemcpyHostToDevice);

  struct single_int_key_sort_comparator sort_comparator(d_int_ptr);
  struct single_int_key_comparator comparator(d_int_ptr);

  thrust::device_vector<uint32_t> d_result_idx(row_num);
  thrust::sequence(d_result_idx.begin(), d_result_idx.end());


  profiler.end_gpu_alloc();
  profiler.end_pci_host2device();
  profiler.start_sort();
  profiler.start_gpu_exec();
  thrust::sort(d_result_idx.begin(), d_result_idx.end(), sort_comparator);

  thrust::host_vector<int> tmp = d_result_idx;
  for (int i: tmp) {
      std::cout << i << std::endl;
  }
  profiler.end_gpu_exec();
  profiler.end_sort();

  std::cout << "GHive-CPP [GroupByPredicate-execute]: Finish gpu sorting" << std::endl;

  thrust::host_vector<uint32_t> h_result_keys_idx;

  std::vector<Column *> result_columns;

  AggregationDesc desc = aggregation_descs[0];
  assert(desc.type == SUM);
  result_columns.push_back(agg_sum(d_result_idx,
    h_result_keys_idx,
    input_tbl->columns[desc.index],
    row_num,
    comparator,
    profiler));

  Table *result_tbl = new Table();
  result_tbl->columns = result_columns;
  result_tbl->row_num = h_result_keys_idx.size();

  profiler.end_data_recover();
  return result_tbl;
}


Column *SingleIntKeyGroupByPredicate::agg_sum(thrust::device_vector<uint32_t> &d_result_idx,
                                          thrust::host_vector<uint32_t> &result_key_idx,
                                          Column *column, uint32_t row_num,
                                          struct single_int_key_comparator &rp, SortGroupByProfiler &profiler) {
  std::cout << "GHive-CPP [GroupByPredicate-aggregation_sum]" << std::endl;
  int *p_data = (int *) column->data_ptr;
  thrust::device_vector<int> d_input_values(p_data, p_data + row_num);
  thrust::device_vector<int> d_result_keys_idx(row_num);
  thrust::device_vector<int> d_result_values(row_num);
  auto new_end = thrust::reduce_by_key(d_result_idx.begin(), d_result_idx.end(),
                                         thrust::make_permutation_iterator(d_input_values.begin(),
                                                                           d_result_idx.begin()),
                                         d_result_keys_idx.begin(), d_result_values.begin(), rp,
                                         agg_plus<int>());
  result_key_idx = thrust::host_vector<int32_t>(d_result_keys_idx.begin(), new_end.first);
  uint32_t result_size = new_end.second - d_result_values.begin();
  int *p_result = new int[result_size];
  hipMemcpy(p_result, thrust::raw_pointer_cast(d_result_values.data()),
             result_size * sizeof(int), hipMemcpyDeviceToHost);
  return new Column(column->type, result_size, p_result);
}

