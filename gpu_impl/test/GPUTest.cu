#include <gtest/gtest.h>
#include <thrust/transform.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/device_vector.h>

using namespace  std;
class GPUTest : public ::testing::Test {

};


struct c2l {


    char *start;

    c2l(char *_start): start(_start) {}

    __host__ __device__
    long operator() (int i) {
        return *(long *)start;

    }
};

TEST_F(GPUTest, EndianTest) {

    char c[8] = {1, 1, 0, 0, 0, 0, 0, 0};
    char *dDataPtr;
    hipMalloc((void **) &dDataPtr, sizeof(char) * 8);
    hipMemcpy(dDataPtr, c, sizeof(char) * 8, hipMemcpyHostToDevice);


    thrust::counting_iterator<int> iter(0);
    thrust::device_vector<long> res(1);
    thrust::transform(iter, iter + 1, res.begin(), c2l(dDataPtr));
    std::cout << res[0] << std::endl;


    char x[8] = {1};
    long *a = (long *)x;
    std::cout << *a << std::endl;





}
