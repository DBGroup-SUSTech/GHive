#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include "Operator/Base64.h"
#include "Operator/RSA.h"
#include "DataFlow/Table.hpp"
#include "Operator/SelectPredicate.hpp"
#include <chrono>
#include <fstream>
#include "Operator/udf/RSADecrypt.h"
#include "Util/Util.hpp"



using namespace  std;
class RSATest : public ::testing::Test {



};



TEST_F(RSATest, RSATest) {
    char exp[128] = {107, -82, -91, -54, 88, 72, -25, 12, -34, 65, -54, -59, 124, 6, -88, -19, 51, -84, 105, -73, 79, 55, -40, -68, 115, 86, 72, -43, 46, 107, -115, 91, 41, -25, -90, 25, -8, -18, -66, -50, -16, -1, 106, 91, 60, 90, 14, -14, -40, 75, 95, -122, -22, -126, -8, -18, -51, 8, -50, -16, -23, 103, 71, -20, 83, 66, 60, 63, -32, 81, -77, 18, 2, -14, -28, 53, -66, 4, 31, -125, -44, 115, 49, 51, -86, 6, 54, 80, 18, 114, 78, 78, -40, 123, -63, -18, -86, -24, 2, 84, -6, -116, 24, -3, -111, 100, 43, -70, 55, 13, 45, 94, 36, -123, -83, -16, -39, -35, 96, -97, 91, 39, 80, 10, 56, 8, -102, 57};
    char mod[128] = {-112, 10, -96, 10, 91, 74, 30, -58, 90, 77, -36, 28, 42, -24, -42, 53, 108, -61, -88, 50, 10, 108, -55, 118, 46, -81, 101, -27, -50, -56, 122, 2, -79, 58, -19, -43, -31, 98, 109, -4, -62, 104, 117, 25, 59, 10, 79, -115, 85, -9, 8, 97, -13, 82, -64, -80, -18, -28, 16, -6, -18, -12, -27, -8, -27, 33, -80, 120, 7, 79, -32, 113, -62, -121, 127, -128, -7, -85, -59, -33, 31, 123, 66, 4, 26, 51, 84, -103, -123, 18, -47, 67, 85, -86, 5, -120, -110, -128, -82, 120, -117, 82, -40, -56, -122, -25, -14, 97, 68, -62, 94, 69, -120, 37, -60, -102, 103, 30, 83, 39, 10, -62, 32, 76, -39, 51, -58, 61};

    std::string Exponent = "75617024498692365584385751462342930957698313562765988763410623235194287559492097847431217744113616978356293714871925947116894300288386889742408415109822044974996134092237936326927814553796840211041819295015584421019306153830736962213019023732670655511549016137442333404992390016420132441347998671388482771513";
    std::string Modulus = "101149384303604554910884781679135581217591427031126150254881046964218639502519361893846179476996655017155905237203768028660650319590154133017271916990905179280478495022740066538101553337912020856443802362559881457536975295984320051395381431923531695001924526297612284788206607224582504897188631446446010517053";
//    std::string cipherText = "iqyntVuXixaY3QkN6eYVVC+Tf4eeDtJAX9eonxpjXdKNIeeGNITmRdB/+NyyJDK8qO8hJpEiC2OoxDS64aUgcJJH1Pv3vmHE0YqT9a8AyLJ3sxGV4ivahUN8pGvdhSTQazfRzVpaGqcUTS92GgWjvWb/kHzQskwBC5FLkkeDj/8=";
    std::string cipherText = "fbUwlfNfDic7V+Wh6DCsfctVsFVaSh+FuShiCOOJC+kYz/eLjNVHQBYAFHv9LykRzrW6ayQTSHKjPjy8VTSKbQg0VAxIUR3XxIQD0uu2wd80+Xx7X2xZD9NIEB2HVRBEmKzSWtLIKU/6k/TYFPhQcS8h90puRvGmZru91rXmFGo=";

    char cipherBuffer[128] = {0};
    macaron::Base64::Decode(cipherText, cipherBuffer);

    char tmp_exp[128], tmp_mod[128], tmp_ciph[128];
    std::cout << "char exp[128] = {";
    for (int i = 0; i < 128; i ++) {
        std::cout << (int) exp[127 - i];
        std::cout << (i == 127 ? "}": ",");
    }
    std::cout << std::endl;
    std::cout << "char mod[128] = {";
    for (int i = 0; i < 128; i ++) {
        std::cout << (int) mod[127 - i];
        std::cout << (i == 127 ? "}": ",");
    }


    for (int i = 0; i < 128; i ++) {
        tmp_exp[i] = exp[127 - i];
        tmp_mod[i] = mod[127 - i];
        tmp_ciph[i] = cipherBuffer[127 - i];
    }
    for (int i = 0; i < 128; i ++) {
        exp[i] = tmp_exp[i];
        mod[i] = tmp_mod[i];
        cipherBuffer[i] = tmp_ciph[i];
    }


    for (int i = 0; i < 128 ; i ++) {
        std::cout << (int)cipherBuffer[i];
    }
    char res[128] = {0};


    auto start = std::chrono::system_clock::now();
    rsa1024(reinterpret_cast<unsigned long *>((long *) res), reinterpret_cast<unsigned long *>(cipherBuffer),
            reinterpret_cast<unsigned long *>(exp), reinterpret_cast<unsigned long *>(mod));

    auto end = std::chrono::system_clock::now();
    std::cout << std::endl;
    std::cout << "result: ";
    for (int i = 0; i <128; i ++) {
        std::cout << (int)res[i];
    }

    std::cout << "time: " << std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() <<std::endl;

//    std::cout << std::endl;
//        int a=1;
//        char b =*(char*)(&a);
//        std::cout << "b: " << b;

}




TEST_F(RSATest, simpleDescrpytTest) {

    Table *tbl = new Table();
    tbl->row_num = 1;

    std::string cipherText = "fbUwlfNfDic7V+Wh6DCsfctVsFVaSh+FuShiCOOJC+kYz/eLjNVHQBYAFHv9LykRzrW6ayQTSHKjPjy8VTSKbQg0VAxIUR3XxIQD0uu2wd80+Xx7X2xZD9NIEB2HVRBEmKzSWtLIKU/6k/TYFPhQcS8h90puRvGmZru91rXmFGo=";

    char cipherBuffer[128] = {0};
    macaron::Base64::Decode(cipherText, cipherBuffer);

    char *strCol = new char[128];
    int *strIdxCol = new int[2];

    for (int i = 0; i < 128; i++) {
        strCol[i] = cipherBuffer[127 - i];
    }
    strIdxCol[0] = 0;
    strIdxCol[1] = 127;


    tbl->columns.push_back(new Column(STRING, 1, strCol, strIdxCol, 128));
    std::cout << tbl->toString(1, true) << std::endl;

    FunctorSelectPredicate predicate(rsa_decrypt_functor(), 0);
    auto start = std::chrono::system_clock::now();
    predicate.process(tbl);
    auto end = std::chrono::system_clock::now();
    std::cout << "time: " << std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() <<std::endl;
    tbl->columns.push_back(predicate.column);

    std::cout << tbl->toString(10) << std::endl;

//    tbl->columns.push_back(predicate.column);
//    std::cout << tbl->toString() << std::endl;

}



TEST_F(RSATest, MultipleDescrpytTest) {

    std::ifstream input( "number.txt" );
    std::vector<std::string> lines;
    for( std::string line; getline( input, line ); )
    {
        std::vector<std::string> each_line_elems;
        split(line, each_line_elems, ",");
        lines.push_back(each_line_elems[2]);
    }


    int row_num = 1000000;
    int char_size = row_num * 128;

    char *strCol = new char[char_size];
    int *strIdxCol = new int[row_num * 2];

    for (int i = 0; i < row_num; i++) {
        if (i % 100000 == 0) {
            std::cout << "number: " << i << std::endl;
        }
        strIdxCol[2 * i] = i * 128;
        strIdxCol[2 * i + 1] = i * 128 + 128;

        char cipherBuffer[128] = {0};
        macaron::Base64::Decode(lines[i], cipherBuffer);
        for (int j = 0; j < 128; j++) {
            strCol[128 * i + j] = cipherBuffer[127 - j];
        }
    }


    Table *tbl = new Table();
    tbl->row_num = row_num;

    tbl->columns.push_back(new Column(STRING, row_num, strCol, strIdxCol, char_size));
//    std::cout << tbl->toString(10, true) << std::endl;

    FunctorSelectPredicate predicate(rsa_decrypt_functor(), 0);
    auto start = std::chrono::system_clock::now();
    predicate.process(tbl);
    auto end = std::chrono::system_clock::now();
    std::cout << "time: " << std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() <<std::endl;
    tbl->columns.push_back(predicate.column);

//    std::cout << tbl->toString(10) << std::endl;

//    tbl->columns.push_back(predicate.column);

}
