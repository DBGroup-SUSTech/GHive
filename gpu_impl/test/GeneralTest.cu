#include <gtest/gtest.h>
#include <regex>
#include <Profile/Profiler.hpp>
#include <thrust/device_vector.h>

using namespace  std;
class GeneralTest : public ::testing::Test {

};

TEST_F(GeneralTest, shouldSuccess) {

  std::cout << "GeneralTest-shouldSuccess" << std::endl;

  std::string conds("SEL_51._col1=RS_49._col0(Inner)");
  std::regex split_re("[\\.=]");
  std::vector<std::string> v(std::sregex_token_iterator(conds.begin(), conds.end(), split_re, -1),
                             std::sregex_token_iterator());



  std::vector<std::string> children_names;
  for (std::string cond_split: v) {
    std::cout << cond_split  << std::endl;
    if (std::regex_match(cond_split, std::regex("[A-Z]+_[0-9]+"))) {
//      if (find(children_names.begin(), children_names.end(), cond_split) != children_names.end()) {
//        break;
//      }
      children_names.push_back(cond_split);
    }
  }
  for (std::string s: children_names) {
    std::cout << s << ";";
  }
}

TEST_F(GeneralTest, performanceTest) {
  uint32_t ROW_NUM = 300000000;
  Profiler profiler;

  profiler.start_op();
  int32_t *d_data;
//  hipMalloc((void **)&d_data, ROW_NUM * sizeof(int32_t));
  thrust::host_vector<int32_t> vec(ROW_NUM);
  profiler.end_op();
  std::cout << profiler.toString() << std::endl;



}